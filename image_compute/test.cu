#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>      /// From "cufft.lib" 
#include "hip/hip_runtime.h"
#include <fstream>
#include "testheader.cuh"
#include ""
#include <chrono>
using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::chrono::milliseconds;


__global__ void range(int* out_data, int min, int arr_size, int inc)	//creates an array of a range of values
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				//max=final value of the array
	if (i < arr_size)																//arr_size==array size
	{																			//inc=increment needed
		out_data[i] = min + (i * inc);
	}
}

__global__ void range(float* out_data, float min, int arr_size, float inc)	//creates an array of a range of values
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				//max=final value of the array
	if (i < arr_size)																//arr_size==array size
	{																			//inc=increment needed
		out_data[i] = min + (i * inc);
	}
}

__global__ void range(double* out_data, double min, int arr_size, double inc)	//creates an array of a range of values
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				//max=final value of the array
	if (i < arr_size)																//arr_size==array size
	{																			//inc=increment needed
		out_data[i] = min + (i * inc);
	}
}

__global__ void range(long double* out_data, long double min, int arr_size, long double inc)	//creates an array of a range of values
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				//max=final value of the array
	if (i < arr_size)																//arr_size==array size
	{																			//inc=increment needed
		out_data[i] = min + (i * inc);
	}
}

__global__ void element_division(float* mat_in, float value, int size, float* mat_out)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				   //max=final value of the array
	if (i < size)
	{
		mat_out[i] = mat_in[i] / value;
	}
}

__global__ void element_division(long double* mat_in, float value, int size, long double* mat_out)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				   //max=final value of the array
	if (i < size)
	{
		mat_out[i] = mat_in[i] / value;
	}
}

__global__ void aperture_distance(float* mat1, float* mat2, int Nx, int channels, float* mat_out)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	int i = x / channels;
	int j = x % channels;

	if (x < Nx * channels)
	{
		mat_out[i * channels + j] = fabs(mat1[i] - mat2[j]);
	}
}

__global__ void apodization(float* distance, float* aperture, int Nz, int Nx, int channels, int pixels, float* apod)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int i = x / Nz;
	int ii = i % Nx;
	int j = x % Nz;
	int nrx = x / pixels;
	float PI = 3.14159;

	if (x < pixels * channels)
	{
		bool temp = distance[ii * channels + nrx] <= (aperture[j] / 2);
		apod[i * Nz + j] = temp * (0.5 + 0.5 * cos(2 * PI * distance[ii * channels + nrx] / aperture[j]));
	}
}

// This function calculates TX central aperture position
__global__ void Tx_cen_pos(float* cen_pos, int N_elements, int N_active, float pitch, int skip_frames, int num_frames, float* probe)
{

	int x = threadIdx.x;

	if (x < num_frames)
	{
		//cen_pos[x] = pitch * ((N_active / 2) + (N_active * (x)-N_elements / 2));
		cen_pos[x] = probe[x * skip_frames + 4];
	}
}

// receive_delay calculation
__global__ void receive_delay(float* probe_ge_x, float* x_axis1, float* z_axis1, int channels, int Nx, int Nz, float del_convert, float* rx_delay)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;

	if (x < Nx * Nz * channels)
	{
		int i = x / Nz;
		int ii = i % Nx;
		int j = x % Nz;
		int nrx = x / (Nx * Nz);
		rx_delay[i * Nz + j] = (sqrt((probe_ge_x[nrx] - x_axis1[ii]) * (probe_ge_x[nrx] - x_axis1[ii]) + ((z_axis1[j]) * (z_axis1[j])))) * del_convert;
		// 1867 - 210 = 1657
		//rx_delay[i * Nx + j] = sqrt(rc * rc + (rc + z_axis[ii]) * (rc + z_axis[ii]) - 2 * rc * (rc + z_axis[ii]) * cos(theta[nrx] - theta1[j])) * del_convert;
	}
}

//  transmit_delay calculation
__global__ void transmit_delay(float* x_axis1, float* z_axis1, float* k1, float zd, int Nx, int Nz, float del_convert, int num_frames, float* tx_delay)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int i = x / Nz;
	int ii = i % Nx;
	int j = x % Nz;
	int f = x / (Nx * Nz);

	if (x < Nx * Nz * num_frames)
	{
		tx_delay[i * Nz + j] = (sqrt(((k1[f] - x_axis1[ii]) * (k1[f] - x_axis1[ii])) + ((zd + z_axis1[j]) * (zd + z_axis1[j])))) * del_convert;
		// 1875-210 = 1665
		//tx_delay[i * Nx + j] = (zd + sqrt(rc * rc + (rc + z_axis[ii]) * (rc + z_axis[ii]) - 2 * rc * (rc + z_axis[ii]) * cos(theta_tx[f] - theta1[j]))) * del_convert;
		//first 256*1024 for frame 1, next 256*1024 for frame 2........
	}
}

__global__ void beamformingLR3(float* beamformed_data1, float* tx_delay, float* rx_delay, float* data, float* rx_apod, int samples, int pixels, int f, int num_frames, int channels)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	int nrx = x / pixels;   // nrx - nth A-line
	int pix = x & (pixels - 1); // x% pixels;     // pixel location

	int pixel_pos = round((float)tx_delay[f * pixels + pix] + (float)rx_delay[x]);   // delay value estimation from tx and rx delay values

	if (pixel_pos < samples)
	{
		beamformed_data1[pix] += rx_apod[x] * data[(nrx * samples + pixel_pos - 1)];   // Extract data based on the delay values and multiplying with apodization value
	}
}

__global__ void isnan_test_array(float* data, int size)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < size)
	{
		if (isnan(data[idx]) == 1)
		{
			data[idx] = 0;
		}
		else
		{
			data[idx] = data[idx];
		}

	}


}

__global__ void BPfilter1SharedMem(float* in, float* filt_coeff, int pixels, float* y1) {

	const int TILE_SIZE = 4;
	int MASK_WIDTH = 364;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float N_s[TILE_SIZE];
	N_s[threadIdx.x] = in[x];
	__syncthreads();

	int PtileStartPt = blockIdx.x * blockDim.x;
	int NtileStartPt = (blockIdx.x + 1) * blockDim.x;
	int n_start_pt = x - (MASK_WIDTH / 2);

	float temp = 0;

	for (int j = 0; j < MASK_WIDTH; j++) {
		int N_index = n_start_pt + j;

		if (N_index >= 0 && N_index < pixels) {
			if ((N_index >= PtileStartPt) && (N_index < NtileStartPt)) {
				temp += N_s[threadIdx.x + j - (MASK_WIDTH / 2)] * filt_coeff[j];
			}
			else {
				temp += in[N_index] * filt_coeff[j];
			}
		}
	}
	y1[x] = temp;
}

__global__ void reshape_columnwise(int col, int row, float* beamformed_data_reshaped, float* d_bfHR)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	int i = x / row;
	int j = x % row;

	if (x < col * row)
	{
		beamformed_data_reshaped[j * col + i] = d_bfHR[x];

	}
}

//** <Curvilinear Prob> **//

// receive_delay calculation
__global__ void receive_delay(float* theta, float* theta1, float rc, float* z_axis, int channels, int Nx, int Nz, float del_convert, float* rx_delay)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;

	if (x < Nz * Nx * channels)
	{
		int i = x / Nz;
		int ii = i % Nx;
		int j = x % Nz;
		int nrx = x / (Nx * Nz);
		rx_delay[i * Nz + j] = sqrt(rc * rc + (rc + z_axis[j]) * (rc + z_axis[j]) - 2 * rc * (rc + z_axis[j]) * cos(theta[nrx] - theta1[ii])) * del_convert;
	}
}

__global__ void theta1(float* theta_active, float* theta, int frames, int N_active, int skip_frames)
{

	int x = threadIdx.x;
	int f = 0;
	for (int i = 1; i <= frames; i += skip_frames)
	{
		theta_active[f] = theta[i + 3 - 1];
		f++;
	}
}

//  transmit_delay calculation
__global__ void transmit_delay(float* theta1, float* z_axis, float rc, float* theta_tx, int Nx, int Nz, float del_convert, int columns, float zd, float* tx_delay)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int i = x / Nz;
	int j = x % Nz;
	int f = x / (Nx * Nz);


	if (x < Nx * Nz * columns)
	{
		tx_delay[i * Nz + j] = (zd + sqrt(rc * rc + (rc + z_axis[j]) * (rc + z_axis[j]) - 2 * rc * (rc + z_axis[j]) * cos(theta_tx[f] - theta1[i % Nx]))) * del_convert;
	}
}

__global__ void add_ele(float* data, int pixels, float* out_data)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < pixels)
	{
		out_data[x] += data[x];
	}
}

__global__ void sample1(float* tx_delay, float* rx_delay, int pixels, int channels, float c, float sampling_frequency, float* data1, float* rx_apod, float* data, int samples, int columns)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int f = blockDim.y * blockIdx.y + threadIdx.y;
	int nrx = x / pixels; //channels
	int pix = x % pixels; //pixels


	if (f < columns)
	{
		float delay = ((float)tx_delay[f * pixels + pix] + (float)rx_delay[(nrx % channels) * pixels + pix]) / c;
		float p = delay * sampling_frequency;
		int pixel_pos = round(p);

		//data1 = rx_apod[(nrx % channels) + (pix * channels)] * data[((nrx % channels) * samples + pixel_pos - 1) * columns + f];
	}
}

__global__ void add_columns_matrix(float* data, int columns, int pixels, float* out_data)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < pixels)
	{
		for (int f = 0; f < columns; f++)
			out_data[x] += data[x * columns + f];
	}
}

__global__ void parallel_try(float* tx_delay, float* rx_delay, float sampling_frequency, float c, int samples,
	int channels, int columns, float* rx_apod, int pixels, float* data, float* beamformed_data)
{
	//__shared__ double* beamformed_data_1;
	int pix = blockDim.x * blockIdx.x + threadIdx.x;
	int nrx = blockDim.y * blockIdx.y + threadIdx.y;
	//int f = x / pixels; int pix = x % pixels; //int nrx = x % 128;
	int f = blockDim.z * blockIdx.z + threadIdx.z;
	//int nrx = x / pixels; //channels
	//int pix = x % pixels; //pixels
	//int f = x / (pixels * channels);

	if (f < columns && pix < pixels && nrx < channels)
	{
		//for (int nrx = 0; nrx < channels; nrx++)
		//{

		float delay = ((float)tx_delay[f * pixels + pix] + (float)rx_delay[(nrx)*pixels + pix]) / c;
		float p = delay * sampling_frequency;
		int pixel_pos = round(p);

		if ((0 < pixel_pos) && (pixel_pos < samples))
		{
			//double ans= beamformed_data[pix] + rx_apod[channels * nrx + pix] * data[nrx * 2600 + pixel_pos];
			beamformed_data[pix * columns + f] += rx_apod[nrx + (pix * channels)] * data[(nrx * samples + pixel_pos - 1) * columns + f];

		}
		//}
	}
}


__global__ void real2complex(float* f, hipfftComplex* fc) {
	//int i = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	fc[i].x = f[i];
	fc[i].y = 0.0f;
}

__global__ void splitComplex(hipfftComplex* inComplex, float* outReal, float* outImag) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	outReal[i] = inComplex[i].x;
	outImag[i] = inComplex[i].y;

}

__global__ void scalarMult(float* inArray, float* outArray, float c) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	outArray[i] = inArray[i] * c;

}

__global__ void magnitide(float* inX, float* inY, float* outW) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	outW[i] = std::sqrtf(std::pow(inX[i], 2) + std::pow(inY[i], 2));

}

__global__ void logCompresion(float* inArray, float* outArray, float c= 20.0) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	outArray[i] = c * std::log10(1 + inArray[i]);

}



//cudaBackEnd::cudaBackEnd() {
//
//	int num_threads = 1024;
//	float rx_f_number = 2.0;
//	int samples = 2040;						// # of samples in depth direction
//	int N_elements = 128;					// # of transducer elements
//	float sampling_frequency = 32e6;		// sampling frequency
//	float c = 1540.0;						// speed of sound [m/s]	
//	int N_active = 8;						// Active transmit elmeents
//	float pitch = 0.465 / 1000;				// spacing between the elements
//	float aper_len = (N_elements - 1) * pitch * 1000;	//aperture foot print 
//	float zd = pitch * N_active / (float)2;				// virtual src distance from transducer array 
//	float sample_spacing = c / sampling_frequency / (float)2;
//	float del_convert = sampling_frequency / c;			// used in delay calculation
//	float rc = 60.1 / 1000;					// radius_of_curvature
//	float scan_angle = (58 * PI) / 180;
//	int channels = 128;						// number of A-lines data used for beamforming
//	int Nx = 256;							// 256 Lateral spacing
//	int Nz = 1024;							//1024 Axial spacing
//	int pixels = Nz * Nx;
//	int pix_cha = pixels * channels;		// Nz*Nx*128 This array size is used for Apodization
//	int frames = 121;
//	int num_frames = 121;					// number of low resolution images
//	int skip_frames = 1;					// 
//	int dBvalue = 60;
//	float rximg[128 * 2040];
//	int croppedBot = 300;
//	//float* filt_coeff = new float[364];
//	//float* d_z_axis = 0;
//	//float* d_x_axis = 0;
//	//float* d_probe = 0;
//	//float* d_rx_aperture = 0;
//	//float* d_rx_ap_distance = 0;
//	//float* d_cen_pos = 0;
//	//float* d_data = 0;   // variable to store raw rf data
//	//float* d_bfHR = 0;  // variable to store beamformed high-resolution beamformed image 
//	//float* d_tx_delay = 0;
//	//float* d_rx_delay = 0;// delay calculation
//	//float* d_rx_apod = 0; //apodization
//	//float* d_filt_coeff = 0; //to read filter coeff CSV
//	//float* d_bfHRBP = 0;  // variable to store beamformed high-resolution bandpass filtered data
//	//float* dev_beamformed_data1 = 0;   // variable to store reshaped beamformed data
//	//float* env = new float[pixels]; // Host memory variable to store beamformed high-resolution bandpass filtered data
//	//float* d_theta = 0;
//	//float* d_theta1 = 0;
//	//float* d_theta_tx = 0;
//
//	USBDevice = new CCyUSBDevice(NULL);
//	// Obtain the control endpoint pointer
//	ept = USBDevice->ControlEndPt;
//	if (!ept) {
//		printf("Could not get Control endpoint.\n");
//		//return 1;
//	}
//
//}

int cudaBackEnd::num_threads = 0;
float cudaBackEnd::rx_f_number = 0;
float cudaBackEnd::PI = 0;
int cudaBackEnd::samples = 0;						// # of samples in depth direction
int cudaBackEnd::N_elements = 0;						// # of transducer elements
float cudaBackEnd::sampling_frequency = 0;			// sampling frequency
float cudaBackEnd::c = 0;							// speed of sound [m/s]	
int cudaBackEnd::N_active = 0;							// Active transmit elmeents
float cudaBackEnd::pitch = 0;				// spacing between the elements
float cudaBackEnd::aper_len = 0;	//aperture foot print 
float cudaBackEnd::zd = 0;			// virtual src distance from transducer array 
float cudaBackEnd::sample_spacing = 0;
float cudaBackEnd::del_convert = 0;			// used in delay calculation
float cudaBackEnd::rc = 0;					// radius_of_curvature
float cudaBackEnd::scan_angle = 0;
int cudaBackEnd::channels = 0;						// number of A-lines data used for beamforming
int cudaBackEnd::Nx = 0;								// 256 Lateral spacing
int cudaBackEnd::Nz = 0;								//1024 Axial spacing
int cudaBackEnd::pixels = 0;
int cudaBackEnd::pix_cha = 0;			// Nz*Nx*128 This array size is used for Apodization
int cudaBackEnd::frames = 0;
int cudaBackEnd::num_frames = 0;						// number of low resolution images
int cudaBackEnd::skip_frames = 0;						// 
int cudaBackEnd::dBvalue = 0;

float* cudaBackEnd::filt_coeff = new float[364];
//float* cudaBackEnd::env = new float[cudaBackEnd::pixels];
float* cudaBackEnd::env = 0;
float* cudaBackEnd::rximg2 = 0;
float* cudaBackEnd::d_filt_coeff = 0;
float* cudaBackEnd::d_z_axis = 0;
float* cudaBackEnd::d_x_axis = 0;
float* cudaBackEnd::d_probe = 0;
float* cudaBackEnd::d_rx_aperture = 0;
float* cudaBackEnd::d_rx_ap_distance = 0;
float* cudaBackEnd::d_cen_pos = 0;
float* cudaBackEnd::d_data = 0;
float* cudaBackEnd::d_bfHR = 0;
float* cudaBackEnd::d_tx_delay = 0;
float* cudaBackEnd::d_rx_delay = 0;
float* cudaBackEnd::d_rx_apod = 0;
float* cudaBackEnd::d_bfHRBP = 0;
float* cudaBackEnd::dev_beamformed_data1 = 0;
//** for curveLiner Prob  **//
float* cudaBackEnd::d_theta = 0;
float* cudaBackEnd::d_theta1 = 0;
float* cudaBackEnd::d_theta_tx = 0;
float cudaBackEnd::rximg[128 * 2040] = { 0 };
FILE* cudaBackEnd::fp = 0;
// for envelop detcetion
//------------------------

float cudaBackEnd::log_c = 20.0;
float* cudaBackEnd::d_envelop;
float* cudaBackEnd::d_logComp;
// init the cufft handles here
int cudaBackEnd::NBK1 = 0;
int cudaBackEnd::NBK2 = 0;
int cudaBackEnd::NBK3 = 0;
int cudaBackEnd::BKZ1 = 0;
int cudaBackEnd::BKZ2 = 0;
int cudaBackEnd::BKZ3 = 0; // declared in function calculateThreads

//dim3 cudaBackEnd::BKZ = 0;
hipfftHandle cudaBackEnd::plan;
hipStream_t cudaBackEnd::stream;
float* cudaBackEnd::d_xflat = 0;
float* cudaBackEnd::d_ifftI = 0;
float* cudaBackEnd::d_ifftR = 0;
hipfftComplex* cudaBackEnd::d_xflatComplex=0;
hipfftComplex* cudaBackEnd::d_fftComplex=0;
hipfftComplex* cudaBackEnd::d_ifftComplex=0;
hipfftComplex* cudaBackEnd::xflatComplex=0;
hipfftComplex* cudaBackEnd::fftComplex=0;
hipfftComplex* cudaBackEnd::ifftComplex=0;


std::ofstream cudaBackEnd::cudaLog("sample_output/cudaLog_file.txt", std::ofstream::out);
//std::ofstream cudaBackEnd::cudaLog.open("sample_output/cudaLog_file.txt");
const char* log_file_path = "cudaLog_file.txt";

CCyUSBDevice* cudaBackEnd::USBDevice = new CCyUSBDevice(NULL);
CCyControlEndPoint* cudaBackEnd::ept = cudaBackEnd::USBDevice->ControlEndPt;
CCyBulkEndPoint* cudaBackEnd::ept_in = cudaBackEnd::USBDevice->BulkInEndPt;


//void cudaDisplay::init(int rows, int cols) {
//
//	// init the cufft handles here
//	NBK = cols;
//	BKZ = dim3(rows);
//	hipfftPlan2d(&plan, cols, rows, HIPFFT_C2C);
//
//	hipMalloc((void**)&d_xflat, sizeof(float) * rows * cols);
//	hipMalloc((void**)&d_ifftI, sizeof(float) * rows * cols);
//	hipMalloc((void**)&d_ifftR, sizeof(float) * rows * cols);
//	//hipMalloc((void**)&d_envelop, sizeof(float) * rows * cols);
//	//hipMalloc((void**)&d_logComp, sizeof(float) * rows * cols);
//
//	xflatComplex = new hipfftComplex[rows * cols];
//	fftComplex = new hipfftComplex[rows * cols];
//	ifftComplex = new hipfftComplex[rows * cols];
//	hipMalloc((void**)&d_fftComplex, sizeof(hipfftComplex) * rows * cols);
//	hipMalloc((void**)&d_ifftComplex, sizeof(hipfftComplex) * rows * cols);
//	hipMalloc((void**)&d_xflatComplex, sizeof(hipfftComplex) * rows * cols);
//}
//
//void cudaDisplay::fetchEnvolep(float* d_inImg, float* d_envelop, int rows, int cols) {
//	// calculate the hilber transform here
//
//	real2complex << <NBK, BKZ >> > (d_inImg, d_xflatComplex);
//	hipfftExecC2C(plan, d_xflatComplex, d_fftComplex, HIPFFT_FORWARD);
//	hipfftExecC2C(plan, d_fftComplex, d_ifftComplex, HIPFFT_BACKWARD);
//	// convert t real and imaginary parts
//	splitComplex << <NBK, BKZ >> > (d_ifftComplex, d_ifftR, d_ifftI);
//	scalarMult << <NBK, BKZ >> > (d_ifftI, d_ifftI, (float)(1.0 / rows));
//	magnitide << <NBK, BKZ >> > (d_inImg, d_ifftI, d_envelop);
//}
//
//void cudaDisplay::logTransform(float* d_inImg, float* d_logComp, float c, float rows, float cols) {
//	// Performing the log transformation to the image to make it enhanced
//	// d_envelop is from previous function
//	logCompresion << <NBK, BKZ >> > (d_inImg, d_logComp, c);
//	//hipMemcpy(outImg, d_envelop, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
//}


void cudaBackEnd::wait(unsigned timeout)
{
	timeout += std::clock();
	while (std::clock() < timeout) continue;
}

void cudaBackEnd::write_rows(CCyControlEndPoint* ept, unsigned char* ptr, unsigned int numRows)
{
	int len = numRows * 16; // each row is 16 bytes to send
	int sent = 0;
	unsigned char* tPtr = ptr;
	while (sent < len) {
		LONG buflen = len - sent;
		if (buflen > 192) buflen = 192;
		//for (int i = 0; i < buflen; i += 16) {
		//	for (int j = 0; j < 16; j++) {
		//		printf("%02X\t ", ptr[i + j]);
		//	}
		//	printf("\n");
		//}
		ept->XferData(tPtr, buflen);
		tPtr += buflen;
		sent += buflen;
	}
	//printf("Sent %d bytes to EPT\n", sent);
}

bool cudaBackEnd::read_chunk(CCyBulkEndPoint* ept_in, unsigned char* recvBuf, LONG& length)
{
	bool result;
	LONG intlen = length;
	result = ept_in->XferData(recvBuf, intlen, NULL, true);
	//if (result) {
	//	printf("Received data ------------- : %d\n", intlen);
	//}
	//else {
	//	printf("***   ERROR receiving data - expected %d, got %d\n", length, intlen);
	//}
	ept_in->Abort();
	ept_in->Reset();
	return result;
}

int cudaBackEnd::insert_row(unsigned char* buf, int row, short addr, int data)
{
	int o = row * 16;
	buf[o] = 0xff; buf[o + 1] = 0xaa; buf[o + 2] = 0x01; buf[o + 3] = 0x07;
	buf[o + 4] = 0x00; buf[o + 5] = 0x00; buf[o + 6] = 0x00; buf[o + 7] = 0x01;
	buf[o + 8] = (addr & 0xff); buf[o + 9] = (addr & 0xff00) >> 8;
	buf[o + 10] = (data & 0xff); buf[o + 11] = (data & 0xff00) >> 8;
	buf[o + 12] = (data & 0xff0000) >> 16; buf[o + 13] = (data & 0xff000000) >> 24;
	buf[o + 14] = 0x00; buf[o + 15] = 0x00;
	return row + 1;
}

void cudaBackEnd::read_csv_mat(float* data, char* filename, int col1)
{
	char buffer[6240];  //6240
	char* token;

	int i = 0, j = 0;
	FILE* file;
	file = fopen(filename, "r");
	if (file == NULL)
	{
		// printf("Can't open the file");
	}
	else
	{
		while (fgets(buffer, sizeof(buffer), file) != 0)            // end-of-file indicator
		{
			token = strtok(buffer, ",");
			j = 0;
			while (token != NULL)
			{
				data[i * col1 + j] = atof(token);     //converts the string argument str to float
				token = strtok(NULL, ",");
				j++;
			}

			i++;
		}
		fclose(file);
		// printf("Complete reading from file %s\n", filename);
	}
}

void cudaBackEnd::read_csv_mat(long double* data, char* filename, int col1)
{
	char buffer[6240];  //6240
	char* token;

	int i = 0, j = 0;
	FILE* file;
	file = fopen(filename, "r");
	if (file == NULL)
	{
		// printf("Can't open the file");
	}
	else
	{
		while (fgets(buffer, sizeof(buffer), file) != 0)            // end-of-file indicator
		{
			token = strtok(buffer, ",");
			j = 0;
			while (token != NULL)
			{
				data[i * col1 + j] = atof(token);     //converts the string argument str to float
				token = strtok(NULL, ",");
				j++;
			}

			i++;
		}
		fclose(file);
		// printf("Complete reading from file %s\n", filename);
	}
}

void cudaBackEnd::read_csv_array(float* data, char* filename)
{
	char buffer[6240];  //6240
	char* token;
	int i = 0;
	FILE* file;

	file = fopen(filename, "r");
	if (file == NULL)
	{
		throw std::exception("File did not open");
	}

	while (fgets(buffer, sizeof(buffer), file) != 0)            // end-of-file indicator
	{
		token = strtok(buffer, ",");
		//j = 0;
		while (token != NULL)
		{
			data[i] = atof(token);     //converts the string argument str to float
			token = strtok(NULL, ",");
			//j++;
		}

		i++;
	}
	fclose(file);
	// printf("Complete reading from file %s\n", filename);

}

void cudaBackEnd::zeroC(float* bfHR, int pixels)
{
	for (int j = 0; j < pixels; j++)
	{
		bfHR[j] = 0;
	}
}

void cudaBackEnd::onesC(float* bfHR, int pixels)
{
	for (int j = 0; j < pixels; j++)
	{
		bfHR[j] = 1.10;
	}
}

//** <Curvilinear Prob> **//

void cudaBackEnd::csv_write_mat(long double* a, const char* filename, int row1, int col1)		//writes data to memory
{
	FILE* fp;
	int i;

	fp = fopen(filename, "w+");

	for (i = 0; i < row1; ++i)
	{
		for (int j = 0; j < col1; j++)
		{
			if (j == col1 - 1)					//for the last value in the column "," is not appended
			{									//matrix dimension error can occur with the presence of extra comma at last of the column
				fprintf(fp, "%g", a[i * col1 + j]);
			}
			else
				fprintf(fp, "%g,", a[i * col1 + j]);
		}



		fprintf(fp, "\n");
	}


	fclose(fp);
	// printf("\n %s file is created\n", filename);
}

void cudaBackEnd::csv_write_mat(double* a, const char* filename, int row1, int col1)		//writes data to memory
{
	FILE* fp;
	int i;

	fp = fopen(filename, "w+");

	for (i = 0; i < row1; ++i)
	{
		for (int j = 0; j < col1; j++)
		{
			if (j == col1 - 1)					//for the last value in the column "," is not appended
			{									//matrix dimension error can occur with the presence of extra comma at last of the column
				fprintf(fp, "%g", a[i * col1 + j]);
			}
			else
				fprintf(fp, "%g,", a[i * col1 + j]);
		}



		fprintf(fp, "\n");
	}


	fclose(fp);
	printf("\n %s file is created\n", filename);
}

void cudaBackEnd::csv_write_mat(float* a, const char* filename, int row1, int col1)	//for writing integer data "FUNCTION OVERLOADING"
{
	FILE* fp;
	int i;

	fp = fopen(filename, "w+");

	for (i = 0; i < row1; ++i)
	{
		for (int j = 0; j < col1; j++)
		{
			if (j == col1 - 1)
			{
				fprintf(fp, "%f", a[i * col1 + j]);
			}
			else

				fprintf(fp, "%f,", a[i * col1 + j]);
		}



		fprintf(fp, "\n");
	}


	fclose(fp);
	// printf("\n %s file is created\n", filename);
}

float** cudaBackEnd::convertsingto2darray(float* imgArray, int rows, int cols) {

	float** array2D = (float**)malloc(rows * sizeof(float*));
	for (int i = 0; i < rows; i++) {
		array2D[i] = (float*)malloc(cols * sizeof(float));
	}

	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			array2D[i][j] = (float)imgArray[i * cols + j];
		}
	}

	return array2D;
}

//int cudaBackEnd::num_threads = 1024;
//float cudaBackEnd::rx_f_number = 2.0;
//float cudaBackEnd::PI = 3.14;
//int cudaBackEnd::samples = 2040;						// # of samples in depth direction
//int cudaBackEnd::N_elements = 128;						// # of transducer elements
//float cudaBackEnd::sampling_frequency = 32e6;			// sampling frequency
//float cudaBackEnd::c = 1540.0;							// speed of sound [m/s]	
//int cudaBackEnd::N_active = 8;							// Active transmit elmeents
//float cudaBackEnd::pitch = 0.465 / 1000;				// spacing between the elements
//float cudaBackEnd::aper_len = (N_elements - 1) * pitch * 1000;	//aperture foot print 
//float cudaBackEnd::zd = pitch * N_active / (float)2;			// virtual src distance from transducer array 
//float cudaBackEnd::sample_spacing = c / sampling_frequency / (float)2;
//float cudaBackEnd::del_convert = sampling_frequency / c;			// used in delay calculation
//float cudaBackEnd::rc = 60.1 / 1000;					// radius_of_curvature
//float cudaBackEnd::scan_angle = (58 * PI) / 180;
//int cudaBackEnd::channels = 128;						// number of A-lines data used for beamforming
//int cudaBackEnd::Nx = 256;								// 256 Lateral spacing
//int cudaBackEnd::Nz = 1024;								//1024 Axial spacing
//int cudaBackEnd::pixels = Nz * Nx;
//int cudaBackEnd::pix_cha = pixels * channels;			// Nz*Nx*128 This array size is used for Apodization
//int cudaBackEnd::frames = 121;
//int cudaBackEnd::num_frames = 121;						// number of low resolution images
//int cudaBackEnd::skip_frames = 1;						// 
//int cudaBackEnd::dBvalue = 60;

//// constructor
//cudaBackEnd::cudaBackEnd() {
//
//	cudaBackEnd::cudaLog->open(log_file_path);
//
//}
//
//// destructor
//cudaBackEnd::~cudaBackEnd() {
//
//	cudaBackEnd::cudaLog->close();
//
//}

int cudaBackEnd::initHW(bool debug)
{
	//cudaBackEnd::USBDevice = new CCyUSBDevice(NULL);
	//cudaBackEnd::ept = cudaBackEnd::USBDevice->ControlEndPt;
	//cudaBackEnd::ept_in = cudaBackEnd::USBDevice->BulkInEndPt;
	//cudaBackEnd::cudaLog.open("sample_output/cudaLog_file.txt");

	if (debug)
		cudaBackEnd::cudaLog << "initHW start " << std::endl;

	if (!cudaBackEnd::ept) {
		//printf("Could not get Control endpoint.\n");
		if (debug)
			cudaBackEnd::cudaLog << "Error : Could not get Control endpoint " << std::endl;
		return 3;
	}

	if (!cudaBackEnd::ept_in) {
		if (debug)
			cudaBackEnd::cudaLog << "Error : No IN endpoint " << std::endl;
		//printf("No IN endpoint??\n");
		return 4;
	}

	// Send a vendor request (bRequest = 0x05) to the device
	cudaBackEnd::ept->Target = TGT_DEVICE;
	cudaBackEnd::ept->ReqType = REQ_VENDOR;
	cudaBackEnd::ept->Direction = DIR_TO_DEVICE;
	cudaBackEnd::ept->ReqCode = 0x05;
	cudaBackEnd::ept->Value = 1;
	cudaBackEnd::ept->Index = 0;
	cudaBackEnd::ept->TimeOut = 100;				// set timeout to 100ms for quick response
	cudaBackEnd::ept_in->MaxPktSize = 16384;
	cudaBackEnd::ept_in->TimeOut = 100;			// set timeout to 100ms for readin

	//std::ofstream mFile;
	//mFile.open("sample_output/initHW.txt");
	//mFile << "H/w init done" << std::endl;
	//mFile.close();

	if (debug)
		cudaBackEnd::cudaLog << "initHW sucessfull1 " << std::endl;

	return 0;
}

int cudaBackEnd::initcudaFFT() {

	//cudaBackEnd::cudaDisplayHandle->init(rows, cols);

	return 0;

}

int cudaBackEnd::initSettingFile(const char* path, bool debug)
{
	static int call_cout = 0;
	errno_t err;
	//FILE* fp;
	// path = "out25_curvi.txt"; for curvilieanr prob
	if ((err = fopen_s(&cudaBackEnd::fp, path, "r")) != 0) {
		if (debug)
			cudaBackEnd::cudaLog << "Could not open config file for reading " << std::endl;
		//printf("Could not open config file for reading.\n");
		return 5;
	}

	//std::ofstream mFile;
	//mFile.open("sample_output/setting.txt");
	//mFile << "setting file rEADING  done" << std::endl;
	//mFile.close();

	if (debug && call_cout == 0)
		cudaBackEnd::cudaLog << "Setting file Reading  done " << std::endl;

	call_cout++;
	return 0;
}

int cudaBackEnd::initGPUprobeC(double* probPrms, bool debug) {

	const int MASK_WIDTH = 364;

	if (debug)
		cudaBackEnd::cudaLog << "CUDA memmory init starting " << std::endl;

	try {
		cudaBackEnd::PI = (float)probPrms[3];
		//cudaBackEnd::MASK_WIDTH		= (int)probPrms[4];// mFile2 << MASK_WIDTH << std::endl;
		//cudaBackEnd::TILE_SIZE		= (int)probPrms[5]; //mFile2 << TILE_SIZE << std::endl;
		cudaBackEnd::num_threads = (int)probPrms[6];
		cudaBackEnd::rx_f_number = (float)probPrms[7];
		cudaBackEnd::samples = (int)probPrms[8];
		cudaBackEnd::N_elements = (int)probPrms[9];
		cudaBackEnd::sampling_frequency = (float)probPrms[10];
		cudaBackEnd::c = (float)probPrms[11];
		cudaBackEnd::N_active = (int)probPrms[12];
		cudaBackEnd::channels = (int)probPrms[13];
		cudaBackEnd::Nx = (int)probPrms[14];
		cudaBackEnd::Nz = (int)probPrms[15];
		cudaBackEnd::frames = (int)probPrms[16];
		cudaBackEnd::num_frames = (int)probPrms[17];
		cudaBackEnd::skip_frames = (int)probPrms[18];
		cudaBackEnd::dBvalue = (int)probPrms[19];
		cudaBackEnd::pitch = (float)probPrms[20];
		cudaBackEnd::aper_len = (float)probPrms[21];
		cudaBackEnd::zd = (float)probPrms[22];
		cudaBackEnd::sample_spacing = (float)probPrms[23];
		cudaBackEnd::del_convert = (float)probPrms[24];
		cudaBackEnd::rc = (float)probPrms[25];
		cudaBackEnd::scan_angle = (float)probPrms[26];
		cudaBackEnd::pixels = (int)probPrms[27];
		cudaBackEnd::pix_cha = (int)probPrms[28];
	}
	catch (std::exception& e) {
		return 6;
	}


	if (debug)
	{
		cudaBackEnd::cudaLog << "PI : " << PI << std::endl;
		cudaBackEnd::cudaLog << "rx_f_number : " << rx_f_number << std::endl;
		cudaBackEnd::cudaLog << "samples : " << samples << std::endl;
		cudaBackEnd::cudaLog << "N_elements : " << N_elements << std::endl;
		cudaBackEnd::cudaLog << "sampling_frequency : " << sampling_frequency << std::endl;
		cudaBackEnd::cudaLog << "c : " << c << std::endl;
		cudaBackEnd::cudaLog << "N_active : " << N_active << std::endl;
		cudaBackEnd::cudaLog << "channels : " << channels << std::endl;
		cudaBackEnd::cudaLog << "Nx : " << Nx << std::endl;
		cudaBackEnd::cudaLog << "Nz : " << Nz << std::endl;
		cudaBackEnd::cudaLog << "frames : " << frames << std::endl;
		cudaBackEnd::cudaLog << "num_frames : " << num_frames << std::endl;
		cudaBackEnd::cudaLog << "skip_frames : " << skip_frames << std::endl;
		cudaBackEnd::cudaLog << "dBvalue : " << dBvalue << std::endl;
		cudaBackEnd::cudaLog << "pitch : " << pitch << std::endl;
		cudaBackEnd::cudaLog << "aper_len : " << aper_len << std::endl;
		cudaBackEnd::cudaLog << "zd : " << zd << std::endl;
		cudaBackEnd::cudaLog << "sample_spacing : " << sample_spacing << std::endl;
		cudaBackEnd::cudaLog << "del_convert : " << del_convert << std::endl;
		cudaBackEnd::cudaLog << "rc : " << rc << std::endl;
		cudaBackEnd::cudaLog << "scan_angle : " << scan_angle << std::endl;
		cudaBackEnd::cudaLog << "pixels : " << pixels << std::endl;
		cudaBackEnd::cudaLog << "pix_cha : " << pix_cha << std::endl;
		cudaBackEnd::cudaLog << "param reading done " << std::endl;
	}


	try
	{
		cudaBackEnd::env = new float[cudaBackEnd::pixels];
		zeroC(cudaBackEnd::rximg, cudaBackEnd::samples * cudaBackEnd::N_elements);   // set rx_img array values to zero.

		char filename1[200];
		sprintf(filename1, "b_10M.csv");
		cudaBackEnd::read_csv_array(cudaBackEnd::filt_coeff, filename1);    // csv file read

		//float* d_filt_coeff = 0;
		hipMalloc((void**)&cudaBackEnd::d_filt_coeff, sizeof(float) * MASK_WIDTH);
		hipMemcpy(cudaBackEnd::d_filt_coeff, filt_coeff, sizeof(float) * MASK_WIDTH, hipMemcpyHostToDevice);

		////////  Intialization &(or) Memory allocation  //////////////////
		//float* d_data = 0;   // variable to store raw rf data
		hipMalloc((void**)&cudaBackEnd::d_data, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels);

		//float* d_bfHR = 0;  // variable to store beamformed high-resolution beamformed image 
		hipMalloc((void**)&cudaBackEnd::d_bfHR, cudaBackEnd::pixels * sizeof(float));
		//zeros << <pixels / num_threads + 1, num_threads >> > (d_bfHR, pixels);  
		hipMemset(cudaBackEnd::d_bfHR, 0, cudaBackEnd::pixels * sizeof(float));

		//float* dev_beamformed_data1 = 0;   // variable to store reshaped beamformed data
		hipMalloc((void**)&cudaBackEnd::dev_beamformed_data1, cudaBackEnd::pixels * sizeof(float));

		//float* d_bfHRBP = 0;  // variable to store beamformed high-resolution bandpass filtered data
		hipMalloc((void**)&cudaBackEnd::d_bfHRBP, sizeof(float) * cudaBackEnd::pixels);

		/////////////////// theta positions for all elements ////////////////////
		//float* d_theta = 0;
		hipMalloc((void**)&cudaBackEnd::d_theta, cudaBackEnd::N_elements * sizeof(float));
		range << <cudaBackEnd::Nx / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_theta, (-cudaBackEnd::scan_angle / 2), cudaBackEnd::N_elements, (cudaBackEnd::scan_angle / (cudaBackEnd::N_elements - 1)));


		///////////// theta for grid /////////////////  theta = -scan_angle / 2 : scan_angle / (elements - 1) : scan_angle / 2;
		//float* d_theta1 = 0;
		hipMalloc((void**)&cudaBackEnd::d_theta1, cudaBackEnd::Nx * sizeof(float));
		range << <cudaBackEnd::Nx / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_theta1, (-cudaBackEnd::scan_angle / 2), cudaBackEnd::Nx, (cudaBackEnd::scan_angle / (cudaBackEnd::Nx - 1)));
		hipGetLastError();
		hipDeviceSynchronize();

		////////////// z value////////////////////
		float dz = cudaBackEnd::sample_spacing * cudaBackEnd::samples / cudaBackEnd::Nz;  // depth / (Nz - 1) / 1000;   // spacing in axial (z) direction in mm;
		//float* d_z_axis = 0;
		hipMalloc((void**)&cudaBackEnd::d_z_axis, cudaBackEnd::Nz * sizeof(float));
		range << <cudaBackEnd::Nz / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_z_axis, 0, cudaBackEnd::Nz, dz);
		hipGetLastError();
		hipDeviceSynchronize();

		//////////////////////////////// x value////////////////////////////////
		float dx = aper_len / (Nx - 1);
		//float* d_x_axis = 0;
		hipMalloc((void**)&cudaBackEnd::d_x_axis, cudaBackEnd::Nx * sizeof(float));
		range << <cudaBackEnd::Nx / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_x_axis, (-cudaBackEnd::aper_len / 2000), cudaBackEnd::Nx, dx / 1000);
		hipGetLastError();
		hipDeviceSynchronize();

		//////////////// Probe geometry, this info can be taken from transducer file ////////////////////
		//float* d_probe = 0;
		hipMalloc((void**)&cudaBackEnd::d_probe, cudaBackEnd::N_elements * sizeof(float));
		//hipMemcpy(d_probe, probe_ge_x, N_elements * sizeof(double), hipMemcpyHostToDevice);
		range << <1, cudaBackEnd::N_elements >> > (cudaBackEnd::d_probe, (-cudaBackEnd::aper_len / 2000), cudaBackEnd::N_elements, cudaBackEnd::pitch);
		hipGetLastError();
		hipDeviceSynchronize();

		/////////////////rx aerture calculation using Fnumber///////////////////////////////
		// rx_aper=rfsca.z/rf_number
		//float* d_rx_aperture = 0;
		hipMalloc((void**)&cudaBackEnd::d_rx_aperture, cudaBackEnd::Nz * sizeof(float));
		element_division << <cudaBackEnd::Nz / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_z_axis, cudaBackEnd::rx_f_number, cudaBackEnd::Nz, cudaBackEnd::d_rx_aperture);
		hipGetLastError();
		hipDeviceSynchronize();

		////////////////////////rx aerture distance////////
		//float* d_rx_ap_distance = 0;
		hipMalloc((void**)&cudaBackEnd::d_rx_ap_distance, cudaBackEnd::channels * cudaBackEnd::Nx * sizeof(float));  //20.087 us
		aperture_distance << <cudaBackEnd::Nx * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_x_axis, cudaBackEnd::d_probe, cudaBackEnd::Nx, cudaBackEnd::channels, cudaBackEnd::d_rx_ap_distance);
		hipGetLastError();
		hipDeviceSynchronize();

		///////////////////apodization/////////////////
		//float* d_rx_apod = 0;
		hipMalloc((void**)&cudaBackEnd::d_rx_apod, sizeof(float) * cudaBackEnd::Nz * cudaBackEnd::channels * cudaBackEnd::Nx);
		apodization << <cudaBackEnd::pixels * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_rx_ap_distance, cudaBackEnd::d_rx_aperture, cudaBackEnd::Nz, cudaBackEnd::Nx, cudaBackEnd::channels, cudaBackEnd::pixels, cudaBackEnd::d_rx_apod);
		hipGetLastError();
		hipDeviceSynchronize();

		//// check for nan values,
		isnan_test_array << <cudaBackEnd::pixels * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_rx_apod, cudaBackEnd::pixels * cudaBackEnd::channels);
		hipGetLastError();
		hipDeviceSynchronize();

		hipFree(cudaBackEnd::d_rx_aperture);
		hipFree(cudaBackEnd::d_rx_ap_distance);

		/////////////receive delay calculation /////////////////////////////////////////////
		//float* d_rx_delay = 0;
		hipMalloc((void**)&cudaBackEnd::d_rx_delay, cudaBackEnd::pix_cha * sizeof(float));
		receive_delay << < cudaBackEnd::pixels * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_theta, cudaBackEnd::d_theta1, cudaBackEnd::rc, cudaBackEnd::d_z_axis, cudaBackEnd::channels, cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::del_convert, cudaBackEnd::d_rx_delay);
		hipGetLastError();
		hipDeviceSynchronize();

		/////////////////// theta positions for all elements ////////////////////
		//float* d_theta_tx = 0;
		hipMalloc((void**)&cudaBackEnd::d_theta_tx, cudaBackEnd::num_frames * sizeof(float));
		theta1 << < 1, cudaBackEnd::num_frames >> > (cudaBackEnd::d_theta_tx, cudaBackEnd::d_theta, cudaBackEnd::frames, cudaBackEnd::N_active, cudaBackEnd::skip_frames);

		/////////////////// Transmit delay calculation ////////////////////
		//float* d_tx_delay = 0;
		hipMalloc((void**)&cudaBackEnd::d_tx_delay, cudaBackEnd::pixels * cudaBackEnd::num_frames * sizeof(float));
		//transmitter delay for 16 frames,  
		transmit_delay << < cudaBackEnd::pixels * cudaBackEnd::num_frames / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_theta1, cudaBackEnd::d_z_axis, cudaBackEnd::rc, cudaBackEnd::d_theta_tx, cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::del_convert, cudaBackEnd::num_frames, cudaBackEnd::zd, cudaBackEnd::d_tx_delay);
		hipGetLastError();
		hipDeviceSynchronize();

		if (debug)
			cudaBackEnd::cudaLog << "CUDA memmory allocation completed succefully " << std::endl;
	}
	catch (std::exception& e) {
		return 7;
	}

	hipFree(cudaBackEnd::d_theta1);
	hipFree(cudaBackEnd::d_probe);
	hipFree(cudaBackEnd::d_x_axis);
	hipFree(cudaBackEnd::d_z_axis);
	hipFree(cudaBackEnd::d_theta_tx);

	return 0;
}

float** cudaBackEnd::computeBModeImgDev(bool debug) {

	const int MAX_LINE = 256;
	const int N_RX = 64;
	unsigned char buf[16 * 1024];
	static int call_count = 0; // parms counts the number of time this function calls

	if (debug && call_count == 0)
		cudaBackEnd::cudaLog << "b_mode image generation starts " << std::endl;

	//-----------------------
	int ok = cudaBackEnd::initSettingFile("out25.txt");
	//-----------------------

	char line[MAX_LINE]; // Max possible line length?
	int iteration = 0;
	int errcount = 0;
	int row = 0;					// Keep track of how many rows have been added
	unsigned int addr, data;
	unsigned char recvbuf[2048 * N_RX * 2];
	const int MAXROWS = 2040;
	LONG rxlen = MAXROWS * N_RX * 2;
	//cudaBackEnd::env = new float[cudaBackEnd::pixels];

	if (debug && call_count == 0)
		cudaBackEnd::cudaLog << "setting file reading done " << std::endl;

	try
	{
		//unsigned int start = clock();
		while (fgets(line, cudaBackEnd::MAX_LINE, cudaBackEnd::fp))
		{
			line[strcspn(line, "\n")] = 0; // Trim trailing newline
			if ((strlen(line) == 0) || (line[0] == ' ') || (line[0] == '#')) {
				//printf("Skipping [%s]\n", line);
			}
			else if (line[0] == 'O') {
				sscanf_s(line, "O %04X %08X ", &addr, &data);
				//printf("Write %08X to Obelix %04X\n", data, addr);
				row = insert_row(buf, row, addr, data);
			}
			else if (line[0] == 'T') {
				sscanf_s(line, "T %04X %08X ", &addr, &data);
				row = insert_row(buf, row, 0x6, 0x40000000 | addr);
				row = insert_row(buf, row, 0x7, data);
				row = insert_row(buf, row, 0x6, 0xC0000000 | addr);
				//printf("Write %08X to TX %04X\n", data, addr);
			}
			else if (line[0] == 'A') {
				sscanf_s(line, "A %04X %08X ", &addr, &data);
				row = insert_row(buf, row, 0x6, 0x00000000 | addr);
				row = insert_row(buf, row, 0x7, data);
				row = insert_row(buf, row, 0x6, 0x80000000 | addr);
				//printf("Write %08X to AFE %04X\n", data, addr);
			}
			else if (line[0] == 'C') {  // CAPTURE STARTS
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x01);
				//write_rows(ept, buf, row);  // Send commands
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x10);
				//write_rows(ept, buf, row);  // Send commands
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x00);

				cudaBackEnd::ept_in->Abort();
				cudaBackEnd::ept_in->Reset();

				write_rows(cudaBackEnd::ept, buf, row);  // Send commands

				wait(1);
				if (read_chunk(cudaBackEnd::ept_in, recvbuf, rxlen)) {
					short* rxdata = (short*)(recvbuf);
					for (int i = 0; i < rxlen / 2; i++) {
						if (rxdata[i] >= 512) rxdata[i] -= 1024;
					}
					// Trying to read only first N-1 rows and discard 1st sample
					for (int i = 0; i < N_RX; i++) {
						for (int j = 0; j < MAXROWS - 1; j++) {
							//rximg[iteration][i][j] = rxdata[j*64+i+2];
							if (iteration < 29) {      // start from 0 index, so 30-1 
								cudaBackEnd::rximg[i * MAXROWS + j] = rxdata[j * N_RX + i + 2];
							}
							else if (iteration > 91) {
								cudaBackEnd::rximg[(i + 64) * MAXROWS + j] = rxdata[j * N_RX + i + 2];
							}
							else {
								cudaBackEnd::rximg[(i + iteration - 28) * MAXROWS + j] = rxdata[j * N_RX + i + 2];
							}
							//rximg[i * MAXROWS + j] = rxdata[j * N_RX + i + 2];
						}
					}
					//saveToFile(iteration, rxlen, recvbuf);
				}
				else {
					errcount++;
				}

				//clock_t begin = clock();   // clock intiated

				hipMemcpy(cudaBackEnd::d_data, cudaBackEnd::rximg, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels, hipMemcpyHostToDevice);

				beamformingLR3 << <(cudaBackEnd::pixels / 256) * cudaBackEnd::channels, 256 >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_tx_delay, cudaBackEnd::d_rx_delay, cudaBackEnd::d_data, cudaBackEnd::d_rx_apod, cudaBackEnd::samples, cudaBackEnd::pixels, iteration, cudaBackEnd::num_frames, cudaBackEnd::channels);
				hipGetLastError();
				hipDeviceSynchronize();

				iteration++;	// Increment iteration after saving to image
				row = 0;		// Reset buffer for next iteration
			}
			else {
				if (debug) {
					cudaBackEnd::cudaLog << "Error : Don't know how to handle [%s] yet " << std::endl;
					cudaBackEnd::cudaLog << line << std::endl;

				}
				return nullptr;
			}


		}
		// while loop completed
		if (debug && call_count == 0)
			cudaBackEnd::cudaLog << "while loop completed " << std::endl;


	}
	catch (std::exception& e)
	{
		return nullptr;
	}

	try
	{
		//////////// Bandpass filtering using shared memory /////////////////////
		BPfilter1SharedMem << <(cudaBackEnd::pixels + cudaBackEnd::TILE_SIZE - 1) / cudaBackEnd::TILE_SIZE, cudaBackEnd::TILE_SIZE >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_filt_coeff, cudaBackEnd::pixels, cudaBackEnd::d_bfHRBP);
		hipGetLastError();
		hipDeviceSynchronize();

		if (debug && call_count == 0)
			cudaBackEnd::cudaLog << "BPF done" << std::endl;

		//////////////// reshape of the beamformed data ///////////////
		reshape_columnwise << <cudaBackEnd::pixels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::d_bfHRBP);
		hipGetLastError();
		hipDeviceSynchronize();
		hipMemcpy(cudaBackEnd::env, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::Nz * cudaBackEnd::Nx * sizeof(float), hipMemcpyDeviceToHost);

		if (debug && call_count == 0)
		{
			char fileout[200];
			sprintf(fileout, "sample_output/b_curve_mode.csv"); //all the 16 inputs are arranged in a single file
			csv_write_mat(cudaBackEnd::env, fileout, cudaBackEnd::Nz, cudaBackEnd::Nx);
			cudaBackEnd::cudaLog << "Copy to host completed" << std::endl;
		}



	}
	catch (std::exception& e)
	{
		return nullptr;

	}

	float** outArray = convertsingto2darray(cudaBackEnd::env, cudaBackEnd::Nz, cudaBackEnd::Nx);

	if (debug && call_count == 0)
	{
		cudaBackEnd::cudaLog << "First image capture completed" << std::endl;
		cudaBackEnd::cudaLog.close();
	}


	// For next iteration
	//hipMalloc((void**)&cudaBackEnd::d_data, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels);
	//hipMemset(cudaBackEnd::d_bfHR, 0, cudaBackEnd::pixels * sizeof(float));
	//hipMemset(cudaBackEnd::dev_beamformed_data1, 0, cudaBackEnd::pixels * sizeof(float));
	zeroC(cudaBackEnd::rximg, cudaBackEnd::samples * cudaBackEnd::N_elements);   // set rx_img array values to zero.
	call_count++;

	return outArray;

}

int cudaBackEnd::initGPUprobeL(double* prob_params, bool debug)
{
	//// perform b-mode generation here using cuda
	//const int TILE_SIZE = 4;
	//int MASK_WIDTH = 364;
	//const int MAX_LINE = 256;
	////// Computer (NIVIDIA) parametrs
	//int num_threads = 1024;
	///// Apodization parameters
	//float rx_f_number = 2.0;
	///////// Ultrasound scanner parametrs
	////float depth = 49.28;      // Depth of imaging in mm
	//int samples = 2040;         // # of samples in depth direction
	//int N_elements = 64;        // # of transducer elements
	//float sampling_frequency = 32e6;   // sampling frequency
	//float c = 1540.0;		 // speed of sound [m/s]	
	//int N_active = 8;        // Active transmit elmeents
	//float pitch = 0.3 / 1000;// spacing between the elements
	//float aper_len = (N_elements - 1) * pitch * 1000;  //aperture foot print 
	//float zd = pitch * N_active / (float)2;            // virtual src distance from transducer array 
	//float sample_spacing = c / sampling_frequency / (float)2;
	//float del_convert = sampling_frequency / c;  // used in delay calculation
	//int channels = 64;							 // number of A-lines data used for beamforming
	////// Beamforming "Grid" parameters
	//int Nx = 256;			// 256 Lateral spacing
	//int Nz = 1024;			//1024 Axial spacing
	//int pixels = Nz * Nx;
	//int pix_cha = pixels * channels;// Nz*Nx*128 This array size is used for Apodization
	//int num_frames = 57;			// number of low resolution images
	//int skip_frames = 1;			//

	if (debug)
		cudaBackEnd::cudaLog << "CUDA memm init for linear prob starts " << std::endl;

	// perform b-mode generation here using cuda
	const int TILE_SIZE = prob_params[0];
	int MASK_WIDTH = prob_params[1];
	const int MAX_LINE = prob_params[2];


	try {
		cudaBackEnd::num_threads = prob_params[3];
		cudaBackEnd::rx_f_number = prob_params[4];	// Apodization parameters
		cudaBackEnd::samples = prob_params[5];	// # of samples in depth direction
		cudaBackEnd::N_elements = prob_params[6];	// # of transducer elements
		cudaBackEnd::sampling_frequency = prob_params[7];   // sampling frequency
		cudaBackEnd::c = prob_params[8];	// speed of sound [m/s]	
		cudaBackEnd::N_active = prob_params[9];   // Active transmit elmeents
		cudaBackEnd::pitch = prob_params[10];	// spacing between the elements
		cudaBackEnd::aper_len = prob_params[11];  // aperture foot print 
		cudaBackEnd::zd = prob_params[12];  // virtual src distance from transducer array 
		cudaBackEnd::sample_spacing = prob_params[13];
		cudaBackEnd::del_convert = prob_params[14];  // used in delay calculation
		cudaBackEnd::channels = prob_params[15];	// number of A-lines data used for beamforming
		cudaBackEnd::Nx = prob_params[16];	// 256 Lateral spacing Beamforming "Grid" parameters
		cudaBackEnd::Nz = prob_params[17];	// 1024 Axial spacing
		cudaBackEnd::pixels = prob_params[18];
		cudaBackEnd::pix_cha = prob_params[19];	// Nz*Nx*128 This array size is used for Apodization
		cudaBackEnd::num_frames = prob_params[20];	// number of low resolution images
		cudaBackEnd::skip_frames = prob_params[21];	//
	}
	catch (std::exception& e) {
		return 8;
	}

	if (debug)
	{
		cudaBackEnd::cudaLog << "num_threads : " << prob_params[3] << std::endl;
		cudaBackEnd::cudaLog << "rx_f_number : " << prob_params[4] << std::endl;
		cudaBackEnd::cudaLog << "samples : " << prob_params[5] << std::endl;
		cudaBackEnd::cudaLog << "N_elements : " << prob_params[6] << std::endl;
		cudaBackEnd::cudaLog << "sampling_frequency : " << prob_params[7] << std::endl;
		cudaBackEnd::cudaLog << "c : " << prob_params[8] << std::endl;
		cudaBackEnd::cudaLog << "N_active : " << prob_params[9] << std::endl;
		cudaBackEnd::cudaLog << "pitch : " << prob_params[10] << std::endl;
		cudaBackEnd::cudaLog << "aper_len : " << prob_params[11] << std::endl;
		cudaBackEnd::cudaLog << "zd : " << prob_params[12] << std::endl;
		cudaBackEnd::cudaLog << "sample_spacing : " << prob_params[13] << std::endl;
		cudaBackEnd::cudaLog << "del_convert : " << prob_params[14] << std::endl;
		cudaBackEnd::cudaLog << "channels : " << prob_params[15] << std::endl;
		cudaBackEnd::cudaLog << "Nx : " << prob_params[16] << std::endl;
		cudaBackEnd::cudaLog << "Nz : " << prob_params[17] << std::endl;
		cudaBackEnd::cudaLog << "pixels : " << prob_params[18] << std::endl;
		cudaBackEnd::cudaLog << "pix_cha : " << prob_params[19] << std::endl;
		cudaBackEnd::cudaLog << "num_frames : " << prob_params[20] << std::endl;
		cudaBackEnd::cudaLog << "skip_frames : " << prob_params[21] << std::endl;
	}

	try
	{
		char filename3[200];
		sprintf(filename3, "b_10M.csv");
		read_csv_array(cudaBackEnd::filt_coeff, filename3);    // csv file read
		//cv::imwrite("okMat3.png", testMat0);

		// float* d_filt_coeff = 0;
		hipMalloc((void**)&cudaBackEnd::d_filt_coeff, sizeof(float) * MASK_WIDTH);
		hipMemcpy(cudaBackEnd::d_filt_coeff, cudaBackEnd::filt_coeff, sizeof(float) * MASK_WIDTH, hipMemcpyHostToDevice);

		////////  Intialization &(or) Memory allocation  //////////////////
		hipMalloc((void**)&cudaBackEnd::d_data, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels);// variable to store raw rf data

		hipMalloc((void**)&cudaBackEnd::d_bfHR, cudaBackEnd::pixels * sizeof(float)); // variable to store beamformed high-resolution beamformed image 
		hipMemset(cudaBackEnd::d_bfHR, 0, cudaBackEnd::pixels * sizeof(float));

		hipMalloc((void**)&cudaBackEnd::dev_beamformed_data1, cudaBackEnd::pixels * sizeof(float));// variable to store reshaped beamformed data

		hipMalloc((void**)&cudaBackEnd::d_bfHRBP, sizeof(float) * cudaBackEnd::pixels);// variable to store beamformed high-resolution bandpass filtered data

		////////////// z value////////////////////
		float dz = sample_spacing * samples / Nz;  // depth / (Nz - 1) / 1000;   // spacing in axial (z) direction in mm;
		hipMalloc((void**)&cudaBackEnd::d_z_axis, cudaBackEnd::Nz * sizeof(float));
		range << <cudaBackEnd::Nz / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_z_axis, 0, cudaBackEnd::Nz, dz);
		hipGetLastError();
		hipDeviceSynchronize();

		//////////////////////////////// x value////////////////////////////////
		float dx = aper_len / (cudaBackEnd::Nx - 1);
		// float* d_x_axis = 0;
		hipMalloc((void**)&cudaBackEnd::d_x_axis, cudaBackEnd::Nx * sizeof(float));    // 167.939 us
		range << <cudaBackEnd::Nx / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_x_axis, (-cudaBackEnd::aper_len / 2000), cudaBackEnd::Nx, dx / 1000);
		hipGetLastError();
		hipDeviceSynchronize();

		//////////////// Probe geometry, this info can be taken from transducer file ////////////////////
		//float* d_probe = 0;
		hipMalloc((void**)&cudaBackEnd::d_probe, cudaBackEnd::N_elements * sizeof(float));
		range << <1, cudaBackEnd::N_elements >> > (cudaBackEnd::d_probe, (-cudaBackEnd::aper_len / 2000), cudaBackEnd::N_elements, cudaBackEnd::pitch);
		hipGetLastError();
		hipDeviceSynchronize();

		/////////////////rx aerture calculation using Fnumber///////////////////////////////
		hipMalloc((void**)&cudaBackEnd::d_rx_aperture, cudaBackEnd::Nz * sizeof(float));
		element_division << <cudaBackEnd::Nz / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_z_axis, cudaBackEnd::rx_f_number, cudaBackEnd::Nz, cudaBackEnd::d_rx_aperture);
		hipGetLastError();
		hipDeviceSynchronize();

		////////////////////////rx aerture distance////////
		hipMalloc((void**)&cudaBackEnd::d_rx_ap_distance, cudaBackEnd::channels * cudaBackEnd::Nx * sizeof(float));
		aperture_distance << <cudaBackEnd::Nx * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_x_axis, cudaBackEnd::d_probe, cudaBackEnd::Nx, cudaBackEnd::channels, cudaBackEnd::d_rx_ap_distance);
		hipGetLastError();
		hipDeviceSynchronize();

		///////////////////apodization/////////////////
		// float* d_rx_apod = 0;
		hipMalloc((void**)&cudaBackEnd::d_rx_apod, sizeof(float) * cudaBackEnd::Nz * cudaBackEnd::channels * cudaBackEnd::Nx);
		apodization << <cudaBackEnd::pixels * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_rx_ap_distance, cudaBackEnd::d_rx_aperture, cudaBackEnd::Nz, cudaBackEnd::Nx, cudaBackEnd::channels, cudaBackEnd::pixels, cudaBackEnd::d_rx_apod);
		hipGetLastError();
		hipDeviceSynchronize();

		hipFree(d_rx_aperture);
		hipFree(d_rx_ap_distance);

		/////////////////// calculate central positions transmit subaperture ////////////////////
		hipMalloc((void**)&cudaBackEnd::d_cen_pos, cudaBackEnd::num_frames * sizeof(float));
		Tx_cen_pos << < 1, cudaBackEnd::num_frames >> > (cudaBackEnd::d_cen_pos, cudaBackEnd::N_elements, cudaBackEnd::N_active, cudaBackEnd::pitch, cudaBackEnd::skip_frames, cudaBackEnd::num_frames, cudaBackEnd::d_probe);

		/////////////receive delay calculation /////////////////////////////////////////////
		hipMalloc((void**)&cudaBackEnd::d_rx_delay, cudaBackEnd::pix_cha * sizeof(float));
		receive_delay << < cudaBackEnd::pixels * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_probe, cudaBackEnd::d_x_axis, cudaBackEnd::d_z_axis, cudaBackEnd::channels, cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::del_convert, cudaBackEnd::d_rx_delay);
		hipGetLastError();
		hipDeviceSynchronize();

		/////////////////// Transmit delay calculation ////////////////////
		hipMalloc((void**)&cudaBackEnd::d_tx_delay, cudaBackEnd::pixels * cudaBackEnd::num_frames * sizeof(float));
		transmit_delay << < cudaBackEnd::pixels * cudaBackEnd::num_frames / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_x_axis, cudaBackEnd::d_z_axis, cudaBackEnd::d_cen_pos, cudaBackEnd::zd, cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::del_convert, cudaBackEnd::num_frames, cudaBackEnd::d_tx_delay);
		hipGetLastError();
		hipDeviceSynchronize();

		//////// Host to device and back /////////
		//cudaBackEnd::env = new float[cudaBackEnd::pixels];
		//cudaBackEnd::rximg2 = new float[cudaBackEnd::N_elements * cudaBackEnd::samples];
		hipHostMalloc(&cudaBackEnd::env, sizeof(float)* cudaBackEnd::pixels);
		hipHostMalloc(&cudaBackEnd::rximg2, sizeof(float)* cudaBackEnd::pixels);
		
		/////////////////////////////////////////////////

		///////  for envelop detection /////////
		//cudaBackEnd::NBK = cudaBackEnd::Nx;
		//cudaBackEnd::BKZ = dim3(cudaBackEnd::Nz);
		hipfftPlan2d(&cudaBackEnd::plan, cudaBackEnd::Nx, cudaBackEnd::Nz, HIPFFT_C2C);
		hipMalloc((void**)&cudaBackEnd::d_xflat, sizeof(float)* cudaBackEnd::Nz* cudaBackEnd::Nx);
		hipMalloc((void**)&cudaBackEnd::d_ifftI, sizeof(float)* cudaBackEnd::Nz* cudaBackEnd::Nx);
		hipMalloc((void**)&cudaBackEnd::d_ifftR, sizeof(float)* cudaBackEnd::Nz* cudaBackEnd::Nx);
		hipMalloc((void**)&cudaBackEnd::d_logComp, sizeof(float)* cudaBackEnd::Nz* cudaBackEnd::Nx);
		hipMalloc((void**)&cudaBackEnd::d_envelop, sizeof(float)* cudaBackEnd::Nz* cudaBackEnd::Nx);


		cudaBackEnd::xflatComplex = new hipfftComplex[cudaBackEnd::Nz * cudaBackEnd::Nx];
		cudaBackEnd::fftComplex = new hipfftComplex[cudaBackEnd::Nz * cudaBackEnd::Nx];
		cudaBackEnd::ifftComplex = new hipfftComplex[cudaBackEnd::Nz * cudaBackEnd::Nx];
		hipMalloc((void**)&cudaBackEnd::d_fftComplex, sizeof(hipfftComplex)* cudaBackEnd::Nz* cudaBackEnd::Nx);
		hipMalloc((void**)&cudaBackEnd::d_ifftComplex, sizeof(hipfftComplex)* cudaBackEnd::Nz* cudaBackEnd::Nx);
		hipMalloc((void**)&cudaBackEnd::d_xflatComplex, sizeof(hipfftComplex)* cudaBackEnd::Nz* cudaBackEnd::Nx);

		hipStreamCreateWithFlags(&cudaBackEnd::stream, hipStreamNonBlocking);
	}
	catch (std::exception& c)
	{
		return 9;
	}


	////////////Free cuda memory (one time use) ///////////////////////////
	hipFree(cudaBackEnd::d_probe);
	hipFree(cudaBackEnd::d_x_axis);
	hipFree(cudaBackEnd::d_z_axis);
	hipFree(cudaBackEnd::d_cen_pos);

	if (debug)
		cudaBackEnd::cudaLog << "CUDA memm init for linear prob completed " << std::endl;

	return 0;

}

float** cudaBackEnd::computeBModeImgLinDiv(bool debug)
{
	static int call_count = 0;
	zeroC(cudaBackEnd::env, cudaBackEnd::Nx);   // set rx_img array values to zero.
	zeroC(cudaBackEnd::rximg2, cudaBackEnd::samples * cudaBackEnd::N_elements);   // set rx_img array values to zero.
	int ok = cudaBackEnd::initSettingFile("out25.txt");

	if (debug && call_count == 0)
		cudaBackEnd::cudaLog << "CUDA B-Mode for linear prob starts setting file read" << std::endl;

	int ok1 = cudaBackEnd::readAndBeamForm(debug);

	if (debug && call_count == 0 && ok1==0)
		cudaBackEnd::cudaLog << "CUDA B-Mode for linear prob while loop completed" << std::endl;
	//-------------------------------------------------------------------------------------


	int ok2 = cudaBackEnd::filterBeamForm(debug);
	if (debug && call_count == 0 && ok2 ==0)
		cudaBackEnd::cudaLog << " filterBeamForm completed " << std::endl;
	//-------------------------------------------------------------------------------------


	int ok3 = cudaBackEnd::envelopAndCompress(debug);
	if (debug && call_count == 0 && ok3 == 0)
		cudaBackEnd::cudaLog << " envelopAndCompress completed " << std::endl;
	//-------------------------------------------------------------------------------------


	float** outArray = convertsingto2darray(cudaBackEnd::env, cudaBackEnd::Nz, cudaBackEnd::Nx);
	if (debug && call_count <= 10){
		cudaBackEnd::cudaLog << " 1 Frame generation completed " << std::endl;
	}

	//// For next iteration
	hipMemset(cudaBackEnd::d_bfHR, 0, pixels * sizeof(float));
	call_count++;

	return outArray;
}

///////// Divided functions /////////

int cudaBackEnd::readAndBeamForm(bool debug) {

	// perform b-mode generation here using cuda
	const int TILE_SIZE = 4;
	int MASK_WIDTH = 364;
	const int MAX_LINE = 256;
	static int call_count = 0;
	//---------------------------------------------------

	static double pix2 = 0.0;
	unsigned char buf[16 * 1024];
	int row = 0;  // Keep track of how many rows have been added
	char line[MAX_LINE]; // Max possible line length?
	int iteration = 0;
	int errcount = 0;
	unsigned int addr, data;
	unsigned char recvbuf[2048 * 64 * 2];
	const int MAXROWS = 2040;
	LONG rxlen = MAXROWS * 64 * 2;
	//-----------------------------------------------------
	//auto start_bmod = std::chrono::high_resolution_clock::now();
	try
	{
		// unsigned int start = clock();
		while (fgets(line, MAX_LINE, cudaBackEnd::fp)) {
			line[strcspn(line, "\n")] = 0; // Trim trailing newline
			if ((strlen(line) == 0) || (line[0] == ' ') || (line[0] == '#')) {
				//printf("Skipping [%s]\n", line);
			}
			else if (line[0] == 'O') {
				sscanf_s(line, "O %04X %08X ", &addr, &data);
				//printf("Write %08X to Obelix %04X\n", data, addr);
				row = insert_row(buf, row, addr, data);
			}
			else if (line[0] == 'T') {
				sscanf_s(line, "T %04X %08X ", &addr, &data);
				row = insert_row(buf, row, 0x6, 0x40000000 | addr);
				row = insert_row(buf, row, 0x7, data);
				row = insert_row(buf, row, 0x6, 0xC0000000 | addr);
				//printf("Write %08X to TX %04X\n", data, addr);
			}
			else if (line[0] == 'A') {
				sscanf_s(line, "A %04X %08X ", &addr, &data);
				row = insert_row(buf, row, 0x6, 0x00000000 | addr);
				row = insert_row(buf, row, 0x7, data);
				row = insert_row(buf, row, 0x6, 0x80000000 | addr);
				//printf("Write %08X to AFE %04X\n", data, addr);
			}
			else if (line[0] == 'C') {  // CAPTURE
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x01);
				//write_rows(ept, buf, row);  // Send commands
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x10);
				//write_rows(ept, buf, row);  // Send commands
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x00);
				cudaBackEnd::ept_in->Abort();
				cudaBackEnd::ept_in->Reset();
				write_rows(cudaBackEnd::ept, buf, row);  // Send commands
				wait(1);
				if (read_chunk(cudaBackEnd::ept_in, recvbuf, rxlen)) {
					short* rxdata = (short*)(recvbuf);
					for (int i = 0; i < rxlen / 2; i++) {
						if (rxdata[i] >= 512) rxdata[i] -= 1024;
					}
					// Trying to read only first N-1 rows and discard 1st sample
					for (int i = 0; i < 64; i++) {
						for (int j = 0; j < MAXROWS - 1; j++) {
							//rximg[iteration][i][j] = rxdata[j*64+i+2];
							cudaBackEnd::rximg2[i * MAXROWS + j] = rxdata[j * 64 + i + 2];
						}
					}
					//saveToFile(iteration, rxlen, recvbuf);
				}
				else {
					errcount++;
				}
				//hipMemcpy(cudaBackEnd::d_data, cudaBackEnd::rximg2, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels, hipMemcpyHostToDevice);
				//beamformingLR3 << <(cudaBackEnd::pixels / 256) * cudaBackEnd::channels, 256 >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_tx_delay, cudaBackEnd::d_rx_delay, cudaBackEnd::d_data, cudaBackEnd::d_rx_apod, cudaBackEnd::samples, cudaBackEnd::pixels, iteration, cudaBackEnd::num_frames, cudaBackEnd::channels);
				//hipGetLastError();
				//hipDeviceSynchronize();

				hipMemcpyAsync(cudaBackEnd::d_data, cudaBackEnd::rximg2, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels, 
					hipMemcpyHostToDevice, cudaBackEnd::stream);

				//beamformingLR3 << <cudaBackEnd::NBK1, cudaBackEnd::BKZ1, 0, cudaBackEnd::stream >> > 
				//	(cudaBackEnd::d_bfHR, cudaBackEnd::d_tx_delay, cudaBackEnd::d_rx_delay, cudaBackEnd::d_data, cudaBackEnd::d_rx_apod,
				//	cudaBackEnd::samples, cudaBackEnd::pixels, iteration, cudaBackEnd::num_frames, cudaBackEnd::channels);

				beamformingLR3 << <(cudaBackEnd::pixels / 256) * cudaBackEnd::channels, 256, 0, cudaBackEnd::stream >> > (cudaBackEnd::d_bfHR,
					cudaBackEnd::d_tx_delay, cudaBackEnd::d_rx_delay, cudaBackEnd::d_data, cudaBackEnd::d_rx_apod, 
					cudaBackEnd::samples, cudaBackEnd::pixels, iteration, cudaBackEnd::num_frames, cudaBackEnd::channels);

				hipGetLastError();
				hipStreamSynchronize(cudaBackEnd::stream);

				iteration++; // Increment iteration after saving to image
				row = 0;   // Reset buffer for next iteration
			}
			else {
				printf("Don't know how to handle [%s] yet.\n", line);
			}
		}

	}
	catch (std::exception& e) {
		return 11;
	}

	return 0;
}

int cudaBackEnd::filterBeamForm(bool debug) {
	try
	{
		//// check for nan values,
		// isnan_test_array << <cudaBackEnd::BKZ3, NBK3, 0, cudaBackEnd::stream >> > (cudaBackEnd::d_bfHR, cudaBackEnd::pixels);
		isnan_test_array << <cudaBackEnd::pixels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads, 0, cudaBackEnd::stream >> > (cudaBackEnd::d_bfHR, cudaBackEnd::pixels);
		hipGetLastError();
		//hipDeviceSynchronize();
		hipStreamSynchronize(cudaBackEnd::stream);

		//////////// Bandpass filtering using shared memory /////////////////////
		// BPfilter1SharedMem << <(cudaBackEnd::pixels + TILE_SIZE - 1) / TILE_SIZE, TILE_SIZE, 0, cudaBackEnd::stream >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_filt_coeff, cudaBackEnd::pixels, cudaBackEnd::d_bfHRBP);
		BPfilter1SharedMem << <cudaBackEnd::NBK2, cudaBackEnd::BKZ2, 0, cudaBackEnd::stream >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_filt_coeff, cudaBackEnd::pixels, cudaBackEnd::d_bfHRBP);
		hipGetLastError();
		//hipDeviceSynchronize();
		hipStreamSynchronize(cudaBackEnd::stream);

		//////////////// reshape of the beamformed data ///////////////
		// reshape_columnwise << <cudaBackEnd::BKZ3, NBK3, 0, cudaBackEnd::stream >> > (cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::d_bfHRBP);
		reshape_columnwise << <cudaBackEnd::pixels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads, 0, cudaBackEnd::stream >> > (cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::d_bfHRBP);
		hipGetLastError();
		//hipDeviceSynchronize();
		hipStreamSynchronize(cudaBackEnd::stream);
	}
	catch (std::exception& e)
	{
		return 12;
	}

	return 0;

}

int cudaBackEnd::envelopAndCompress(bool debug) {

	// Adding envelop detection and log compression
	//auto start_env = std::chrono::high_resolution_clock::now();
	real2complex << <cudaBackEnd::NBK3, cudaBackEnd::BKZ3, 0, cudaBackEnd::stream >> > (cudaBackEnd::dev_beamformed_data1, cudaBackEnd::d_xflatComplex);
	hipGetLastError();
	hipStreamSynchronize(cudaBackEnd::stream);
	hipfftExecC2C(cudaBackEnd::plan, cudaBackEnd::d_xflatComplex, cudaBackEnd::d_fftComplex, HIPFFT_FORWARD);
	hipGetLastError();
	hipStreamSynchronize(cudaBackEnd::stream);
	hipfftExecC2C(cudaBackEnd::plan, cudaBackEnd::d_fftComplex, cudaBackEnd::d_ifftComplex, HIPFFT_BACKWARD);
	hipGetLastError();
	hipStreamSynchronize(cudaBackEnd::stream);
	// convert t real and imaginary parts
	splitComplex << <cudaBackEnd::NBK3, cudaBackEnd::BKZ3, 0, cudaBackEnd::stream >> > (cudaBackEnd::d_ifftComplex, cudaBackEnd::d_ifftR, cudaBackEnd::d_ifftI);
	hipGetLastError();
	hipStreamSynchronize(cudaBackEnd::stream);
	scalarMult << <cudaBackEnd::NBK3, cudaBackEnd::BKZ3, 0, cudaBackEnd::stream >> > (cudaBackEnd::d_ifftI, cudaBackEnd::d_ifftI, (float)(1.0 / (float)cudaBackEnd::Nz));
	hipGetLastError();
	hipStreamSynchronize(cudaBackEnd::stream);
	magnitide << <cudaBackEnd::NBK3, cudaBackEnd::BKZ3, 0, cudaBackEnd::stream >> > (cudaBackEnd::dev_beamformed_data1, cudaBackEnd::d_ifftI, cudaBackEnd::d_envelop);
	hipGetLastError();
	hipStreamSynchronize(cudaBackEnd::stream);
	//auto stop_env = std::chrono::high_resolution_clock::now();
	// Performing the log transformation to the image to make it enhanced
	// d_envelop is from previous function
	logCompresion << <cudaBackEnd::NBK3, cudaBackEnd::BKZ3, 0, cudaBackEnd::stream >> > (cudaBackEnd::d_envelop, cudaBackEnd::d_logComp, cudaBackEnd::log_c);
	hipStreamSynchronize(cudaBackEnd::stream);

	//auto stop_com = std::chrono::high_resolution_clock::now();

	//auto duration_bmod = std::chrono::duration_cast<std::chrono::microseconds>(start_env - start_bmod);
	//auto duration_env = std::chrono::duration_cast<std::chrono::microseconds>(stop_env - start_env);
	//auto duration_log = std::chrono::duration_cast<std::chrono::microseconds>(stop_com - stop_env);
	//cudaBackEnd::cudaLog << " duration_bmod time taken:  " << duration_bmod.count() << std::endl;
	//cudaBackEnd::cudaLog << " duration_env time taken:  " << duration_env.count() << std::endl;
	//cudaBackEnd::cudaLog << " duration_log time taken:" << duration_log.count() << std::endl;

	//hipMemcpy(cudaBackEnd::env, cudaBackEnd::d_logComp, cudaBackEnd::Nz * cudaBackEnd::Nx * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpyAsync(cudaBackEnd::env, cudaBackEnd::d_logComp, cudaBackEnd::Nz * cudaBackEnd::Nx * sizeof(float), hipMemcpyDeviceToHost, cudaBackEnd::stream);
	//hipMemcpy(cudaBackEnd::env, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::Nz * cudaBackEnd::Nx * sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(env2, cudaBackEnd::d_logComp, cudaBackEnd::Nz * cudaBackEnd::Nx * sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(env3, cudaBackEnd::d_envelop, cudaBackEnd::Nz * cudaBackEnd::Nx * sizeof(float), hipMemcpyDeviceToHost);

	return 0;

}

int cudaBackEnd::warmUp() {
	// warp up the GPU upon initiating the prob
	
	// init with const values and perform the exact same operation
	onesC(cudaBackEnd::rximg2, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels);

	hipMemcpyAsync(cudaBackEnd::d_data, cudaBackEnd::rximg2, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels,
		hipMemcpyHostToDevice, cudaBackEnd::stream);
	beamformingLR3 << <(cudaBackEnd::pixels / 256) * cudaBackEnd::channels, 256, 0, cudaBackEnd::stream >> > (cudaBackEnd::d_bfHR,
		cudaBackEnd::d_tx_delay, cudaBackEnd::d_rx_delay, cudaBackEnd::d_data, cudaBackEnd::d_rx_apod,
		cudaBackEnd::samples, cudaBackEnd::pixels, iteration, cudaBackEnd::num_frames, cudaBackEnd::channels);
	
	// Basic operation completed
	hipGetLastError();
	hipStreamSynchronize(cudaBackEnd::stream);
	hipMemset(cudaBackEnd::d_bfHR, 0, pixels * sizeof(float));

}

// function to init the block size
int cudaBackEnd::calculateThreads() {
	// Fiuntion declaring the number of blocks and threads for each operation
	// diff size threads and blocks are used to three diff opperations
	// 1: h/w reading and initialization
	// 2: Filtering of the B-mode image
	// envelop detection and compression of B-mode image
	
	// pixels:262144, channels:64, num_threads:1024, TILE_SIZE:4


	cudaBackEnd::NBK1 = (cudaBackEnd::pixels / cudaBackEnd::Nx) * cudaBackEnd::channels;// 65536
	cudaBackEnd::BKZ1 = cudaBackEnd::Nx;												// cols size 256

	cudaBackEnd::NBK2 = (cudaBackEnd::pixels + TILE_SIZE - 1) / TILE_SIZE;				// 65536.36
	cudaBackEnd::BKZ2 = TILE_SIZE;

	cudaBackEnd::NBK3 = cudaBackEnd::Nz;
	cudaBackEnd::BKZ3 = cudaBackEnd::Nx;

	// cudaBackEnd::NBK = cudaBackEnd::Nx;
	// cudaBackEnd::BKZ = dim3(cudaBackEnd::Nz);


}


/////////////  old function ///////////////
///////////////////////////////////////////

float** cudaBackEnd::computeBModeImg() {

	//hipMalloc((void**)&cudaBackEnd::d_bfHR, cudaBackEnd::pixels * sizeof(float));
	//hipMemset(cudaBackEnd::d_bfHR, 0, cudaBackEnd::pixels * sizeof(float));
	//hipMalloc((void**)&cudaBackEnd::dev_beamformed_data1, cudaBackEnd::pixels * sizeof(float));
	//hipMalloc((void**)&cudaBackEnd::d_bfHRBP, sizeof(float)* cudaBackEnd::pixels);

	//////////---<H/W INIT>---////////

	//CCyUSBDevice* USBDevice;	// H/W initilization1
	//CCyControlEndPoint* ept;	// H/W initilization2
	//CCyBulkEndPoint* ept_in;	// Endpoint for reading back data
	//USBDevice = new CCyUSBDevice(NULL);
	//ept = USBDevice->ControlEndPt; // Obtain the control endpoint pointer
	//ept_in = USBDevice->BulkInEndPt;

	if (!ept) {
		printf("Could not get Control endpoint.\n");
		//return 1;
	}
	// Send a vendor request (bRequest = 0x05) to the device
	ept->Target = TGT_DEVICE;
	ept->ReqType = REQ_VENDOR;
	ept->Direction = DIR_TO_DEVICE;
	ept->ReqCode = 0x05;
	ept->Value = 1;
	ept->Index = 0;
	ept->TimeOut = 100;				// set timeout to 100ms for quick response

	if (!ept_in) {
		//printf("No IN endpoint??\n");
		return nullptr;
	}
	ept_in->MaxPktSize = 16384;
	ept_in->TimeOut = 100;			// set timeout to 100ms for readin


	//////////-<set reading params>-/////////

	const int MAX_LINE = 256;
	const int N_RX = 64;
	unsigned char buf[16 * 1024];

	errno_t err;
	char line[MAX_LINE]; // Max possible line length?
	FILE* fp;
	if ((err = fopen_s(&fp, "out25_curvi.txt", "r")) != 0) {
		//printf("Could not open config file for reading.\n");
		return nullptr;
	}

	int iteration = 0;
	int errcount = 0;
	int row = 0;					// Keep track of how many rows have been added
	unsigned int addr, data;
	unsigned char recvbuf[2048 * N_RX * 2];
	const int MAXROWS = 2040;
	LONG rxlen = MAXROWS * N_RX * 2;
	cudaBackEnd::env = new float[cudaBackEnd::pixels];


	//unsigned int start = clock();
	while (fgets(line, cudaBackEnd::MAX_LINE, fp)) {
		line[strcspn(line, "\n")] = 0; // Trim trailing newline
		if ((strlen(line) == 0) || (line[0] == ' ') || (line[0] == '#')) {
		}
		else if (line[0] == 'O') {
			sscanf_s(line, "O %04X %08X ", &addr, &data);
			row = insert_row(buf, row, addr, data);
		}
		else if (line[0] == 'T') {
			sscanf_s(line, "T %04X %08X ", &addr, &data);
			row = insert_row(buf, row, 0x6, 0x40000000 | addr);
			row = insert_row(buf, row, 0x7, data);
			row = insert_row(buf, row, 0x6, 0xC0000000 | addr);
		}
		else if (line[0] == 'A') {
			sscanf_s(line, "A %04X %08X ", &addr, &data);
			row = insert_row(buf, row, 0x6, 0x00000000 | addr);
			row = insert_row(buf, row, 0x7, data);
			row = insert_row(buf, row, 0x6, 0x80000000 | addr);
		}
		else if (line[0] == 'C') {  // CAPTURE STARTS
			row = insert_row(buf, row, 0x4, 0x01);
			row = insert_row(buf, row, 0x4, 0x10);
			row = insert_row(buf, row, 0x4, 0x00);

			ept_in->Abort();
			ept_in->Reset();

			write_rows(ept, buf, row);  // Send commands

			wait(1);
			if (read_chunk(ept_in, recvbuf, rxlen)) {
				short* rxdata = (short*)(recvbuf);
				for (int i = 0; i < rxlen / 2; i++) {
					if (rxdata[i] >= 512) rxdata[i] -= 1024;
				}
				// Trying to read only first N-1 rows and discard 1st sample
				for (int i = 0; i < N_RX; i++) {
					for (int j = 0; j < MAXROWS - 1; j++) {
						//rximg[iteration][i][j] = rxdata[j*64+i+2];
						if (iteration < 29) {      // start from 0 index, so 30-1 
							cudaBackEnd::rximg[i * MAXROWS + j] = rxdata[j * N_RX + i + 2];
						}
						else if (iteration > 91) {
							cudaBackEnd::rximg[(i + 64) * MAXROWS + j] = rxdata[j * N_RX + i + 2];
						}
						else {
							cudaBackEnd::rximg[(i + iteration - 28) * MAXROWS + j] = rxdata[j * N_RX + i + 2];
						}
						//rximg[i * MAXROWS + j] = rxdata[j * N_RX + i + 2];
					}
				}
				//saveToFile(iteration, rxlen, recvbuf);
			}
			else {
				errcount++;
			}

			hipMemcpy(cudaBackEnd::d_data, cudaBackEnd::rximg, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels, hipMemcpyHostToDevice);
			beamformingLR3 << <(cudaBackEnd::pixels / 256) * cudaBackEnd::channels, 256 >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_tx_delay, cudaBackEnd::d_rx_delay, cudaBackEnd::d_data, cudaBackEnd::d_rx_apod, cudaBackEnd::samples, cudaBackEnd::pixels, iteration, cudaBackEnd::num_frames, cudaBackEnd::channels);
			hipGetLastError();
			hipDeviceSynchronize();

			iteration++;	// Increment iteration after saving to image
			row = 0;		// Reset buffer for next iteration
		}
		else {
			printf("Don't know how to handle [%s] yet.\n", line);
		}
	}

	//////////// Bandpass filtering using shared memory /////////////////////
	BPfilter1SharedMem << <(cudaBackEnd::pixels + cudaBackEnd::TILE_SIZE - 1) / cudaBackEnd::TILE_SIZE, cudaBackEnd::TILE_SIZE >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_filt_coeff, cudaBackEnd::pixels, cudaBackEnd::d_bfHRBP);
	hipGetLastError();
	hipDeviceSynchronize();

	//////////////// reshape of the beamformed data ///////////////
	reshape_columnwise << <cudaBackEnd::pixels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::d_bfHRBP);
	hipGetLastError();
	hipDeviceSynchronize();
	hipMemcpy(cudaBackEnd::env, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::Nz * cudaBackEnd::Nx * sizeof(float), hipMemcpyDeviceToHost);

	//char fileout[200];
	//sprintf(fileout, "sample_output/b_curve_mode.csv"); //all the 16 inputs are arranged in a single file
	//csv_write_mat(cudaBackEnd::env, fileout, cudaBackEnd::Nz, cudaBackEnd::Nx);

	///////-<free up for next iteration>-/////////
	zeroC(cudaBackEnd::rximg, cudaBackEnd::samples * cudaBackEnd::N_elements);   // set rx_img array values to zero.
	hipMemset(cudaBackEnd::d_bfHR, 0, cudaBackEnd::pixels * sizeof(float));
	hipMemset(cudaBackEnd::dev_beamformed_data1, 0, cudaBackEnd::pixels * sizeof(float));

	float** outArray = convertsingto2darray(env, Nz, Nx);
	return outArray;

}
float** cudaBackEnd::computeBModeImgLinDev()
{
	// perform b-mode generation here using cuda
	const int TILE_SIZE = 4;
	int MASK_WIDTH = 364;
	const int MAX_LINE = 256;
	//// Computer (NIVIDIA) parametrs
	int num_threads = 1024;
	/// Apodization parameters
	float rx_f_number = 2.0;
	/////// Ultrasound scanner parametrs
	//float depth = 49.28;      // Depth of imaging in mm
	int samples = 2040;         // # of samples in depth direction
	int N_elements = 64;        // # of transducer elements
	float sampling_frequency = 32e6;   // sampling frequency
	float c = 1540.0;		 // speed of sound [m/s]	
	int N_active = 8;        // Active transmit elmeents
	float pitch = 0.3 / 1000;// spacing between the elements
	float aper_len = (N_elements - 1) * pitch * 1000;  //aperture foot print 
	float zd = pitch * N_active / (float)2;            // virtual src distance from transducer array 
	float sample_spacing = c / sampling_frequency / (float)2;
	float del_convert = sampling_frequency / c;  // used in delay calculation
	int channels = 64;							 // number of A-lines data used for beamforming
	//// Beamforming "Grid" parameters
	int Nx = 256;			// 256 Lateral spacing
	int Nz = 1024;			//1024 Axial spacing
	int pixels = Nz * Nx;
	int pix_cha = pixels * channels;// Nz*Nx*128 This array size is used for Apodization
	int num_frames = 57;			// number of low resolution images
	int skip_frames = 1;			//
	// Global variable to store the full image.  Cannot be declared local as memory alloc may fail due to large size.
	float rximg[64 * 2040];

	CCyUSBDevice* USBDevice;	// H/W initilization1
	CCyControlEndPoint* ept;	// H/W initilization2
	CCyBulkEndPoint* ept_in;	// Endpoint for reading back data
	USBDevice = new CCyUSBDevice(NULL);
	ept = USBDevice->ControlEndPt;	// Obtain the control endpoint pointer
	if (!ept) {
		printf("Could not get Control endpoint.\n");
		//return 1;
	}
	ept->Target = TGT_DEVICE;		// Send a vendor request (bRequest = 0x05) to the device
	ept->ReqType = REQ_VENDOR;
	ept->Direction = DIR_TO_DEVICE;
	ept->ReqCode = 0x05;
	ept->Value = 1;
	ept->Index = 0;
	ept->TimeOut = 100;  // set timeout to 100ms for quick response

	ept_in = USBDevice->BulkInEndPt;
	if (!ept_in) {
		//printf("No IN endpoint??\n");
		exit(1);
	}
	ept_in->MaxPktSize = 16384;
	ept_in->TimeOut = 100;  // set timeout to 100ms for reading

	static double pix2 = 0.0;
	unsigned char buf[16 * 1024];
	int row = 0;  // Keep track of how many rows have been added
	errno_t err;
	char line[MAX_LINE]; // Max possible line length?
	FILE* fp;
	if ((err = fopen_s(&fp, "out25.txt", "r")) != 0) {
		//printf("Could not open config file for reading.\n");
		//cv::imwrite("errorMat2.png", testMat0);
		exit(1);
	}

	int iteration = 0;
	int errcount = 0;
	unsigned int addr, data;
	unsigned char recvbuf[2048 * 64 * 2];
	const int MAXROWS = 2040;
	LONG rxlen = MAXROWS * 64 * 2;

	// unsigned int start = clock();
	while (fgets(line, MAX_LINE, fp)) {
		line[strcspn(line, "\n")] = 0; // Trim trailing newline
		if ((strlen(line) == 0) || (line[0] == ' ') || (line[0] == '#')) {
			//printf("Skipping [%s]\n", line);
		}
		else if (line[0] == 'O') {
			sscanf_s(line, "O %04X %08X ", &addr, &data);
			//printf("Write %08X to Obelix %04X\n", data, addr);
			row = insert_row(buf, row, addr, data);
		}
		else if (line[0] == 'T') {
			sscanf_s(line, "T %04X %08X ", &addr, &data);
			row = insert_row(buf, row, 0x6, 0x40000000 | addr);
			row = insert_row(buf, row, 0x7, data);
			row = insert_row(buf, row, 0x6, 0xC0000000 | addr);
			//printf("Write %08X to TX %04X\n", data, addr);
		}
		else if (line[0] == 'A') {
			sscanf_s(line, "A %04X %08X ", &addr, &data);
			row = insert_row(buf, row, 0x6, 0x00000000 | addr);
			row = insert_row(buf, row, 0x7, data);
			row = insert_row(buf, row, 0x6, 0x80000000 | addr);
			//printf("Write %08X to AFE %04X\n", data, addr);
		}
		else if (line[0] == 'C') {  // CAPTURE
			//wait(100);
			row = insert_row(buf, row, 0x4, 0x01);
			//write_rows(ept, buf, row);  // Send commands
			//wait(100);
			row = insert_row(buf, row, 0x4, 0x10);
			//write_rows(ept, buf, row);  // Send commands
			//wait(100);
			row = insert_row(buf, row, 0x4, 0x00);
			ept_in->Abort();
			ept_in->Reset();
			write_rows(ept, buf, row);  // Send commands
			wait(1);
			if (read_chunk(ept_in, recvbuf, rxlen)) {
				short* rxdata = (short*)(recvbuf);
				for (int i = 0; i < rxlen / 2; i++) {
					if (rxdata[i] >= 512) rxdata[i] -= 1024;
				}
				// Trying to read only first N-1 rows and discard 1st sample
				for (int i = 0; i < 64; i++) {
					for (int j = 0; j < MAXROWS - 1; j++) {
						//rximg[iteration][i][j] = rxdata[j*64+i+2];
						rximg[i * MAXROWS + j] = rxdata[j * 64 + i + 2];
					}
				}
				//saveToFile(iteration, rxlen, recvbuf);
			}
			else {
				errcount++;
			}
			hipMemcpy(d_data, rximg, sizeof(float) * samples * channels, hipMemcpyHostToDevice);
			beamformingLR3 << <(pixels / 256) * channels, 256 >> > (d_bfHR, d_tx_delay, d_rx_delay, d_data, d_rx_apod, samples, pixels, iteration, num_frames, channels);
			hipGetLastError();
			hipDeviceSynchronize();
			iteration++; // Increment iteration after saving to image
			row = 0;   // Reset buffer for next iteration
		}
		else {
			printf("Don't know how to handle [%s] yet.\n", line);
		}
	}

	//// check for nan values,
	isnan_test_array << <pixels / num_threads + 1, num_threads >> > (d_bfHR, pixels);
	hipGetLastError();
	hipDeviceSynchronize();
	//////////// Bandpass filtering using shared memory /////////////////////
	BPfilter1SharedMem << <(pixels + TILE_SIZE - 1) / TILE_SIZE, TILE_SIZE >> > (d_bfHR, d_filt_coeff, pixels, d_bfHRBP);
	hipGetLastError();
	hipDeviceSynchronize();
	//////////////// reshape of the beamformed data ///////////////
	reshape_columnwise << <pixels / num_threads + 1, num_threads >> > (Nx, Nz, dev_beamformed_data1, d_bfHRBP);
	hipGetLastError();
	hipDeviceSynchronize();
	hipMemcpy(env, dev_beamformed_data1, Nz * Nx * sizeof(float), hipMemcpyDeviceToHost);
	//char fileout[200];
	//sprintf(fileout, "sample_output\\b_mode_%d.csv", 1); //all the 16 inputs are arranged in a single file
	//csv_write_mat(env, fileout, Nz, Nx);
	float** outArray = convertsingto2darray(env, Nz, Nx);

	return outArray;
}
float** cudaBackEnd::computeBModeImgLinDev2(bool debug)
{
	// perform b-mode generation here using cuda
	const int TILE_SIZE = 4;
	int MASK_WIDTH = 364;
	const int MAX_LINE = 256;
	static int call_count = 0;


	////// Computer (NIVIDIA) parametrs
	//int num_threads = 1024;
	///// Apodization parameters
	//float rx_f_number = 2.0;
	///////// Ultrasound scanner parametrs
	////float depth = 49.28;      // Depth of imaging in mm
	//int samples = 2040;         // # of samples in depth direction
	//int N_elements = 64;        // # of transducer elements
	//float sampling_frequency = 32.0e6;   // sampling frequency
	//float c = 1540.0;		 // speed of sound [m/s]	
	//int N_active = 8;        // Active transmit elmeents
	//float pitch = 0.3 / 1000;// spacing between the elements
	//float aper_len = (N_elements - 1) * pitch * 1000;  //aperture foot print 
	//float zd = pitch * N_active / (float)2;            // virtual src distance from transducer array 
	//float sample_spacing = c / sampling_frequency / (float)2;
	//float del_convert = sampling_frequency / c;  // used in delay calculation
	//int channels = 64;							 // number of A-lines data used for beamforming
	////// Beamforming "Grid" parameters
	//int Nx = 256;			// 256 Lateral spacing
	//int Nz = 1024;			//1024 Axial spacing
	//int pixels = Nz * Nx;
	//int pix_cha = pixels * channels;// Nz*Nx*128 This array size is used for Apodization
	//int num_frames = 57;			// number of low resolution images
	//int skip_frames = 1;			//

	cudaBackEnd::env = new float[cudaBackEnd::pixels];
	//float* env2 = new float[cudaBackEnd::pixels];
	//float* env3 = new float[cudaBackEnd::pixels];
	// Global variable to store the full image.  Cannot be declared local as memory alloc may fail due to large size.
	cudaBackEnd::rximg2 = new float[cudaBackEnd::N_elements * cudaBackEnd::samples];
	zeroC(cudaBackEnd::rximg2, cudaBackEnd::samples * cudaBackEnd::N_elements);   // set rx_img array values to zero.
	//--------------------------------------------------

	int ok = cudaBackEnd::initSettingFile("out25.txt");
	if (debug && call_count == 0)
		cudaBackEnd::cudaLog << "CUDA B-Mode for linear prob starts setting file read" << std::endl;
	//---------------------------------------------------

	static double pix2 = 0.0;
	unsigned char buf[16 * 1024];
	int row = 0;  // Keep track of how many rows have been added
	char line[MAX_LINE]; // Max possible line length?
	int iteration = 0;
	int errcount = 0;
	unsigned int addr, data;
	unsigned char recvbuf[2048 * 64 * 2];
	const int MAXROWS = 2040;
	LONG rxlen = MAXROWS * 64 * 2;
	//-----------------------------------------------------
	//auto start_bmod = std::chrono::high_resolution_clock::now();
	try
	{
		// unsigned int start = clock();
		while (fgets(line, MAX_LINE, cudaBackEnd::fp)) {
			line[strcspn(line, "\n")] = 0; // Trim trailing newline
			if ((strlen(line) == 0) || (line[0] == ' ') || (line[0] == '#')) {
				//printf("Skipping [%s]\n", line);
			}
			else if (line[0] == 'O') {
				sscanf_s(line, "O %04X %08X ", &addr, &data);
				//printf("Write %08X to Obelix %04X\n", data, addr);
				row = insert_row(buf, row, addr, data);
			}
			else if (line[0] == 'T') {
				sscanf_s(line, "T %04X %08X ", &addr, &data);
				row = insert_row(buf, row, 0x6, 0x40000000 | addr);
				row = insert_row(buf, row, 0x7, data);
				row = insert_row(buf, row, 0x6, 0xC0000000 | addr);
				//printf("Write %08X to TX %04X\n", data, addr);
			}
			else if (line[0] == 'A') {
				sscanf_s(line, "A %04X %08X ", &addr, &data);
				row = insert_row(buf, row, 0x6, 0x00000000 | addr);
				row = insert_row(buf, row, 0x7, data);
				row = insert_row(buf, row, 0x6, 0x80000000 | addr);
				//printf("Write %08X to AFE %04X\n", data, addr);
			}
			else if (line[0] == 'C') {  // CAPTURE
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x01);
				//write_rows(ept, buf, row);  // Send commands
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x10);
				//write_rows(ept, buf, row);  // Send commands
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x00);
				cudaBackEnd::ept_in->Abort();
				cudaBackEnd::ept_in->Reset();
				write_rows(cudaBackEnd::ept, buf, row);  // Send commands
				wait(1);
				if (read_chunk(cudaBackEnd::ept_in, recvbuf, rxlen)) {
					short* rxdata = (short*)(recvbuf);
					for (int i = 0; i < rxlen / 2; i++) {
						if (rxdata[i] >= 512) rxdata[i] -= 1024;
					}
					// Trying to read only first N-1 rows and discard 1st sample
					for (int i = 0; i < 64; i++) {
						for (int j = 0; j < MAXROWS - 1; j++) {
							//rximg[iteration][i][j] = rxdata[j*64+i+2];
							cudaBackEnd::rximg2[i * MAXROWS + j] = rxdata[j * 64 + i + 2];
						}
					}
					//saveToFile(iteration, rxlen, recvbuf);
				}
				else {
					errcount++;
				}
				hipMemcpy(cudaBackEnd::d_data, cudaBackEnd::rximg2, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels, hipMemcpyHostToDevice);
				beamformingLR3 << <(cudaBackEnd::pixels / 256) * cudaBackEnd::channels, 256 >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_tx_delay, cudaBackEnd::d_rx_delay, cudaBackEnd::d_data, cudaBackEnd::d_rx_apod, cudaBackEnd::samples, cudaBackEnd::pixels, iteration, cudaBackEnd::num_frames, cudaBackEnd::channels);
				hipGetLastError();
				hipDeviceSynchronize();
				iteration++; // Increment iteration after saving to image
				row = 0;   // Reset buffer for next iteration
			}
			else {
				printf("Don't know how to handle [%s] yet.\n", line);
			}
		}

		if (debug && call_count == 0)
			cudaBackEnd::cudaLog << "CUDA B-Mode for linear prob while loop completed" << std::endl;
	}
	catch (std::exception& e) {
		return nullptr;
	}


	try
	{
		//// check for nan values,
		isnan_test_array << <cudaBackEnd::pixels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_bfHR, cudaBackEnd::pixels);
		hipGetLastError();
		hipDeviceSynchronize();

		//////////// Bandpass filtering using shared memory /////////////////////
		BPfilter1SharedMem << <(cudaBackEnd::pixels + TILE_SIZE - 1) / TILE_SIZE, TILE_SIZE >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_filt_coeff, cudaBackEnd::pixels, cudaBackEnd::d_bfHRBP);
		hipGetLastError();
		hipDeviceSynchronize();
		if (debug && call_count == 0)
			cudaBackEnd::cudaLog << " BPfilter1SharedMem completed " << std::endl;

		//////////////// reshape of the beamformed data ///////////////
		reshape_columnwise << <cudaBackEnd::pixels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::d_bfHRBP);
		hipGetLastError();
		hipDeviceSynchronize();

		// Adding envelop detection and log compression
		//auto start_env = std::chrono::high_resolution_clock::now();
		real2complex << <cudaBackEnd::NBK, cudaBackEnd::BKZ >> > (cudaBackEnd::dev_beamformed_data1, cudaBackEnd::d_xflatComplex);
		hipGetLastError();
		hipDeviceSynchronize();
		hipfftExecC2C(cudaBackEnd::plan, cudaBackEnd::d_xflatComplex, cudaBackEnd::d_fftComplex, HIPFFT_FORWARD);
		hipGetLastError();
		hipDeviceSynchronize();
		hipfftExecC2C(cudaBackEnd::plan, cudaBackEnd::d_fftComplex, cudaBackEnd::d_ifftComplex, HIPFFT_BACKWARD);
		hipGetLastError();
		hipDeviceSynchronize();
		// convert t real and imaginary parts
		splitComplex << <cudaBackEnd::NBK, cudaBackEnd::BKZ >> > (cudaBackEnd::d_ifftComplex, cudaBackEnd::d_ifftR, cudaBackEnd::d_ifftI);
		hipGetLastError();
		hipDeviceSynchronize();
		scalarMult << <cudaBackEnd::NBK, cudaBackEnd::BKZ >> > (cudaBackEnd::d_ifftI, cudaBackEnd::d_ifftI, (float)(1.0 / (float)cudaBackEnd::Nz));
		hipGetLastError();
		hipDeviceSynchronize();
		magnitide << <cudaBackEnd::NBK, cudaBackEnd::BKZ >> > (cudaBackEnd::dev_beamformed_data1, cudaBackEnd::d_ifftI, cudaBackEnd::d_envelop);
		hipGetLastError();
		hipDeviceSynchronize();
		//auto stop_env = std::chrono::high_resolution_clock::now();
		// Performing the log transformation to the image to make it enhanced
		// d_envelop is from previous function
		logCompresion << <cudaBackEnd::NBK, cudaBackEnd::BKZ >> > (cudaBackEnd::d_envelop, cudaBackEnd::d_logComp, cudaBackEnd::log_c);
		//auto stop_com = std::chrono::high_resolution_clock::now();

		//auto duration_bmod = std::chrono::duration_cast<std::chrono::microseconds>(start_env - start_bmod);
		//auto duration_env = std::chrono::duration_cast<std::chrono::microseconds>(stop_env - start_env);
		//auto duration_log = std::chrono::duration_cast<std::chrono::microseconds>(stop_com - stop_env);
		//cudaBackEnd::cudaLog << " duration_bmod time taken:  " << duration_bmod.count() << std::endl;
		//cudaBackEnd::cudaLog << " duration_env time taken:  " << duration_env.count() << std::endl;
		//cudaBackEnd::cudaLog << " duration_log time taken:" << duration_log.count() << std::endl;

		//hipMemcpy(cudaBackEnd::env, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::Nz * cudaBackEnd::Nx * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(cudaBackEnd::env, cudaBackEnd::d_logComp, cudaBackEnd::Nz * cudaBackEnd::Nx * sizeof(float), hipMemcpyDeviceToHost);
		//hipMemcpy(env2, cudaBackEnd::d_logComp, cudaBackEnd::Nz * cudaBackEnd::Nx * sizeof(float), hipMemcpyDeviceToHost);
		//hipMemcpy(env3, cudaBackEnd::d_envelop, cudaBackEnd::Nz * cudaBackEnd::Nx * sizeof(float), hipMemcpyDeviceToHost);

		if (debug && call_count == 0)
			cudaBackEnd::cudaLog << " Reshape_columnwise completed " << std::endl;
		//cudaBackEnd::cudaLog << " Nz*Nx" << cudaBackEnd::Nz * cudaBackEnd::Nx << std::endl;

		if (debug && call_count == 0)
		{
			char fileout[200];
			sprintf(fileout, "sample_output\\b_mode_%d.csv", 1); //all the 16 inputs are arranged in a single file
			csv_write_mat(env, fileout, Nz, Nx);
			//sprintf(fileout, "sample_output\\log_com%d.csv", 1); //all the 16 inputs are arranged in a single file
			//csv_write_mat(env2, fileout, Nz, Nx);
			//sprintf(fileout, "sample_output\\mag_%d.csv", 1); //all the 16 inputs are arranged in a single file
			//csv_write_mat(env3, fileout, Nz, Nx);
		}

	}
	catch (std::exception& e)
	{
		return nullptr;
	}


	//double** outArray = convertsingto2darray(cudaBackEnd::env, cudaBackEnd::Nz, cudaBackEnd::Nx);
	float** outArray = convertsingto2darray(cudaBackEnd::env, cudaBackEnd::Nz, cudaBackEnd::Nx);
	if (debug && call_count <= 10)
	{
		cudaBackEnd::cudaLog << " 1 Frame generation completed " << std::endl;
	}




	//// For next iteration
	hipMemset(cudaBackEnd::d_bfHR, 0, pixels * sizeof(float));
	call_count++;

	return outArray;
}

/////// function without debuging /////////
///////////////////////////////////////////

//int cudaBackEnd::initGPUprobeC(double* probPrms, bool debug) {
//
//	const int MASK_WIDTH = 364;
//	std::ofstream mFile2;
//	mFile2.open("sample_output/testclass.txt");
//	mFile2 << "OK" << std::endl;
//
//	cudaBackEnd::PI = (float)probPrms[3]; mFile2 << PI << std::endl;
//	//cudaBackEnd::MASK_WIDTH		= (int)probPrms[4]; mFile2 << MASK_WIDTH << std::endl;
//	//cudaBackEnd::TILE_SIZE		= (int)probPrms[5]; mFile2 << TILE_SIZE << std::endl;
//	cudaBackEnd::num_threads = (int)probPrms[6]; mFile2 << num_threads << std::endl;
//	cudaBackEnd::rx_f_number = (float)probPrms[7]; mFile2 << rx_f_number << std::endl;
//	cudaBackEnd::samples = (int)probPrms[8]; mFile2 << samples << std::endl;
//	cudaBackEnd::N_elements = (int)probPrms[9]; mFile2 << N_elements << std::endl;
//	cudaBackEnd::sampling_frequency = (float)probPrms[10]; mFile2 << sampling_frequency << std::endl;
//	cudaBackEnd::c = (float)probPrms[11]; mFile2 << c << std::endl;
//	cudaBackEnd::N_active = (int)probPrms[12]; mFile2 << N_active << std::endl;
//	cudaBackEnd::channels = (int)probPrms[13]; mFile2 << channels << std::endl;
//	cudaBackEnd::Nx = (int)probPrms[14]; mFile2 << Nx << std::endl;
//	cudaBackEnd::Nz = (int)probPrms[15]; mFile2 << Nz << std::endl;
//	cudaBackEnd::frames = (int)probPrms[16]; mFile2 << frames << std::endl;
//	cudaBackEnd::num_frames = (int)probPrms[17]; mFile2 << num_frames << std::endl;
//	cudaBackEnd::skip_frames = (int)probPrms[18]; mFile2 << skip_frames << std::endl;
//	cudaBackEnd::dBvalue = (int)probPrms[19]; mFile2 << dBvalue << std::endl;
//	cudaBackEnd::pitch = (float)probPrms[20]; mFile2 << pitch << std::endl;
//	cudaBackEnd::aper_len = (float)probPrms[21]; mFile2 << aper_len << std::endl;
//	cudaBackEnd::zd = (float)probPrms[22]; mFile2 << zd << std::endl;
//	cudaBackEnd::sample_spacing = (float)probPrms[23]; mFile2 << sample_spacing << std::endl;
//	cudaBackEnd::del_convert = (float)probPrms[24]; mFile2 << del_convert << std::endl;
//	cudaBackEnd::rc = (float)probPrms[25]; mFile2 << rc << std::endl;
//	cudaBackEnd::scan_angle = (float)probPrms[26]; mFile2 << scan_angle  << std::endl;
//	cudaBackEnd::pixels = (int)probPrms[27]; mFile2 << pixels << std::endl;
//	cudaBackEnd::pix_cha = (int)probPrms[28]; mFile2 << pix_cha << std::endl;
//	
//
//	
//	cudaBackEnd::env = new float[cudaBackEnd::pixels];
//
//	char filename1[200];
//	sprintf(filename1, "b_10M.csv");
//	cudaBackEnd::read_csv_array(cudaBackEnd::filt_coeff, filename1);    // csv file read
//
//	//float* d_filt_coeff = 0;
//	hipMalloc((void**)&cudaBackEnd::d_filt_coeff, sizeof(float) * MASK_WIDTH);
//	hipMemcpy(cudaBackEnd::d_filt_coeff, filt_coeff, sizeof(float) * MASK_WIDTH, hipMemcpyHostToDevice);
//
//	////////  Intialization &(or) Memory allocation  //////////////////
//	//float* d_data = 0;   // variable to store raw rf data
//	hipMalloc((void**)&cudaBackEnd::d_data, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels);
//
//	//float* d_bfHR = 0;  // variable to store beamformed high-resolution beamformed image 
//	hipMalloc((void**)&cudaBackEnd::d_bfHR, cudaBackEnd::pixels * sizeof(float));
//	//zeros << <pixels / num_threads + 1, num_threads >> > (d_bfHR, pixels);  
//	hipMemset(cudaBackEnd::d_bfHR, 0, cudaBackEnd::pixels * sizeof(float));
//
//	//float* dev_beamformed_data1 = 0;   // variable to store reshaped beamformed data
//	hipMalloc((void**)&cudaBackEnd::dev_beamformed_data1, cudaBackEnd::pixels * sizeof(float));
//
//	//float* d_bfHRBP = 0;  // variable to store beamformed high-resolution bandpass filtered data
//	hipMalloc((void**)&cudaBackEnd::d_bfHRBP, sizeof(float) * cudaBackEnd::pixels);
//
//	/////////////////// theta positions for all elements ////////////////////
//	//float* d_theta = 0;
//	hipMalloc((void**)&cudaBackEnd::d_theta, cudaBackEnd::N_elements * sizeof(float));
//	range << <cudaBackEnd::Nx / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_theta, (-cudaBackEnd::scan_angle / 2), cudaBackEnd::N_elements, (cudaBackEnd::scan_angle / (cudaBackEnd::N_elements - 1)));
//
//
//	///////////// theta for grid /////////////////  theta = -scan_angle / 2 : scan_angle / (elements - 1) : scan_angle / 2;
//	//float* d_theta1 = 0;
//	hipMalloc((void**)&cudaBackEnd::d_theta1, cudaBackEnd::Nx * sizeof(float));
//	range << <cudaBackEnd::Nx / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_theta1, (-cudaBackEnd::scan_angle / 2), cudaBackEnd::Nx, (cudaBackEnd::scan_angle / (cudaBackEnd::Nx - 1)));
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	////////////// z value////////////////////
//	float dz = cudaBackEnd::sample_spacing * cudaBackEnd::samples / cudaBackEnd::Nz;  // depth / (Nz - 1) / 1000;   // spacing in axial (z) direction in mm;
//	//float* d_z_axis = 0;
//	hipMalloc((void**)&cudaBackEnd::d_z_axis, cudaBackEnd::Nz * sizeof(float));
//	range << <cudaBackEnd::Nz / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_z_axis, 0, cudaBackEnd::Nz, dz);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	//////////////////////////////// x value////////////////////////////////
//	float dx = aper_len / (Nx - 1);
//	//float* d_x_axis = 0;
//	hipMalloc((void**)&cudaBackEnd::d_x_axis, cudaBackEnd::Nx * sizeof(float));
//	range << <cudaBackEnd::Nx / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_x_axis, (-cudaBackEnd::aper_len / 2000), cudaBackEnd::Nx, dx / 1000);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	//////////////// Probe geometry, this info can be taken from transducer file ////////////////////
//	//float* d_probe = 0;
//	hipMalloc((void**)&cudaBackEnd::d_probe, cudaBackEnd::N_elements * sizeof(float));
//	//hipMemcpy(d_probe, probe_ge_x, N_elements * sizeof(double), hipMemcpyHostToDevice);
//	range << <1, cudaBackEnd::N_elements >> > (cudaBackEnd::d_probe, (-cudaBackEnd::aper_len / 2000), cudaBackEnd::N_elements, cudaBackEnd::pitch);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	/////////////////rx aerture calculation using Fnumber///////////////////////////////
//	// rx_aper=rfsca.z/rf_number
//	//float* d_rx_aperture = 0;
//	hipMalloc((void**)&cudaBackEnd::d_rx_aperture, cudaBackEnd::Nz * sizeof(float));
//	element_division << <cudaBackEnd::Nz / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_z_axis, cudaBackEnd::rx_f_number, cudaBackEnd::Nz, cudaBackEnd::d_rx_aperture);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	////////////////////////rx aerture distance////////
//	//float* d_rx_ap_distance = 0;
//	hipMalloc((void**)&cudaBackEnd::d_rx_ap_distance, cudaBackEnd::channels * cudaBackEnd::Nx * sizeof(float));  //20.087 us
//	aperture_distance << <cudaBackEnd::Nx * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_x_axis, cudaBackEnd::d_probe, cudaBackEnd::Nx, cudaBackEnd::channels, cudaBackEnd::d_rx_ap_distance);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	///////////////////apodization/////////////////
//	//float* d_rx_apod = 0;
//	hipMalloc((void**)&cudaBackEnd::d_rx_apod, sizeof(float) * cudaBackEnd::Nz * cudaBackEnd::channels * cudaBackEnd::Nx);
//	apodization << <cudaBackEnd::pixels * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_rx_ap_distance, cudaBackEnd::d_rx_aperture, cudaBackEnd::Nz, cudaBackEnd::Nx, cudaBackEnd::channels, cudaBackEnd::pixels, cudaBackEnd::d_rx_apod);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	//// check for nan values,
//	isnan_test_array << <cudaBackEnd::pixels * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_rx_apod, cudaBackEnd::pixels * cudaBackEnd::channels);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	hipFree(cudaBackEnd::d_rx_aperture);
//	hipFree(cudaBackEnd::d_rx_ap_distance);
//
//	/////////////receive delay calculation /////////////////////////////////////////////
//	//float* d_rx_delay = 0;
//	hipMalloc((void**)&cudaBackEnd::d_rx_delay, cudaBackEnd::pix_cha * sizeof(float));
//	receive_delay << < cudaBackEnd::pixels * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_theta, cudaBackEnd::d_theta1, cudaBackEnd::rc, cudaBackEnd::d_z_axis, cudaBackEnd::channels, cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::del_convert, cudaBackEnd::d_rx_delay);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	/////////////////// theta positions for all elements ////////////////////
//	//float* d_theta_tx = 0;
//	hipMalloc((void**)&cudaBackEnd::d_theta_tx, cudaBackEnd::num_frames * sizeof(float));
//	theta1 << < 1, cudaBackEnd::num_frames >> > (cudaBackEnd::d_theta_tx, cudaBackEnd::d_theta, cudaBackEnd::frames, cudaBackEnd::N_active, cudaBackEnd::skip_frames);
//
//	/////////////////// Transmit delay calculation ////////////////////
//	//float* d_tx_delay = 0;
//	hipMalloc((void**)&cudaBackEnd::d_tx_delay, cudaBackEnd::pixels * cudaBackEnd::num_frames * sizeof(float));
//	//transmitter delay for 16 frames,  
//	transmit_delay << < cudaBackEnd::pixels * cudaBackEnd::num_frames / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_theta1, cudaBackEnd::d_z_axis, cudaBackEnd::rc, cudaBackEnd::d_theta_tx, cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::del_convert, cudaBackEnd::num_frames, cudaBackEnd::zd, cudaBackEnd::d_tx_delay);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	hipFree(cudaBackEnd::d_theta1);
//	hipFree(cudaBackEnd::d_probe);
//	hipFree(cudaBackEnd::d_x_axis);
//	hipFree(cudaBackEnd::d_z_axis);
//	hipFree(cudaBackEnd::d_theta_tx);
//
//	zeroC(cudaBackEnd::rximg, cudaBackEnd::samples * cudaBackEnd::N_elements);   // set rx_img array values to zero.
//
//	mFile2 << "Memmory init completed" << std::endl;
//	mFile2.close();
//
//	return 0;
//}

//double** cudaBackEnd::computeBModeImgDev(bool debug) {
//
//	const int MAX_LINE = 256;
//	const int N_RX = 64;
//	unsigned char buf[16 * 1024];
//	std::ofstream mFile;
//	mFile.open("sample_output/testcomputeimg.txt");
//	mFile << "OK" << std::endl;
//
//	//-----------------------
//	int ok = cudaBackEnd::initSettingFile("out25_curvi.txt");
//	//-----------------------
//
//	//mFile << "h/w init done" << std::endl;
//
//	char line[MAX_LINE]; // Max possible line length?
//	int iteration = 0;
//	int errcount = 0;
//	int row = 0;					// Keep track of how many rows have been added
//	unsigned int addr, data;
//	unsigned char recvbuf[2048 * N_RX * 2];
//	const int MAXROWS = 2040;
//	LONG rxlen = MAXROWS * N_RX * 2;
//	//cudaBackEnd::env = new float[cudaBackEnd::pixels];
//	zeroC(cudaBackEnd::rximg, cudaBackEnd::samples * cudaBackEnd::N_elements);   // set rx_img array values to zero.
//
//	//unsigned int start = clock();
//	while (fgets(line, cudaBackEnd::MAX_LINE, cudaBackEnd::fp)) 
//	{
//		//mFile << line << std::endl;
//		line[strcspn(line, "\n")] = 0; // Trim trailing newline
//		if ((strlen(line) == 0) || (line[0] == ' ') || (line[0] == '#')) {
//			//printf("Skipping [%s]\n", line);
//		}
//		else if (line[0] == 'O') {
//			sscanf_s(line, "O %04X %08X ", &addr, &data);
//			//printf("Write %08X to Obelix %04X\n", data, addr);
//			row = insert_row(buf, row, addr, data);
//		}
//		else if (line[0] == 'T') {
//			sscanf_s(line, "T %04X %08X ", &addr, &data);
//			row = insert_row(buf, row, 0x6, 0x40000000 | addr);
//			row = insert_row(buf, row, 0x7, data);
//			row = insert_row(buf, row, 0x6, 0xC0000000 | addr);
//			//printf("Write %08X to TX %04X\n", data, addr);
//		}
//		else if (line[0] == 'A') {
//			sscanf_s(line, "A %04X %08X ", &addr, &data);
//			row = insert_row(buf, row, 0x6, 0x00000000 | addr);
//			row = insert_row(buf, row, 0x7, data);
//			row = insert_row(buf, row, 0x6, 0x80000000 | addr);
//			//printf("Write %08X to AFE %04X\n", data, addr);
//		}
//		else if (line[0] == 'C') {  // CAPTURE STARTS
//			//wait(100);
//			row = insert_row(buf, row, 0x4, 0x01);
//			//write_rows(ept, buf, row);  // Send commands
//			//wait(100);
//			row = insert_row(buf, row, 0x4, 0x10);
//			//write_rows(ept, buf, row);  // Send commands
//			//wait(100);
//			row = insert_row(buf, row, 0x4, 0x00);
//			//mFile << "insert_row" << std::endl;
//
//			cudaBackEnd::ept_in->Abort();
//			cudaBackEnd::ept_in->Reset();
//
//			//mFile << "abort reset " << std::endl;
//
//			write_rows(cudaBackEnd::ept, buf, row);  // Send commands
//			//mFile << "write_rows" << std::endl;
//			//wait(100);
//			//row = insert_row(buf, row, 0x4, 0x03);
//			//row = insert_row(buf, row, 0x4, 0x10);
//			//row = insert_row(buf, row, 0x4, 0x00);
//			//printf("CAPTURE %2d: ", iteration);
//			//write_rows(ept, buf, row);  // Send commands
//			
//			// One iteration should have 2048 samples * 64 channels * 2 bytes each
//
//			wait(1);
//			if (read_chunk(cudaBackEnd::ept_in, recvbuf, rxlen)) {
//				short* rxdata = (short*)(recvbuf);
//				for (int i = 0; i < rxlen / 2; i++) {
//					if (rxdata[i] >= 512) rxdata[i] -= 1024;
//				}
//				// Trying to read only first N-1 rows and discard 1st sample
//				for (int i = 0; i < N_RX; i++) {
//					for (int j = 0; j < MAXROWS - 1; j++) {
//						//rximg[iteration][i][j] = rxdata[j*64+i+2];
//						if (iteration < 29) {      // start from 0 index, so 30-1 
//							cudaBackEnd::rximg[i * MAXROWS + j] = rxdata[j * N_RX + i + 2];
//						}
//						else if (iteration > 91) {
//							cudaBackEnd::rximg[(i + 64) * MAXROWS + j] = rxdata[j * N_RX + i + 2];
//						}
//						else {
//							cudaBackEnd::rximg[(i + iteration - 28) * MAXROWS + j] = rxdata[j * N_RX + i + 2];
//						}
//						//rximg[i * MAXROWS + j] = rxdata[j * N_RX + i + 2];
//					}
//				}
//				//saveToFile(iteration, rxlen, recvbuf);
//			}
//			else {
//				errcount++;
//			}
//			//mFile << "read_chunk" << std::endl;
//
//			//clock_t begin = clock();   // clock intiated
//
//			hipMemcpy(cudaBackEnd::d_data, cudaBackEnd::rximg, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels, hipMemcpyHostToDevice);
//
//			beamformingLR3 << <(cudaBackEnd::pixels / 256) * cudaBackEnd::channels, 256 >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_tx_delay, cudaBackEnd::d_rx_delay, cudaBackEnd::d_data, cudaBackEnd::d_rx_apod, cudaBackEnd::samples, cudaBackEnd::pixels, iteration, cudaBackEnd::num_frames, cudaBackEnd::channels);
//			//mFile << "beamformingLR3" << std::endl;
//			hipGetLastError();
//			hipDeviceSynchronize();
//
//			//clock_t end = clock();
//			//float elapsed_secs = float(end - begin) / CLOCKS_PER_SEC;
//			//printf("Time for beamforming in ms: %f\n", elapsed_secs * 1000);
//
//			iteration++;	// Increment iteration after saving to image
//			row = 0;		// Reset buffer for next iteration
//		}
//		else {
//			mFile << "Don't know how to handle" << std::endl;
//			printf("Don't know how to handle [%s] yet.\n", line);
//		}
//	}
//
//	//mFile << "while loop completed" << std::endl;
//
//	//////////// Bandpass filtering using shared memory /////////////////////
//	BPfilter1SharedMem << <(cudaBackEnd::pixels + cudaBackEnd::TILE_SIZE - 1) / cudaBackEnd::TILE_SIZE, cudaBackEnd::TILE_SIZE >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_filt_coeff, cudaBackEnd::pixels, cudaBackEnd::d_bfHRBP);
//	hipGetLastError();
//	hipDeviceSynchronize();
//	//mFile << "BPF done" << std::endl;
//
//	//////////////// reshape of the beamformed data ///////////////
//	reshape_columnwise << <cudaBackEnd::pixels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::d_bfHRBP);
//	hipGetLastError();
//	hipDeviceSynchronize();
//	hipMemcpy(cudaBackEnd::env, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::Nz * cudaBackEnd::Nx * sizeof(float), hipMemcpyDeviceToHost);
//	//mFile << "reshape done" << std::endl;
//	char fileout[200];
//	sprintf(fileout, "sample_output/b_curve_mode.csv"); //all the 16 inputs are arranged in a single file
//	csv_write_mat(cudaBackEnd::env, fileout, cudaBackEnd::Nz, cudaBackEnd::Nx);
//	//mFile << "CSV written" << std::endl;
//
//	double** outArray = convertsingto2darray(cudaBackEnd::env, cudaBackEnd::Nz, cudaBackEnd::Nx);
//
//
//	// For next iteration
//	
//	//hipMalloc((void**)&cudaBackEnd::d_data, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels);
//	//hipMemset(cudaBackEnd::d_bfHR, 0, cudaBackEnd::pixels * sizeof(float));
//	//hipMemset(cudaBackEnd::dev_beamformed_data1, 0, cudaBackEnd::pixels * sizeof(float));
//	zeroC(cudaBackEnd::rximg, cudaBackEnd::samples* cudaBackEnd::N_elements);   // set rx_img array values to zero.
//	//cudaBackEnd::fp->
//
//	mFile.close();
//	return outArray;
//
//}

//int cudaBackEnd::initGPUprobeL(double* prob_params, bool debug)
//{
//	//// perform b-mode generation here using cuda
//	//const int TILE_SIZE = 4;
//	//int MASK_WIDTH = 364;
//	//const int MAX_LINE = 256;
//	////// Computer (NIVIDIA) parametrs
//	//int num_threads = 1024;
//	///// Apodization parameters
//	//float rx_f_number = 2.0;
//	///////// Ultrasound scanner parametrs
//	////float depth = 49.28;      // Depth of imaging in mm
//	//int samples = 2040;         // # of samples in depth direction
//	//int N_elements = 64;        // # of transducer elements
//	//float sampling_frequency = 32e6;   // sampling frequency
//	//float c = 1540.0;		 // speed of sound [m/s]	
//	//int N_active = 8;        // Active transmit elmeents
//	//float pitch = 0.3 / 1000;// spacing between the elements
//	//float aper_len = (N_elements - 1) * pitch * 1000;  //aperture foot print 
//	//float zd = pitch * N_active / (float)2;            // virtual src distance from transducer array 
//	//float sample_spacing = c / sampling_frequency / (float)2;
//	//float del_convert = sampling_frequency / c;  // used in delay calculation
//	//int channels = 64;							 // number of A-lines data used for beamforming
//	////// Beamforming "Grid" parameters
//	//int Nx = 256;			// 256 Lateral spacing
//	//int Nz = 1024;			//1024 Axial spacing
//	//int pixels = Nz * Nx;
//	//int pix_cha = pixels * channels;// Nz*Nx*128 This array size is used for Apodization
//	//int num_frames = 57;			// number of low resolution images
//	//int skip_frames = 1;			//
//
//	// perform b-mode generation here using cuda
//	const int TILE_SIZE = prob_params[0];
//	int MASK_WIDTH = prob_params[1];
//	const int MAX_LINE = prob_params[2];
//
//	cudaBackEnd::num_threads = prob_params[3];
//	cudaBackEnd::rx_f_number = prob_params[4];	// Apodization parameters
//	cudaBackEnd::samples = prob_params[5];	// # of samples in depth direction
//	cudaBackEnd::N_elements = prob_params[6];	// # of transducer elements
//	cudaBackEnd::sampling_frequency = prob_params[7];   // sampling frequency
//	cudaBackEnd::c = prob_params[8];	// speed of sound [m/s]	
//	cudaBackEnd::N_active = prob_params[9];   // Active transmit elmeents
//	cudaBackEnd::pitch = prob_params[10];	// spacing between the elements
//	cudaBackEnd::aper_len = prob_params[11];  // aperture foot print 
//	cudaBackEnd::zd = prob_params[12];  // virtual src distance from transducer array 
//	cudaBackEnd::sample_spacing = prob_params[13];
//	cudaBackEnd::del_convert = prob_params[14];  // used in delay calculation
//	cudaBackEnd::channels = prob_params[15];	// number of A-lines data used for beamforming
//	cudaBackEnd::Nx = prob_params[16];	// 256 Lateral spacing Beamforming "Grid" parameters
//	cudaBackEnd::Nz = prob_params[17];	// 1024 Axial spacing
//	cudaBackEnd::pixels = prob_params[18];
//	cudaBackEnd::pix_cha = prob_params[19];	// Nz*Nx*128 This array size is used for Apodization
//	cudaBackEnd::num_frames = prob_params[20];	// number of low resolution images
//	cudaBackEnd::skip_frames = prob_params[21];	//
//
//
//	if (debug)
//	{
//		cudaBackEnd::cudaLog << "num_threads : " << prob_params[3] << std::endl;
//		cudaBackEnd::cudaLog << "rx_f_number : " << prob_params[4] << std::endl;
//		cudaBackEnd::cudaLog << "samples : " << prob_params[5] << std::endl;
//		cudaBackEnd::cudaLog << "N_elements : " << prob_params[6] << std::endl;
//		cudaBackEnd::cudaLog << "sampling_frequency : " << prob_params[7] << std::endl;
//		cudaBackEnd::cudaLog << "c : " << prob_params[8] << std::endl;
//		cudaBackEnd::cudaLog << "N_active : " << prob_params[9] << std::endl;
//		cudaBackEnd::cudaLog << "pitch : " << prob_params[10] << std::endl;
//		cudaBackEnd::cudaLog << "aper_len : " << prob_params[11] << std::endl;
//		cudaBackEnd::cudaLog << "zd : " << prob_params[12] << std::endl;
//		cudaBackEnd::cudaLog << "sample_spacing : " << prob_params[13] << std::endl;
//		cudaBackEnd::cudaLog << "del_convert : " << prob_params[14] << std::endl;
//		cudaBackEnd::cudaLog << "channels : " << prob_params[15] << std::endl;
//		cudaBackEnd::cudaLog << "Nx : " << prob_params[16] << std::endl;
//		cudaBackEnd::cudaLog << "Nz : " << prob_params[17] << std::endl;
//		cudaBackEnd::cudaLog << "pixels : " << prob_params[18] << std::endl;
//		cudaBackEnd::cudaLog << "pix_cha : " << prob_params[19] << std::endl;
//		cudaBackEnd::cudaLog << "num_frames : " << prob_params[20] << std::endl;
//		cudaBackEnd::cudaLog << "skip_frames : " << prob_params[21] << std::endl;
//	}
//
//	char filename3[200];
//	sprintf(filename3, "b_10M.csv");
//	read_csv_array(cudaBackEnd::filt_coeff, filename3);    // csv file read
//	//cv::imwrite("okMat3.png", testMat0);
//
//	// float* d_filt_coeff = 0;
//	hipMalloc((void**)&cudaBackEnd::d_filt_coeff, sizeof(float) * MASK_WIDTH);
//	hipMemcpy(cudaBackEnd::d_filt_coeff, cudaBackEnd::filt_coeff, sizeof(float) * MASK_WIDTH, hipMemcpyHostToDevice);
//
//	////////  Intialization &(or) Memory allocation  //////////////////
//	hipMalloc((void**)&cudaBackEnd::d_data, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels);// variable to store raw rf data
//
//	hipMalloc((void**)&cudaBackEnd::d_bfHR, cudaBackEnd::pixels * sizeof(float)); // variable to store beamformed high-resolution beamformed image 
//	hipMemset(cudaBackEnd::d_bfHR, 0, cudaBackEnd::pixels * sizeof(float));
//
//	hipMalloc((void**)&cudaBackEnd::dev_beamformed_data1, cudaBackEnd::pixels * sizeof(float));// variable to store reshaped beamformed data
//
//	hipMalloc((void**)&cudaBackEnd::d_bfHRBP, sizeof(float) * cudaBackEnd::pixels);// variable to store beamformed high-resolution bandpass filtered data
//
//	////////////// z value////////////////////
//	float dz = sample_spacing * samples / Nz;  // depth / (Nz - 1) / 1000;   // spacing in axial (z) direction in mm;
//	hipMalloc((void**)&cudaBackEnd::d_z_axis, cudaBackEnd::Nz * sizeof(float));
//	range << <cudaBackEnd::Nz / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_z_axis, 0, cudaBackEnd::Nz, dz);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	//////////////////////////////// x value////////////////////////////////
//	float dx = aper_len / (cudaBackEnd::Nx - 1);
//	// float* d_x_axis = 0;
//	hipMalloc((void**)&cudaBackEnd::d_x_axis, cudaBackEnd::Nx * sizeof(float));    // 167.939 us
//	range << <cudaBackEnd::Nx / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_x_axis, (-cudaBackEnd::aper_len / 2000), cudaBackEnd::Nx, dx / 1000);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	//////////////// Probe geometry, this info can be taken from transducer file ////////////////////
//	//float* d_probe = 0;
//	hipMalloc((void**)&cudaBackEnd::d_probe, cudaBackEnd::N_elements * sizeof(float));
//	range << <1, cudaBackEnd::N_elements >> > (cudaBackEnd::d_probe, (-cudaBackEnd::aper_len / 2000), cudaBackEnd::N_elements, cudaBackEnd::pitch);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	/////////////////rx aerture calculation using Fnumber///////////////////////////////
//	hipMalloc((void**)&cudaBackEnd::d_rx_aperture, cudaBackEnd::Nz * sizeof(float));
//	element_division << <cudaBackEnd::Nz / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_z_axis, cudaBackEnd::rx_f_number, cudaBackEnd::Nz, cudaBackEnd::d_rx_aperture);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	////////////////////////rx aerture distance////////
//	hipMalloc((void**)&cudaBackEnd::d_rx_ap_distance, cudaBackEnd::channels * cudaBackEnd::Nx * sizeof(float));
//	aperture_distance << <cudaBackEnd::Nx * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_x_axis, cudaBackEnd::d_probe, cudaBackEnd::Nx, cudaBackEnd::channels, cudaBackEnd::d_rx_ap_distance);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	///////////////////apodization/////////////////
//	// float* d_rx_apod = 0;
//	hipMalloc((void**)&cudaBackEnd::d_rx_apod, sizeof(float) * cudaBackEnd::Nz * cudaBackEnd::channels * cudaBackEnd::Nx);
//	apodization << <cudaBackEnd::pixels * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_rx_ap_distance, cudaBackEnd::d_rx_aperture, cudaBackEnd::Nz, cudaBackEnd::Nx, cudaBackEnd::channels, cudaBackEnd::pixels, cudaBackEnd::d_rx_apod);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	hipFree(d_rx_aperture);
//	hipFree(d_rx_ap_distance);
//
//	/////////////////// calculate central positions transmit subaperture ////////////////////
//	hipMalloc((void**)&cudaBackEnd::d_cen_pos, cudaBackEnd::num_frames * sizeof(float));
//	Tx_cen_pos << < 1, cudaBackEnd::num_frames >> > (cudaBackEnd::d_cen_pos, cudaBackEnd::N_elements, cudaBackEnd::N_active, cudaBackEnd::pitch, cudaBackEnd::skip_frames, cudaBackEnd::num_frames, cudaBackEnd::d_probe);
//
//	/////////////receive delay calculation /////////////////////////////////////////////
//	hipMalloc((void**)&cudaBackEnd::d_rx_delay, cudaBackEnd::pix_cha * sizeof(float));
//	receive_delay << < cudaBackEnd::pixels * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_probe, cudaBackEnd::d_x_axis, cudaBackEnd::d_z_axis, cudaBackEnd::channels, cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::del_convert, cudaBackEnd::d_rx_delay);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	/////////////////// Transmit delay calculation ////////////////////
//	hipMalloc((void**)&cudaBackEnd::d_tx_delay, cudaBackEnd::pixels * cudaBackEnd::num_frames * sizeof(float));
//	transmit_delay << < cudaBackEnd::pixels * cudaBackEnd::num_frames / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_x_axis, cudaBackEnd::d_z_axis, cudaBackEnd::d_cen_pos, cudaBackEnd::zd, cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::del_convert, cudaBackEnd::num_frames, cudaBackEnd::d_tx_delay);
//	hipGetLastError();
//	hipDeviceSynchronize();
//
//	////////////Free cuda memory (one time use) ///////////////////////////
//	hipFree(cudaBackEnd::d_probe);
//	hipFree(cudaBackEnd::d_x_axis);
//	hipFree(cudaBackEnd::d_z_axis);
//	hipFree(cudaBackEnd::d_cen_pos);
//
//	//char filename3[200];
//	//sprintf(filename3, "b_10M.csv");
//	//read_csv_array(filt_coeff, filename3);    // csv file read
//	////cv::imwrite("okMat3.png", testMat0);
//	//// float* d_filt_coeff = 0;
//	//hipMalloc((void**)&d_filt_coeff, sizeof(float) * MASK_WIDTH);
//	//hipMemcpy(d_filt_coeff, filt_coeff, sizeof(float) * MASK_WIDTH, hipMemcpyHostToDevice);
//	//////////  Intialization &(or) Memory allocation  //////////////////
//	//// float* d_data = 0;   // variable to store raw rf data
//	//hipMalloc((void**)&d_data, sizeof(float) * samples * channels);
//	//// float* d_bfHR = 0;  // variable to store beamformed high-resolution beamformed image 
//	//hipMalloc((void**)&d_bfHR, pixels * sizeof(float));
//	////zeros << <pixels / num_threads + 1, num_threads >> > (d_bfHR, pixels);  
//	//hipMemset(d_bfHR, 0, pixels * sizeof(float));
//	//// float* dev_beamformed_data1 = 0;   // variable to store reshaped beamformed data
//	//hipMalloc((void**)&dev_beamformed_data1, pixels * sizeof(float));
//	//// float* d_bfHRBP = 0;  // variable to store beamformed high-resolution bandpass filtered data
//	//hipMalloc((void**)&d_bfHRBP, sizeof(float) * pixels);
//	//////////////// z value////////////////////
//	//float dz = sample_spacing * samples / Nz;  // depth / (Nz - 1) / 1000;   // spacing in axial (z) direction in mm;
//	//// float* d_z_axis = 0;
//	//hipMalloc((void**)&d_z_axis, Nz * sizeof(float));
//	//range << <Nz / num_threads + 1, num_threads >> > (d_z_axis, 0, Nz, dz);
//	//hipGetLastError();
//	//hipDeviceSynchronize();
//	////////////////////////////////// x value////////////////////////////////
//	//float dx = aper_len / (Nx - 1);
//	//// float* d_x_axis = 0;
//	//hipMalloc((void**)&d_x_axis, Nx * sizeof(float));    // 167.939 us
//	//range << <Nx / num_threads + 1, num_threads >> > (d_x_axis, (-aper_len / 2000), Nx, dx / 1000);
//	//hipGetLastError();
//	//hipDeviceSynchronize();
//	////////////////// Probe geometry, this info can be taken from transducer file ////////////////////
//	////float* d_probe = 0;
//	//hipMalloc((void**)&d_probe, N_elements * sizeof(float));
//	//range << <1, N_elements >> > (d_probe, (-aper_len / 2000), N_elements, pitch);
//	//hipGetLastError();
//	//hipDeviceSynchronize();
//	///////////////////rx aerture calculation using Fnumber///////////////////////////////
//	//// rx_aper=rfsca.z/rf_number
//	//// float* d_rx_aperture = 0;
//	//hipMalloc((void**)&d_rx_aperture, Nz * sizeof(float));
//	//element_division << <Nz / num_threads + 1, num_threads >> > (d_z_axis, rx_f_number, Nz, d_rx_aperture);
//	//hipGetLastError();
//	//hipDeviceSynchronize();
//	//////////////////////////rx aerture distance////////
//	//// float* d_rx_ap_distance = 0;
//	//hipMalloc((void**)&d_rx_ap_distance, channels * Nx * sizeof(float));
//	//aperture_distance << <Nx * channels / num_threads + 1, num_threads >> > (d_x_axis, d_probe, Nx, channels, d_rx_ap_distance);
//	//hipGetLastError();
//	//hipDeviceSynchronize();
//	/////////////////////apodization/////////////////
//	//// float* d_rx_apod = 0;
//	//hipMalloc((void**)&d_rx_apod, sizeof(float) * Nz * channels * Nx);
//	//apodization << <pixels * channels / num_threads + 1, num_threads >> > (d_rx_ap_distance, d_rx_aperture, Nz, Nx, channels, pixels, d_rx_apod);
//	//hipGetLastError();
//	//hipDeviceSynchronize();
//	//hipFree(d_rx_aperture);
//	//hipFree(d_rx_ap_distance);
//	///////////////////// calculate central positions transmit subaperture ////////////////////
//	//// float* d_cen_pos = 0;
//	//hipMalloc((void**)&d_cen_pos, num_frames * sizeof(float));
//	//Tx_cen_pos << < 1, num_frames >> > (d_cen_pos, N_elements, N_active, pitch, skip_frames, num_frames, d_probe);
//	///////////////receive delay calculation /////////////////////////////////////////////
//	//// float* d_rx_delay = 0;
//	//hipMalloc((void**)&d_rx_delay, pix_cha * sizeof(float));
//	//receive_delay << < pixels * channels / num_threads + 1, num_threads >> > (d_probe, d_x_axis, d_z_axis, channels, Nx, Nz, del_convert, d_rx_delay);
//	//hipGetLastError();
//	//hipDeviceSynchronize();
//	///////////////////// Transmit delay calculation ////////////////////
//	//// float* d_tx_delay = 0;
//	//hipMalloc((void**)&d_tx_delay, pixels * num_frames * sizeof(float));
//	////transmit delay for all frames,   
//	//transmit_delay << < pixels * num_frames / num_threads + 1, num_threads >> > (d_x_axis, d_z_axis, d_cen_pos, zd, Nx, Nz, del_convert, num_frames, d_tx_delay);
//	//hipGetLastError();
//	//hipDeviceSynchronize();
//	//////////////Free cuda memory (one time use) ///////////////////////////
//	//hipFree(d_probe);
//	//hipFree(d_x_axis);
//	//hipFree(d_z_axis);
//	//hipFree(d_cen_pos);
//
//	return 0;
//}

//double** cudaBackEnd::computeBModeImgLinDev2(bool debug)
//{
//	// perform b-mode generation here using cuda
//	const int TILE_SIZE = 4;
//	int MASK_WIDTH = 364;
//	const int MAX_LINE = 256;
//
//	////// Computer (NIVIDIA) parametrs
//	//int num_threads = 1024;
//	///// Apodization parameters
//	//float rx_f_number = 2.0;
//	///////// Ultrasound scanner parametrs
//	////float depth = 49.28;      // Depth of imaging in mm
//	//int samples = 2040;         // # of samples in depth direction
//	//int N_elements = 64;        // # of transducer elements
//	//float sampling_frequency = 32.0e6;   // sampling frequency
//	//float c = 1540.0;		 // speed of sound [m/s]	
//	//int N_active = 8;        // Active transmit elmeents
//	//float pitch = 0.3 / 1000;// spacing between the elements
//	//float aper_len = (N_elements - 1) * pitch * 1000;  //aperture foot print 
//	//float zd = pitch * N_active / (float)2;            // virtual src distance from transducer array 
//	//float sample_spacing = c / sampling_frequency / (float)2;
//	//float del_convert = sampling_frequency / c;  // used in delay calculation
//	//int channels = 64;							 // number of A-lines data used for beamforming
//	////// Beamforming "Grid" parameters
//	//int Nx = 256;			// 256 Lateral spacing
//	//int Nz = 1024;			//1024 Axial spacing
//	//int pixels = Nz * Nx;
//	//int pix_cha = pixels * channels;// Nz*Nx*128 This array size is used for Apodization
//	//int num_frames = 57;			// number of low resolution images
//	//int skip_frames = 1;			//
//	
//	cudaBackEnd::env = new float[cudaBackEnd::pixels];
//	// Global variable to store the full image.  Cannot be declared local as memory alloc may fail due to large size.
//	cudaBackEnd::rximg2 = new float[cudaBackEnd::N_elements * cudaBackEnd::samples];
//	zeroC(cudaBackEnd::rximg2, cudaBackEnd::samples * cudaBackEnd::N_elements);   // set rx_img array values to zero.
//	//--------------------------------------------------
//	
//	int ok = cudaBackEnd::initSettingFile("out25.txt");
//	
//	//---------------------------------------------------
//
//	static double pix2 = 0.0;
//	unsigned char buf[16 * 1024];
//	int row = 0;  // Keep track of how many rows have been added
//	char line[MAX_LINE]; // Max possible line length?
//	int iteration = 0;
//	int errcount = 0;
//	unsigned int addr, data;
//	unsigned char recvbuf[2048 * 64 * 2];
//	const int MAXROWS = 2040;
//	LONG rxlen = MAXROWS * 64 * 2;
//	//-----------------------------------------------------
//
//	// unsigned int start = clock();
//	while (fgets(line, MAX_LINE, cudaBackEnd::fp)) {
//		line[strcspn(line, "\n")] = 0; // Trim trailing newline
//		if ((strlen(line) == 0) || (line[0] == ' ') || (line[0] == '#')) {
//			//printf("Skipping [%s]\n", line);
//		}
//		else if (line[0] == 'O') {
//			sscanf_s(line, "O %04X %08X ", &addr, &data);
//			//printf("Write %08X to Obelix %04X\n", data, addr);
//			row = insert_row(buf, row, addr, data);
//		}
//		else if (line[0] == 'T') {
//			sscanf_s(line, "T %04X %08X ", &addr, &data);
//			row = insert_row(buf, row, 0x6, 0x40000000 | addr);
//			row = insert_row(buf, row, 0x7, data);
//			row = insert_row(buf, row, 0x6, 0xC0000000 | addr);
//			//printf("Write %08X to TX %04X\n", data, addr);
//		}
//		else if (line[0] == 'A') {
//			sscanf_s(line, "A %04X %08X ", &addr, &data);
//			row = insert_row(buf, row, 0x6, 0x00000000 | addr);
//			row = insert_row(buf, row, 0x7, data);
//			row = insert_row(buf, row, 0x6, 0x80000000 | addr);
//			//printf("Write %08X to AFE %04X\n", data, addr);
//		}
//		else if (line[0] == 'C') {  // CAPTURE
//			//wait(100);
//			row = insert_row(buf, row, 0x4, 0x01);
//			//write_rows(ept, buf, row);  // Send commands
//			//wait(100);
//			row = insert_row(buf, row, 0x4, 0x10);
//			//write_rows(ept, buf, row);  // Send commands
//			//wait(100);
//			row = insert_row(buf, row, 0x4, 0x00);
//			cudaBackEnd::ept_in->Abort();
//			cudaBackEnd::ept_in->Reset();
//			write_rows(cudaBackEnd::ept, buf, row);  // Send commands
//			wait(1);
//			if (read_chunk(cudaBackEnd::ept_in, recvbuf, rxlen)) {
//				short* rxdata = (short*)(recvbuf);
//				for (int i = 0; i < rxlen / 2; i++) {
//					if (rxdata[i] >= 512) rxdata[i] -= 1024;
//				}
//				// Trying to read only first N-1 rows and discard 1st sample
//				for (int i = 0; i < 64; i++) {
//					for (int j = 0; j < MAXROWS - 1; j++) {
//						//rximg[iteration][i][j] = rxdata[j*64+i+2];
//						cudaBackEnd::rximg2[i * MAXROWS + j] = rxdata[j * 64 + i + 2];
//					}
//				}
//				//saveToFile(iteration, rxlen, recvbuf);
//			}
//			else {
//				errcount++;
//			}
//			hipMemcpy(cudaBackEnd::d_data, cudaBackEnd::rximg2, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels, hipMemcpyHostToDevice);
//			beamformingLR3 << <(cudaBackEnd::pixels / 256) * cudaBackEnd::channels, 256 >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_tx_delay, cudaBackEnd::d_rx_delay, cudaBackEnd::d_data, cudaBackEnd::d_rx_apod, cudaBackEnd::samples, cudaBackEnd::pixels, iteration, cudaBackEnd::num_frames, cudaBackEnd::channels);
//			hipGetLastError();
//			hipDeviceSynchronize();
//			iteration++; // Increment iteration after saving to image
//			row = 0;   // Reset buffer for next iteration
//		}
//		else {
//			printf("Don't know how to handle [%s] yet.\n", line);
//		}
//	}
//
//	//// check for nan values,
//	isnan_test_array << <cudaBackEnd::pixels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_bfHR, cudaBackEnd::pixels);
//	hipGetLastError();
//	hipDeviceSynchronize();
//	//////////// Bandpass filtering using shared memory /////////////////////
//	BPfilter1SharedMem << <(cudaBackEnd::pixels + TILE_SIZE - 1) / TILE_SIZE, TILE_SIZE >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_filt_coeff, cudaBackEnd::pixels, cudaBackEnd::d_bfHRBP);
//	hipGetLastError();
//	hipDeviceSynchronize();
//	//////////////// reshape of the beamformed data ///////////////
//	reshape_columnwise << <cudaBackEnd::pixels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::d_bfHRBP);
//	hipGetLastError();
//	hipDeviceSynchronize();
//	hipMemcpy(cudaBackEnd::env, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::Nz * cudaBackEnd::Nx * sizeof(float), hipMemcpyDeviceToHost);
//	//char fileout[200];
//	//sprintf(fileout, "sample_output\\b_mode_%d.csv", 1); //all the 16 inputs are arranged in a single file
//	//csv_write_mat(env, fileout, Nz, Nx);
//	double** outArray = convertsingto2darray(cudaBackEnd::env, cudaBackEnd::Nz, cudaBackEnd::Nx);
//
//	//// For next iteration
//	hipMemset(cudaBackEnd::d_bfHR, 0, pixels * sizeof(float));
//
//	return outArray;
//}