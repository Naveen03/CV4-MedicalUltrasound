#include "hip/hip_runtime.h"

// It reconstructs LR images separately and then add them to get HR image.
// Delay values and apodization are reshaped into [1 1024, 1 1024, ... pixels] ...
#include <stdlib.h>
#include <stdio.h>
#include <iomanip>
#include <ctime>
#include <math.h>
#include <string.h>
#include <time.h>
#include <exception>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>      /// From "cufft.lib" 
#include "hip/hip_runtime.h"
#include <windows.h>

#include "cudaHeader.cuh"
#include <stdlib.h>
#include <stdio.h>
#include <iomanip>
#include <ctime>
#include <math.h>
#include <string.h>
#include <time.h>
#include <fstream>

// include OpenCV Header
#include <opencv2/opencv.hpp>
// #include <opencv2/highgui.hpp>
// #include "mat_operations.h"
 #include "CyAPI.h"
 #include "CyUSB30_def.h"

namespace imageComputeCudaWrap {

	////////////////////////////////
	// Parameters for Linear Prob///
	////////////////////////////////

	// perform b-mode generation here using cuda
	const int TILE_SIZE = 4;
	int MASK_WIDTH = 364;
	const int MAX_LINE = 256;
	//// Computer (NIVIDIA) parametrs
	int num_threads = 1024;
	/// Apodization parameters
	float rx_f_number = 2.0;
	/////// Ultrasound scanner parametrs
	//float depth = 49.28;      // Depth of imaging in mm
	int samples = 2040;         // # of samples in depth direction
	int N_elements = 64;        // # of transducer elements
	float sampling_frequency = 32e6;   // sampling frequency
	float c = 1540.0;		 // speed of sound [m/s]	
	int N_active = 8;        // Active transmit elmeents
	float pitch = 0.3 / 1000;// spacing between the elements
	float aper_len = (N_elements - 1) * pitch * 1000;  //aperture foot print 
	float zd = pitch * N_active / (float)2;            // virtual src distance from transducer array 
	float sample_spacing = c / sampling_frequency / (float)2;
	float del_convert = sampling_frequency / c;  // used in delay calculation
	int channels = 64;							 // number of A-lines data used for beamforming
	//// Beamforming "Grid" parameters
	int Nx = 256;			// 256 Lateral spacing
	int Nz = 1024;			//1024 Axial spacing
	int pixels = Nz * Nx;
	int pix_cha = pixels * channels;// Nz*Nx*128 This array size is used for Apodization
	int num_frames = 57;			// number of low resolution images
	int skip_frames = 1;			//
	// Global variable to store the full image.  Cannot be declared local as memory alloc may fail due to large size.
	float rximg[64 * 2040];
	// Device and Host memmoey used in initializer
	float* filt_coeff = new float[MASK_WIDTH];
	float* d_z_axis = 0;
	float* d_x_axis = 0;
	float* d_probe = 0;
	float* d_rx_aperture = 0;
	float* d_rx_ap_distance = 0;
	float* d_cen_pos = 0;
	float* d_data = 0;					// variable to store raw rf data
	float* d_bfHR = 0;					// variable to store beamformed high-resolution beamformed image 
	float* d_tx_delay = 0;
	float* d_rx_delay = 0;				// delay calculation
	float* d_rx_apod = 0;				//apodization
	float* d_filt_coeff = 0;			//to read filter coeff CSV
	float* d_bfHRBP = 0;				// variable to store beamformed high-resolution bandpass filtered data
	float* dev_beamformed_data1 = 0;	// variable to store reshaped beamformed data
	float* env = new float[pixels];		// Host memory variable to store beamformed high-resolution bandpass filtered data		// Host memory variable to store beamformed high-resolution bandpass filtered data
	// for curveLiner Prob
	float* d_theta = 0;
	float* d_theta1 = 0;
	float* d_theta_tx = 0;
	// H/W initilization
	CCyUSBDevice* USBDevice;
	CCyControlEndPoint* ept;

	////////////////////////////////////
	/// Parameters for CurvLinear Prob//
	////////////////////////////////////

	////const int MAX_ITER = 128;
	////const int N_RX = 64;
	////const int MAX_LINE = 256;
	////float PI = 3.14;
	//const int MASK_WIDTH = 364;
	////const int TILE_SIZE = 4;
	//////// Computer (NIVIDIA) parametrs
	////int num_threads = 1024;
	/////// Apodization parameters
	////float rx_f_number = 2.0;
	/////////// Ultrasound scanner parametrs
	////int samples = 2040;						// # of samples in depth direction
	////int N_elements = 128;					// # of transducer elements
	////float sampling_frequency = 32e6;		// sampling frequency
	////float c = 1540.0;						// speed of sound [m/s]	
	////int N_active = 8;                       // Active transmit elmeents
	////float pitch = 0.465 / 1000;				// spacing between the elements
	////float aper_len = (N_elements - 1) * pitch * 1000;		//aperture foot print 
	////float zd = pitch * N_active / (float)2;					// virtual src distance from transducer array 
	////float sample_spacing = c / sampling_frequency / (float)2;
	////float del_convert = sampling_frequency / c;				// used in delay calculation
	////float rc = 60.1 / 1000;									// radius_of_curvature
	////float scan_angle = (58 * PI) / 180;
	////int channels = 128;										// number of A-lines data used for beamforming
	//////// Beamforming "Grid" parameters
	////int Nx = 256;							// 256 Lateral spacing
	////int Nz = 1024;							//1024 Axial spacing
	////int pixels = Nz * Nx;
	////int pix_cha = pixels * channels;		// Nz*Nx*128 This array size is used for Apodization
	////int frames = 121;
	////int num_frames = 121;					// number of low resolution images
	////int skip_frames = 1;  
	////// Post processing parameters.
	////int dBvalue = 60;
	//// Global variable to store the full image.  Cannot be declared local as memory alloc may fail due to large size.
	//float rximg[128 * 2040];
	//// parameters for matrix processing
	//int croppedBot = 300;
	//// cv::Mat outMat, outMatCrp, envolepMat, logcMat;



	////////////////////////////////////
	/// Parameters to read from CSV   //
	////////////////////////////////////

	//// perform b-mode generation here using cuda
	//const int TILE_SIZE = 4;
	//int MASK_WIDTH = 364;
	//const int MAX_LINE = 256;
	////// Computer (NIVIDIA) parametrs
	//int num_threads = 1024;
	///// Apodization parameters
	//float rx_f_number = 2.0;
	///////// Ultrasound scanner parametrs
	////float depth = 49.28;      // Depth of imaging in mm
	//int samples = 2040;         // # of samples in depth direction
	//int N_elements = 64;        // # of transducer elements
	//float sampling_frequency = 32e6;   // sampling frequency
	//float c = 1540.0;		 // speed of sound [m/s]	
	//int N_active = 8;        // Active transmit elmeents
	//float pitch = 0.3 / 1000;// spacing between the elements
	//float aper_len = (N_elements - 1) * pitch * 1000;  //aperture foot print 
	//float zd = pitch * N_active / (float)2;            // virtual src distance from transducer array 
	//float sample_spacing = c / sampling_frequency / (float)2;
	//float del_convert = sampling_frequency / c;  // used in delay calculation
	//int channels = 64;							 // number of A-lines data used for beamforming
	////// Beamforming "Grid" parameters
	//int Nx = 256;			// 256 Lateral spacing
	//int Nz = 1024;			//1024 Axial spacing
	//int pixels = Nz * Nx;
	//int pix_cha = pixels * channels;// Nz*Nx*128 This array size is used for Apodization
	//int num_frames = 57;			// number of low resolution images
	//int skip_frames = 1;			//
	//// Global variable to store the full image.  Cannot be declared local as memory alloc may fail due to large size.
	//float rximg[64 * 2040];
	//float* filt_coeff = new float[MASK_WIDTH];
	//float* d_z_axis = 0;
	//float* d_x_axis = 0;
	//float* d_probe = 0;
	//float* d_rx_aperture = 0;
	//float* d_rx_ap_distance = 0;
	//float* d_cen_pos = 0;
	//float* d_data = 0;   // variable to store raw rf data
	//float* d_bfHR = 0;  // variable to store beamformed high-resolution beamformed image 
	//float* d_tx_delay = 0;
	//float* d_rx_delay = 0;// delay calculation
	//float* d_rx_apod = 0; //apodization
	//float* d_filt_coeff = 0; //to read filter coeff CSV
	//float* d_bfHRBP = 0;  // variable to store beamformed high-resolution bandpass filtered data
	//float* dev_beamformed_data1 = 0;   // variable to store reshaped beamformed data
	//float* env = new float[pixels]; // Host memory variable to store beamformed high-resolution bandpass filtered data



	void wait(unsigned timeout)
	{
		timeout += std::clock();
		while (std::clock() < timeout) continue;
	}

	void write_rows(CCyControlEndPoint* ept, unsigned char* ptr, unsigned int numRows)
	{
		int len = numRows * 16; // each row is 16 bytes to send
		int sent = 0;
		unsigned char* tPtr = ptr;
		while (sent < len) {
			LONG buflen = len - sent;
			if (buflen > 192) buflen = 192;
			//for (int i = 0; i < buflen; i += 16) {
			//	for (int j = 0; j < 16; j++) {
			//		printf("%02X\t ", ptr[i + j]);
			//	}
			//	printf("\n");
			//}
			ept->XferData(tPtr, buflen);
			tPtr += buflen;
			sent += buflen;
		}
		//printf("Sent %d bytes to EPT\n", sent);
	}

	bool read_chunk(CCyBulkEndPoint* ept_in, unsigned char* recvBuf, LONG& length)
	{
		bool result;
		LONG intlen = length;
		result = ept_in->XferData(recvBuf, intlen, NULL, true);
		//if (result) {
		//	printf("Received data ------------- : %d\n", intlen);
		//}
		//else {
		//	printf("***   ERROR receiving data - expected %d, got %d\n", length, intlen);
		//}
		ept_in->Abort();
		ept_in->Reset();
		return result;
	}

	int insert_row(unsigned char* buf, int row, short addr, int data)
	{
		int o = row * 16;
		buf[o] = 0xff; buf[o + 1] = 0xaa; buf[o + 2] = 0x01; buf[o + 3] = 0x07;
		buf[o + 4] = 0x00; buf[o + 5] = 0x00; buf[o + 6] = 0x00; buf[o + 7] = 0x01;
		buf[o + 8] = (addr & 0xff); buf[o + 9] = (addr & 0xff00) >> 8;
		buf[o + 10] = (data & 0xff); buf[o + 11] = (data & 0xff00) >> 8;
		buf[o + 12] = (data & 0xff0000) >> 16; buf[o + 13] = (data & 0xff000000) >> 24;
		buf[o + 14] = 0x00; buf[o + 15] = 0x00;
		return row + 1;
	}

	__global__ void range(int* out_data, int min, int arr_size, int inc)	//creates an array of a range of values
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																					//max=final value of the array
		if (i < arr_size)																//arr_size==array size
		{																			//inc=increment needed
			out_data[i] = min + (i * inc);
		}
	}

	__global__ void range(float* out_data, float min, int arr_size, float inc)	//creates an array of a range of values
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																					//max=final value of the array
		if (i < arr_size)																//arr_size==array size
		{																			//inc=increment needed
			out_data[i] = min + (i * inc);
		}
	}

	__global__ void range(double* out_data, double min, int arr_size, double inc)	//creates an array of a range of values
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																					//max=final value of the array
		if (i < arr_size)																//arr_size==array size
		{																			//inc=increment needed
			out_data[i] = min + (i * inc);
		}
	}

	__global__ void range(long double* out_data, long double min, int arr_size, long double inc)	//creates an array of a range of values
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																					//max=final value of the array
		if (i < arr_size)																//arr_size==array size
		{																			//inc=increment needed
			out_data[i] = min + (i * inc);
		}
	}

	__global__ void element_division(float* mat_in, float value, int size, float* mat_out)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																					   //max=final value of the array
		if (i < size)
		{
			mat_out[i] = mat_in[i] / value;
		}
	}

	__global__ void element_division(long double* mat_in, float value, int size, long double* mat_out)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																					   //max=final value of the array
		if (i < size)
		{
			mat_out[i] = mat_in[i] / value;
		}
	}

	__global__ void aperture_distance(float* mat1, float* mat2, int Nx, int channels, float* mat_out)
	{
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		//int y = blockDim.y * blockIdx.y + threadIdx.y;
		int i = x / channels;
		int j = x % channels;

		if (x < Nx * channels)
		{
			mat_out[i * channels + j] = fabs(mat1[i] - mat2[j]);
		}
	}

	__global__ void apodization(float* distance, float* aperture, int Nz, int Nx, int channels, int pixels, float* apod)
	{
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		int i = x / Nz;
		int ii = i % Nx;
		int j = x % Nz;
		int nrx = x / pixels;
		float PI = 3.14159;

		if (x < pixels * channels)
		{
			bool temp = distance[ii * channels + nrx] <= (aperture[j] / 2);
			apod[i * Nz + j] = temp * (0.5 + 0.5 * cos(2 * PI * distance[ii * channels + nrx] / aperture[j]));
		}
	}

	// This function calculates TX central aperture position
	__global__ void Tx_cen_pos(float* cen_pos, int N_elements, int N_active, float pitch, int skip_frames, int num_frames, float* probe)
	{

		int x = threadIdx.x;

		if (x < num_frames)
		{
			//cen_pos[x] = pitch * ((N_active / 2) + (N_active * (x)-N_elements / 2));
			cen_pos[x] = probe[x * skip_frames + 4];
		}
	}

	// receive_delay calculation
	__global__ void receive_delay(float* probe_ge_x, float* x_axis1, float* z_axis1, int channels, int Nx, int Nz, float del_convert, float* rx_delay)
	{
		unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;

		if (x < Nx * Nz * channels)
		{
			int i = x / Nz;
			int ii = i % Nx;
			int j = x % Nz;
			int nrx = x / (Nx * Nz);
			rx_delay[i * Nz + j] = (sqrt((probe_ge_x[nrx] - x_axis1[ii]) * (probe_ge_x[nrx] - x_axis1[ii]) + ((z_axis1[j]) * (z_axis1[j])))) * del_convert;
			// 1867 - 210 = 1657
			//rx_delay[i * Nx + j] = sqrt(rc * rc + (rc + z_axis[ii]) * (rc + z_axis[ii]) - 2 * rc * (rc + z_axis[ii]) * cos(theta[nrx] - theta1[j])) * del_convert;
		}
	}

	//  transmit_delay calculation
	__global__ void transmit_delay(float* x_axis1, float* z_axis1, float* k1, float zd, int Nx, int Nz, float del_convert, int num_frames, float* tx_delay)
	{
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		int i = x / Nz;
		int ii = i % Nx;
		int j = x % Nz;
		int f = x / (Nx * Nz);

		if (x < Nx * Nz * num_frames)
		{
			tx_delay[i * Nz + j] = (sqrt(((k1[f] - x_axis1[ii]) * (k1[f] - x_axis1[ii])) + ((zd + z_axis1[j]) * (zd + z_axis1[j])))) * del_convert;
			// 1875-210 = 1665
			//tx_delay[i * Nx + j] = (zd + sqrt(rc * rc + (rc + z_axis[ii]) * (rc + z_axis[ii]) - 2 * rc * (rc + z_axis[ii]) * cos(theta_tx[f] - theta1[j]))) * del_convert;
			//first 256*1024 for frame 1, next 256*1024 for frame 2........
		}
	}

	void read_csv_mat(float* data, char* filename, int col1)
	{
		char buffer[6240];  //6240
		char* token;

		int i = 0, j = 0;
		FILE* file;
		file = fopen(filename, "r");
		if (file == NULL)
		{
			// printf("Can't open the file");
		}
		else
		{
			while (fgets(buffer, sizeof(buffer), file) != 0)            // end-of-file indicator
			{
				token = strtok(buffer, ",");
				j = 0;
				while (token != NULL)
				{
					data[i * col1 + j] = atof(token);     //converts the string argument str to float
					token = strtok(NULL, ",");
					j++;
				}

				i++;
			}
			fclose(file);
			// printf("Complete reading from file %s\n", filename);
		}
	}

	void read_csv_mat(long double* data, char* filename, int col1)
	{
		char buffer[6240];  //6240
		char* token;

		int i = 0, j = 0;
		FILE* file;
		file = fopen(filename, "r");
		if (file == NULL)
		{
			// printf("Can't open the file");
		}
		else
		{
			while (fgets(buffer, sizeof(buffer), file) != 0)            // end-of-file indicator
			{
				token = strtok(buffer, ",");
				j = 0;
				while (token != NULL)
				{
					data[i * col1 + j] = atof(token);     //converts the string argument str to float
					token = strtok(NULL, ",");
					j++;
				}

				i++;
			}
			fclose(file);
			// printf("Complete reading from file %s\n", filename);
		}
	}

	void read_csv_array(float* data, char* filename)
	{
		char buffer[6240];  //6240
		char* token;
		int i = 0;
		FILE* file;

		file = fopen(filename, "r");
		if (file == NULL)
		{
			throw std::exception("File did not open");
		}

		while (fgets(buffer, sizeof(buffer), file) != 0)            // end-of-file indicator
		{
			token = strtok(buffer, ",");
			//j = 0;
			while (token != NULL)
			{
				data[i] = atof(token);     //converts the string argument str to float
				token = strtok(NULL, ",");
				//j++;
			}

			i++;
		}
		fclose(file);
		// printf("Complete reading from file %s\n", filename);

	}

	__global__ void beamformingLR3(float* beamformed_data1, float* tx_delay, float* rx_delay, float* data, float* rx_apod, int samples, int pixels, int f, int num_frames, int channels)
	{
		unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
		int nrx = x / pixels;   // nrx - nth A-line
		int pix = x & (pixels - 1); // x% pixels;     // pixel location

		int pixel_pos = round((float)tx_delay[f * pixels + pix] + (float)rx_delay[x]);   // delay value estimation from tx and rx delay values

		if (pixel_pos < samples)
		{
			beamformed_data1[pix] += rx_apod[x] * data[(nrx * samples + pixel_pos - 1)];   // Extract data based on the delay values and multiplying with apodization value
		}
	}

	__global__ void isnan_test_array(float* data, int size)
	{
		int idx = threadIdx.x + blockDim.x * blockIdx.x;
		if (idx < size)
		{
			if (isnan(data[idx]) == 1)
			{
				data[idx] = 0;
			}
			else
			{
				data[idx] = data[idx];
			}

		}


	}

	__global__ void BPfilter1SharedMem(float* in, float* filt_coeff, int pixels, float* y1) {

		const int TILE_SIZE = 4;
		int MASK_WIDTH = 364;

		int x = blockIdx.x * blockDim.x + threadIdx.x;
		__shared__ float N_s[TILE_SIZE];
		N_s[threadIdx.x] = in[x];
		__syncthreads();

		int PtileStartPt = blockIdx.x * blockDim.x;
		int NtileStartPt = (blockIdx.x + 1) * blockDim.x;
		int n_start_pt = x - (MASK_WIDTH / 2);

		float temp = 0;

		for (int j = 0; j < MASK_WIDTH; j++) {
			int N_index = n_start_pt + j;

			if (N_index >= 0 && N_index < pixels) {
				if ((N_index >= PtileStartPt) && (N_index < NtileStartPt)) {
					temp += N_s[threadIdx.x + j - (MASK_WIDTH / 2)] * filt_coeff[j];
				}
				else {
					temp += in[N_index] * filt_coeff[j];
				}
			}
		}
		y1[x] = temp;
	}

	__global__ void reshape_columnwise(int col, int row, float* beamformed_data_reshaped, float* d_bfHR)
	{
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		//int y = blockDim.y * blockIdx.y + threadIdx.y;
		int i = x / row;
		int j = x % row;

		if (x < col * row)
		{
			beamformed_data_reshaped[j * col + i] = d_bfHR[x];

		}
	}

	/// <Curvilinear Prob>

	// receive_delay calculation
	__global__ void receive_delay(float* theta, float* theta1, float rc, float* z_axis, int channels, int Nx, int Nz, float del_convert, float* rx_delay)
	{
		int x = blockDim.x * blockIdx.x + threadIdx.x;

		if (x < Nz * Nx * channels)
		{
			int i = x / Nz;
			int ii = i % Nx;
			int j = x % Nz;
			int nrx = x / (Nx * Nz);
			rx_delay[i * Nz + j] = sqrt(rc * rc + (rc + z_axis[j]) * (rc + z_axis[j]) - 2 * rc * (rc + z_axis[j]) * cos(theta[nrx] - theta1[ii])) * del_convert;
		}
	}

	__global__ void theta1(float* theta_active, float* theta, int frames, int N_active, int skip_frames)
	{

		int x = threadIdx.x;
		int f = 0;
		for (int i = 1; i <= frames; i += skip_frames)
		{
			theta_active[f] = theta[i + 3 - 1];
			f++;
		}
	}

	//  transmit_delay calculation
	__global__ void transmit_delay(float* theta1, float* z_axis, float rc, float* theta_tx, int Nx, int Nz, float del_convert, int columns, float zd, float* tx_delay)
	{
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		int i = x / Nz;
		int j = x % Nz;
		int f = x / (Nx * Nz);


		if (x < Nx * Nz * columns)
		{
			tx_delay[i * Nz + j] = (zd + sqrt(rc * rc + (rc + z_axis[j]) * (rc + z_axis[j]) - 2 * rc * (rc + z_axis[j]) * cos(theta_tx[f] - theta1[i % Nx]))) * del_convert;
		}
	}

	__global__ void add_ele(float* data, int pixels, float* out_data)
	{
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		//int y = blockDim.y * blockIdx.y + threadIdx.y;
		if (x < pixels)
		{
			out_data[x] += data[x];
		}
	}

	__global__ void sample1(float* tx_delay, float* rx_delay, int pixels, int channels, float c, float sampling_frequency, float* data1, float* rx_apod, float* data, int samples, int columns)
	{
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		int f = blockDim.y * blockIdx.y + threadIdx.y;
		int nrx = x / pixels; //channels
		int pix = x % pixels; //pixels


		if (f < columns)
		{
			float delay = ((float)tx_delay[f * pixels + pix] + (float)rx_delay[(nrx % channels) * pixels + pix]) / c;
			float p = delay * sampling_frequency;
			int pixel_pos = round(p);

			//data1 = rx_apod[(nrx % channels) + (pix * channels)] * data[((nrx % channels) * samples + pixel_pos - 1) * columns + f];
		}
	}

	__global__ void add_columns_matrix(float* data, int columns, int pixels, float* out_data)
	{
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		//int y = blockDim.y * blockIdx.y + threadIdx.y;
		if (x < pixels)
		{
			for (int f = 0; f < columns; f++)
				out_data[x] += data[x * columns + f];
		}
	}

	__global__ void parallel_try(float* tx_delay, float* rx_delay, float sampling_frequency, float c, int samples,
		int channels, int columns, float* rx_apod, int pixels, float* data, float* beamformed_data)
	{
		//__shared__ double* beamformed_data_1;
		int pix = blockDim.x * blockIdx.x + threadIdx.x;
		int nrx = blockDim.y * blockIdx.y + threadIdx.y;
		//int f = x / pixels; int pix = x % pixels; //int nrx = x % 128;
		int f = blockDim.z * blockIdx.z + threadIdx.z;
		//int nrx = x / pixels; //channels
		//int pix = x % pixels; //pixels
		//int f = x / (pixels * channels);

		if (f < columns && pix < pixels && nrx < channels)
		{
			//for (int nrx = 0; nrx < channels; nrx++)
			//{

			float delay = ((float)tx_delay[f * pixels + pix] + (float)rx_delay[(nrx)*pixels + pix]) / c;
			float p = delay * sampling_frequency;
			int pixel_pos = round(p);

			if ((0 < pixel_pos) && (pixel_pos < samples))
			{
				//double ans= beamformed_data[pix] + rx_apod[channels * nrx + pix] * data[nrx * 2600 + pixel_pos];
				beamformed_data[pix * columns + f] += rx_apod[nrx + (pix * channels)] * data[(nrx * samples + pixel_pos - 1) * columns + f];

			}
			//}
		}
	}

	void zeroC(float* bfHR, int pixels)
	{
		for (int j = 0; j < pixels; j++)
		{
			bfHR[j] = 0;
		}
	}

	/// </Curvilinear Prob>

	void csv_write_mat(long double* a, const char* filename, int row1, int col1)		//writes data to memory
	{
		FILE* fp;
		int i;

		fp = fopen(filename, "w+");

		for (i = 0; i < row1; ++i)
		{
			for (int j = 0; j < col1; j++)
			{
				if (j == col1 - 1)					//for the last value in the column "," is not appended
				{									//matrix dimension error can occur with the presence of extra comma at last of the column
					fprintf(fp, "%g", a[i * col1 + j]);
				}
				else
					fprintf(fp, "%g,", a[i * col1 + j]);
			}



			fprintf(fp, "\n");
		}


		fclose(fp);
		// printf("\n %s file is created\n", filename);
	}

	void csv_write_mat(double* a, const char* filename, int row1, int col1)		//writes data to memory
	{
		FILE* fp;
		int i;

		fp = fopen(filename, "w+");

		for (i = 0; i < row1; ++i)
		{
			for (int j = 0; j < col1; j++)
			{
				if (j == col1 - 1)					//for the last value in the column "," is not appended
				{									//matrix dimension error can occur with the presence of extra comma at last of the column
					fprintf(fp, "%g", a[i * col1 + j]);
				}
				else
					fprintf(fp, "%g,", a[i * col1 + j]);
			}



			fprintf(fp, "\n");
		}


		fclose(fp);
		printf("\n %s file is created\n", filename);
	}

	void csv_write_mat(float* a, const char* filename, int row1, int col1)	//for writing integer data "FUNCTION OVERLOADING"
	{
		FILE* fp;
		int i;

		fp = fopen(filename, "w+");

		for (i = 0; i < row1; ++i)
		{
			for (int j = 0; j < col1; j++)
			{
				if (j == col1 - 1)
				{
					fprintf(fp, "%f", a[i * col1 + j]);
				}
				else

					fprintf(fp, "%f,", a[i * col1 + j]);
			}



			fprintf(fp, "\n");
		}


		fclose(fp);
		// printf("\n %s file is created\n", filename);
	}

	//double** ConvertMatto2DArray(cv::Mat img)
	//{
	//	double** array2D = (double**)malloc(img.rows * sizeof(double*));
	//	for (int i = 0; i < img.rows; i++) {
	//		array2D[i] = (double*)malloc(img.cols * sizeof(double));
	//	}
	//	// Fill the values
	//	for (int i = 0; i < img.rows; i++) {
	//		for (int j = 0; j < img.cols; j++) {
	//			array2D[i][j] = img.at<double>(i, j);
	//		}
	//	}
	//	return array2D;
	//}

	double** convertsingto2darray(float* imgArray, int rows, int cols) {

		double** array2D = (double**)malloc(rows * sizeof(double*));
		for (int i = 0; i < rows; i++) {
			array2D[i] = (double*)malloc(cols * sizeof(double));
		}

		for (int i = 0; i < rows; i++) {
			for (int j = 0; j < cols; j++) {
				array2D[i][j] = (double)imgArray[i * cols + j];
			}
		}

		return array2D;
	}

	// Function to initialize the CUDA memmory if read from CSV
	extern std::string cuMemInit() {

		const int MAX_ITER = 128;
		const int N_RX = 64;
		const int MAX_LINE = 256;
		float PI = 3.14;
		const int MASK_WIDTH = 364;
		const int TILE_SIZE = 4;
		int num_threads = 1024;
		float rx_f_number = 2.0;
		int samples = 2040;
		int N_elements = 128;
		float sampling_frequency = 32.0e6;
		float c = 1540.0;
		int N_active = 8;
		int channels = 128;
		int	Nx = 256;
		int Nz = 1024;
		int frames = 121;
		//int num_frames = 121;
		int skip_frames = 1;
		int	dBvalue = 60;
		float pitch = 0.000465;
		float aper_len = 59.055;
		float zd = 0.00186;
		float sample_spacing = 2.40625e-05;
		float del_convert = 20779.2;
		float rc = 0.0601;
		float scan_angle = 1.01178;
		int pixels = 262144;
		int pix_cha = 33554432;

		std::ofstream mFile;
		mFile.open("sample_output/test_meminit.txt");
		std::string out_string = "OK";

			int num_frames = 57;

		try {
			char filename1[200];
			sprintf(filename1, "b_10M.csv");
			filt_coeff = new float[MASK_WIDTH];
			read_csv_array(filt_coeff, filename1);    // csv file read
		}

		catch (std::exception& err) {
			return err.what();
		}

		hipMalloc((void**)&d_filt_coeff, sizeof(float) * MASK_WIDTH);
		hipMemcpy(d_filt_coeff, filt_coeff, sizeof(float) * MASK_WIDTH, hipMemcpyHostToDevice);

		////////////// z value////////////////////
		float dz = sample_spacing * samples / Nz;  // depth / (Nz - 1) / 1000;   // spacing in axial (z) direction in mm;
		hipMalloc((void**)&d_z_axis, Nz * sizeof(float));
		range << <Nz / num_threads + 1, num_threads >> > (d_z_axis, 0, Nz, dz);
		hipGetLastError();
		hipDeviceSynchronize();

		//////////////////////////////// x value////////////////////////////////
		float dx = aper_len / (Nx - 1);
		hipMalloc((void**)&d_x_axis, Nx * sizeof(float));    // 167.939 us
		range << <Nx / num_threads + 1, num_threads >> > (d_x_axis, (-aper_len / 2000), Nx, dx / 1000);
		hipGetLastError();
		hipDeviceSynchronize();

		//////////////// Probe geometry, this info can be taken from transducer file ////////////////////
		hipMalloc((void**)&d_probe, N_elements * sizeof(float));
		range << <1, N_elements >> > (d_probe, (-aper_len / 2000), N_elements, pitch);
		hipGetLastError();
		hipDeviceSynchronize();

		/////////////////rx aperture calculation using Fnumber///////////////////////////////
		// rx_aper=rfsca.z/rf_number
		hipMalloc((void**)&d_rx_aperture, Nz * sizeof(float));
		element_division << <Nz / num_threads + 1, num_threads >> > (d_z_axis, rx_f_number, Nz, d_rx_aperture);
		hipGetLastError();
		hipDeviceSynchronize();

		////////////////////////rx aerture distance////////
		hipMalloc((void**)&d_rx_ap_distance, channels * Nx * sizeof(float));
		aperture_distance << <Nx * channels / num_threads + 1, num_threads >> > (d_x_axis, d_probe, Nx, channels, d_rx_ap_distance);
		hipGetLastError();
		hipDeviceSynchronize();

		///////////////////apodization/////////////////
		hipMalloc((void**)&d_rx_apod, sizeof(float) * Nz * channels * Nx);
		apodization << <pixels * channels / num_threads + 1, num_threads >> > (d_rx_ap_distance, d_rx_aperture, Nz, Nx, channels, pixels, d_rx_apod);
		hipGetLastError();
		hipDeviceSynchronize();
		hipFree(d_rx_aperture);
		hipFree(d_rx_ap_distance);

		/////////////////// calculate central positions transmit subaperture ////////////////////
		hipMalloc((void**)&d_cen_pos, num_frames * sizeof(float));
		Tx_cen_pos << < 1, num_frames >> > (d_cen_pos, N_elements, N_active, pitch, skip_frames, num_frames, d_probe);
		/////////////receive delay calculation /////////////////////////////////////////////
		hipMalloc((void**)&d_rx_delay, pix_cha * sizeof(float));
		receive_delay << < pixels * channels / num_threads + 1, num_threads >> > (d_probe, d_x_axis, d_z_axis, channels, Nx, Nz, del_convert, d_rx_delay);
		hipGetLastError();
		hipDeviceSynchronize();

		////////////Initialize d_bfHR to store final high-resolution beamformed image /////////////////////////////
		hipMalloc((void**)&d_bfHR, pixels * sizeof(float));
		//zeros << <pixels / num_threads + 1, num_threads >> > (d_bfHR, pixels);  
		hipMemset(d_bfHR, 0, pixels * sizeof(float));

		/////////////////// Transmit delay calculation ////////////////////
		hipMalloc((void**)&d_tx_delay, pixels * num_frames * sizeof(float));
		//transmit delay for all frames,   
		transmit_delay << < pixels * num_frames / num_threads + 1, num_threads >> > (d_x_axis, d_z_axis, d_cen_pos, zd, Nx, Nz, del_convert, num_frames, d_tx_delay);
		hipGetLastError();
		hipDeviceSynchronize();

		////////////Free cuda memory (one time use) ///////////////////////////
		hipFree(d_probe);
		hipFree(d_x_axis);
		hipFree(d_z_axis);
		hipFree(d_cen_pos);

		mFile << "minit end" << std::endl;
		mFile.close();

		return out_string;
	}

	// Function to compute the B-mode image if read from CSV
	extern double** computeImg() {

		// test values
		const int MAX_ITER = 128;
		const int N_RX = 64;
		const int MAX_LINE = 256;
		float PI = 3.14;
		const int MASK_WIDTH = 364;
		const int TILE_SIZE = 4;
		int num_threads = 1024;
		float rx_f_number = 2.0;
		int samples = 2040;
		int N_elements = 128;
		float sampling_frequency = 32.0e6;
		float c = 1540.0;
		int N_active = 8;
		int channels = 128;
		int	Nx = 256;
		int Nz = 1024;
		int frames = 121;
		//int num_frames = 121;
		int skip_frames = 1;
		int	dBvalue = 60;
		float pitch = 0.000465;
		float aper_len = 59.055;
		float zd = 0.00186;
		float sample_spacing = 2.40625e-05;
		float del_convert = 20779.2;
		float rc = 0.0601;
		float scan_angle = 1.01178;
		int pixels = 262144;
		int pix_cha = 33554432;

		float* data = new float[samples * channels];
		float* d_data = 0;
		hipMalloc((void**)&d_data, sizeof(float) * samples * channels);
		int num_frames = 57;

		for (int f = 0; f < num_frames; f++)
		{
			char filename[200];
			sprintf(filename, "inputs/raw_rf_dbsat_Ptsca_arr_%d.csv", f); //all the LR inputs are arranged in a single file

			read_csv_mat(data, filename, 1);    // csv file read

			clock_t begin = clock();   // clock intiated

			hipMemcpy(d_data, data, sizeof(float) * samples * channels, hipMemcpyHostToDevice);

			beamformingLR3 << <(pixels / 256) * channels, 256 >> > (d_bfHR, d_tx_delay, d_rx_delay, d_data, d_rx_apod, samples, pixels, f, num_frames, channels);
			hipGetLastError();
			hipDeviceSynchronize();

			clock_t end = clock();
			float elapsed_secs = float(end - begin) / CLOCKS_PER_SEC;
			//printf("Time for beamforming in ms: %f\n", elapsed_secs * 1000);

		}

		//// check for nan values,
		isnan_test_array << <pixels / num_threads + 1, num_threads >> > (d_bfHR, pixels);
		hipGetLastError();
		hipDeviceSynchronize();

		hipMalloc((void**)&d_bfHRBP, sizeof(float) * pixels);

		//////////// Bandpass filtering using shared memory /////////////////////
		BPfilter1SharedMem << <(pixels + TILE_SIZE - 1) / TILE_SIZE, TILE_SIZE >> > (d_bfHR, d_filt_coeff, pixels, d_bfHRBP);
		hipGetLastError();
		hipDeviceSynchronize();

		//////////////// reshape of the beamformed data ///////////////
		hipMalloc((void**)&dev_beamformed_data1, pixels * sizeof(float));   //234.130 us
		reshape_columnwise << <pixels / num_threads + 1, num_threads >> > (Nx, Nz, dev_beamformed_data1, d_bfHRBP);  //48.864 us
		hipGetLastError();
		hipDeviceSynchronize();

		hipMemcpy(env, dev_beamformed_data1, Nz * Nx * sizeof(float), hipMemcpyDeviceToHost);
		const char* fileout = "sample_output/b_csv_mode.csv";
		csv_write_mat(env, fileout, Nz, Nx);

		//////////////// Free cuda memory (that will be used again) ///////////////
		hipFree(d_data);
		hipFree(d_bfHR);
		hipFree(d_tx_delay);
		hipFree(d_rx_delay);
		hipFree(d_rx_apod);
		hipFree(dev_beamformed_data1);
		hipFree(d_bfHRBP);

		double** outArray = convertsingto2darray(env, Nz, Nx);


		return outArray;
	}

	//// Function to initialize the CUDA memmory if read from Linear prob

	extern int cuMemInitLinear() {

	
		char filename3[200];
		sprintf(filename3, "b_10M.csv");
		read_csv_array(filt_coeff, filename3);    // csv file read
		//cv::imwrite("okMat3.png", testMat0);
	
		// float* d_filt_coeff = 0;
		hipMalloc((void**)&d_filt_coeff, sizeof(float) * MASK_WIDTH);
		hipMemcpy(d_filt_coeff, filt_coeff, sizeof(float) * MASK_WIDTH, hipMemcpyHostToDevice);
	
		////////  Intialization &(or) Memory allocation  //////////////////
		// float* d_data = 0;   // variable to store raw rf data
		hipMalloc((void**)&d_data, sizeof(float) * samples * channels);
	
		// float* d_bfHR = 0;  // variable to store beamformed high-resolution beamformed image 
		hipMalloc((void**)&d_bfHR, pixels * sizeof(float));
		//zeros << <pixels / num_threads + 1, num_threads >> > (d_bfHR, pixels);  
		hipMemset(d_bfHR, 0, pixels * sizeof(float));
	
		// float* dev_beamformed_data1 = 0;   // variable to store reshaped beamformed data
		hipMalloc((void**)&dev_beamformed_data1, pixels * sizeof(float));
	
		// float* d_bfHRBP = 0;  // variable to store beamformed high-resolution bandpass filtered data
		hipMalloc((void**)&d_bfHRBP, sizeof(float) * pixels);
	
		// float* env = new float[pixels]; // Host memory variable to store beamformed high-resolution bandpass filtered data
	
		////////////// z value////////////////////
		float dz = sample_spacing * samples / Nz;  // depth / (Nz - 1) / 1000;   // spacing in axial (z) direction in mm;
		// float* d_z_axis = 0;
		hipMalloc((void**)&d_z_axis, Nz * sizeof(float));
		range << <Nz / num_threads + 1, num_threads >> > (d_z_axis, 0, Nz, dz);
		hipGetLastError();
		hipDeviceSynchronize();
	
		//////////////////////////////// x value////////////////////////////////
		float dx = aper_len / (Nx - 1);
		// float* d_x_axis = 0;
		hipMalloc((void**)&d_x_axis, Nx * sizeof(float));    // 167.939 us
		range << <Nx / num_threads + 1, num_threads >> > (d_x_axis, (-aper_len / 2000), Nx, dx / 1000);
		hipGetLastError();
		hipDeviceSynchronize();
	
		//////////////// Probe geometry, this info can be taken from transducer file ////////////////////
		//float* d_probe = 0;
		hipMalloc((void**)&d_probe, N_elements * sizeof(float));
		range << <1, N_elements >> > (d_probe, (-aper_len / 2000), N_elements, pitch);
		hipGetLastError();
		hipDeviceSynchronize();
	
		/////////////////rx aerture calculation using Fnumber///////////////////////////////
		// rx_aper=rfsca.z/rf_number
		// float* d_rx_aperture = 0;
		hipMalloc((void**)&d_rx_aperture, Nz * sizeof(float));
		element_division << <Nz / num_threads + 1, num_threads >> > (d_z_axis, rx_f_number, Nz, d_rx_aperture);
		hipGetLastError();
		hipDeviceSynchronize();
	
		////////////////////////rx aerture distance////////
		// float* d_rx_ap_distance = 0;
		hipMalloc((void**)&d_rx_ap_distance, channels * Nx * sizeof(float));
		aperture_distance << <Nx * channels / num_threads + 1, num_threads >> > (d_x_axis, d_probe, Nx, channels, d_rx_ap_distance);
		hipGetLastError();
		hipDeviceSynchronize();
	
		///////////////////apodization/////////////////
		// float* d_rx_apod = 0;
		hipMalloc((void**)&d_rx_apod, sizeof(float) * Nz * channels * Nx);
		apodization << <pixels * channels / num_threads + 1, num_threads >> > (d_rx_ap_distance, d_rx_aperture, Nz, Nx, channels, pixels, d_rx_apod);
		hipGetLastError();
		hipDeviceSynchronize();
	
		hipFree(d_rx_aperture);
		hipFree(d_rx_ap_distance);
	
		/////////////////// calculate central positions transmit subaperture ////////////////////
		// float* d_cen_pos = 0;
		hipMalloc((void**)&d_cen_pos, num_frames * sizeof(float));
		Tx_cen_pos << < 1, num_frames >> > (d_cen_pos, N_elements, N_active, pitch, skip_frames, num_frames, d_probe);
	
		/////////////receive delay calculation /////////////////////////////////////////////
		// float* d_rx_delay = 0;
		hipMalloc((void**)&d_rx_delay, pix_cha * sizeof(float));
		receive_delay << < pixels * channels / num_threads + 1, num_threads >> > (d_probe, d_x_axis, d_z_axis, channels, Nx, Nz, del_convert, d_rx_delay);
		hipGetLastError();
		hipDeviceSynchronize();
	
		/////////////////// Transmit delay calculation ////////////////////
		// float* d_tx_delay = 0;
		hipMalloc((void**)&d_tx_delay, pixels * num_frames * sizeof(float));
		//transmit delay for all frames,   
		transmit_delay << < pixels * num_frames / num_threads + 1, num_threads >> > (d_x_axis, d_z_axis, d_cen_pos, zd, Nx, Nz, del_convert, num_frames, d_tx_delay);
		hipGetLastError();
		hipDeviceSynchronize();
	
		////////////Free cuda memory (one time use) ///////////////////////////
		hipFree(d_probe);
		hipFree(d_x_axis);
		hipFree(d_z_axis);
		hipFree(d_cen_pos);
	
		//cv::Mat testMat0 = cv::Mat::zeros(250, 200, CV_8UC1);
		//cv::imwrite("btestMat5.png", testMat0);
		//// Memmory allocation problem in cuda
		//cv::Mat testMat0 = cv::Mat::zeros(250, 200, CV_8UC1);
		//cv::imwrite("btestMat0.png", testMat0);
		//return -1;
	
		USBDevice = new CCyUSBDevice(NULL);
		// Obtain the control endpoint pointer
		ept = USBDevice->ControlEndPt;
		if (!ept) {
			// Could not get Control endpoin
			// printf("Could not get Control endpoint.\n");
			//cv::Mat testMat0 = cv::Mat::zeros(250, 200, CV_8UC1);
			//cv::imwrite("errorMat1.png", testMat0);
			return 2;
		}
	
		//	Any h/w initialization Error
		//	cv::Mat testMat0 = cv::Mat::zeros(250, 200, CV_8UC1);
		//	cv::imwrite("btestMat3.png", testMat0);
		//	return -3;
	
		return 0;
	
	}

	// Function to initialize the CUDA memmory if read from CurvLinear prob

	extern int cuMemInitCurv(double* probPrms) {
	
		//std::ofstream mFile1, mFile2;
		//mFile1.open("sample_output/curve_params.txt");
		//mFile2.open("sample_output/curve_params2.txt");

		//for (int i = 0; i < 29; i++) {
		//	mFile1  << probPrms[i] << std::endl;
		//}

		const int MAX_ITER = (int)probPrms[0]; //mFile2 << MAX_ITER << std::endl;
		const int N_RX = (int)probPrms[1];// mFile2 << N_RX << std::endl;
		const int MAX_LINE = (int)probPrms[2]; //mFile2 << MAX_LINE << std::endl;
		float PI = (float)probPrms[3]; //mFile2 << PI << std::endl;
		const int MASK_WIDTH = (int)probPrms[4];// mFile2 << MASK_WIDTH << std::endl;
		const int TILE_SIZE = (int)probPrms[5]; //mFile2 << TILE_SIZE << std::endl;
		int num_threads = (int)probPrms[6]; //mFile2 << num_threads << std::endl;
		float rx_f_number = (float)probPrms[7]; //mFile2 << rx_f_number << std::endl;
		int samples = (int)probPrms[8]; ///mFile2 << samples << std::endl;
		int N_elements = (int)probPrms[9]; //mFile2 << N_elements << std::endl;
		float sampling_frequency = (float)probPrms[10]; ///mFile2 << sampling_frequency << std::endl;
		float c = (float)probPrms[11]; //mFile2 << c << std::endl;
		int N_active = (int)probPrms[12]; //mFile2 << N_active << std::endl;
		int channels = (int)probPrms[13]; //mFile2 << channels << std::endl;
		int	Nx = (int)probPrms[14]; //mFile2 << Nx << std::endl;
		int Nz = (int)probPrms[15]; //mFile2 << Nz << std::endl;
		int frames = (int)probPrms[16]; //mFile2 << frames << std::endl;
		int num_frames = (int)probPrms[17]; //mFile2 << num_frames << std::endl;
		int skip_frames = (int)probPrms[18]; //mFile2 << skip_frames << std::endl;
		int	dBvalue = (int)probPrms[19];// mFile2 << dBvalue << std::endl;
		float pitch = (float)probPrms[20]; //mFile2 << pitch << std::endl;
		float aper_len = (float)probPrms[21];// mFile2 << aper_len << std::endl;
		float zd = (float)probPrms[22]; //mFile2 << zd << std::endl;
		float sample_spacing = (float)probPrms[23]; //mFile2 << sample_spacing << std::endl;
		float del_convert = (float)probPrms[24]; //mFile2 << del_convert << std::endl;
		float rc = (float)probPrms[25]; //mFile2 << rc << std::endl;
		float scan_angle = (float)probPrms[26]; //mFile2 << scan_angle  << std::endl;
		int pixels = (int)probPrms[27]; //mFile2 << pixels << std::endl;
		int pix_cha = (int)probPrms[28]; //mFile2 << pix_cha << std::endl;

		//mFile1.close();
		//mFile2.close();


		//const int MAX_ITER = 128;
		//const int N_RX =  64;
		//const int MAX_LINE = 256;
		//float PI = 3.14;
		//const int MASK_WIDTH = 364;
		//const int TILE_SIZE = 4;
		//int num_threads = 1024;
		//float rx_f_number = 2.0;
		//int samples = 2040;
		//int N_elements = 128;
		//float sampling_frequency = 32.0e6;
		//float c = 1540.0;
		//int N_active = 8;
		//int channels = 128;
		//int	Nx = 256;
		//int Nz = 1024;
		//int frames = 121;
		//int num_frames = 121;
		//int skip_frames = 1;
		//int	dBvalue = 60;
		//float pitch = 0.000465;
		//float aper_len = 59.055;
		//float zd = 0.00186;
		//float sample_spacing = 2.40625e-05;
		//float del_convert = 20779.2;
		//float rc = 0.0601;
		//float scan_angle = 1.01178;
		//int pixels = 262144;
		//int pix_cha = 33554432;

		float* filt_coeff = new float[MASK_WIDTH];
		char filename1[200];
		sprintf(filename1, "b_10M.csv");
		read_csv_array(filt_coeff, filename1);    // csv file read
	
		//float* d_filt_coeff = 0;
		hipMalloc((void**)&d_filt_coeff, sizeof(float) * MASK_WIDTH);
		hipMemcpy(d_filt_coeff, filt_coeff, sizeof(float) * MASK_WIDTH, hipMemcpyHostToDevice);
	
		////////  Intialization &(or) Memory allocation  //////////////////
		//float* d_data = 0;   // variable to store raw rf data
		hipMalloc((void**)&d_data, sizeof(float) * samples * channels);
	
		//float* d_bfHR = 0;  // variable to store beamformed high-resolution beamformed image 
		hipMalloc((void**)&d_bfHR, pixels * sizeof(float));
		//zeros << <pixels / num_threads + 1, num_threads >> > (d_bfHR, pixels);  
		hipMemset(d_bfHR, 0, pixels * sizeof(float));
	
		//float* dev_beamformed_data1 = 0;   // variable to store reshaped beamformed data
		hipMalloc((void**)&dev_beamformed_data1, pixels * sizeof(float));
	
		//float* d_bfHRBP = 0;  // variable to store beamformed high-resolution bandpass filtered data
		hipMalloc((void**)&d_bfHRBP, sizeof(float) * pixels);
	
		//float* env = new float[pixels]; // Host memory variable to store beamformed high-resolution bandpass filtered data
	
		/////////////////// theta positions for all elements ////////////////////
		//float* d_theta = 0;
		hipMalloc((void**)&d_theta, N_elements * sizeof(float));
		range << <Nx / num_threads + 1, num_threads >> > (d_theta, (-scan_angle / 2), N_elements, (scan_angle / (N_elements - 1)));
	
	
		///////////// theta for grid /////////////////  theta = -scan_angle / 2 : scan_angle / (elements - 1) : scan_angle / 2;
		//float* d_theta1 = 0;
		hipMalloc((void**)&d_theta1, Nx * sizeof(float));
		range << <Nx / num_threads + 1, num_threads >> > (d_theta1, (-scan_angle / 2), Nx, (scan_angle / (Nx - 1)));
		hipGetLastError();
		hipDeviceSynchronize();
	
		////////////// z value////////////////////
		float dz = sample_spacing * samples / Nz;  // depth / (Nz - 1) / 1000;   // spacing in axial (z) direction in mm;
		float* d_z_axis = 0;
		hipMalloc((void**)&d_z_axis, Nz * sizeof(float));
		range << <Nz / num_threads + 1, num_threads >> > (d_z_axis, 0, Nz, dz);
		hipGetLastError();
		hipDeviceSynchronize();
	
		//////////////////////////////// x value////////////////////////////////
		float dx = aper_len / (Nx - 1);
		float* d_x_axis = 0;
		hipMalloc((void**)&d_x_axis, Nx * sizeof(float));
		range << <Nx / num_threads + 1, num_threads >> > (d_x_axis, (-aper_len / 2000), Nx, dx / 1000);
		hipGetLastError();
		hipDeviceSynchronize();
	
		//////////////// Probe geometry, this info can be taken from transducer file ////////////////////
		//float* d_probe = 0;
		hipMalloc((void**)&d_probe, N_elements * sizeof(float));
		//hipMemcpy(d_probe, probe_ge_x, N_elements * sizeof(double), hipMemcpyHostToDevice);
		range << <1, N_elements >> > (d_probe, (-aper_len / 2000), N_elements, pitch);
		hipGetLastError();
		hipDeviceSynchronize();
	
		/////////////////rx aerture calculation using Fnumber///////////////////////////////
		// rx_aper=rfsca.z/rf_number
		//float* d_rx_aperture = 0;
		hipMalloc((void**)&d_rx_aperture, Nz * sizeof(float));
		element_division << <Nz / num_threads + 1, num_threads >> > (d_z_axis, rx_f_number, Nz, d_rx_aperture);
		hipGetLastError();
		hipDeviceSynchronize();
	
		////////////////////////rx aerture distance////////
		//float* d_rx_ap_distance = 0;
		hipMalloc((void**)&d_rx_ap_distance, channels * Nx * sizeof(float));  //20.087 us
		aperture_distance << <Nx * channels / num_threads + 1, num_threads >> > (d_x_axis, d_probe, Nx, channels, d_rx_ap_distance);
		hipGetLastError();
		hipDeviceSynchronize();
	
		///////////////////apodization/////////////////
		//float* d_rx_apod = 0;
		hipMalloc((void**)&d_rx_apod, sizeof(float) * Nz * channels * Nx);
		apodization << <pixels * channels / num_threads + 1, num_threads >> > (d_rx_ap_distance, d_rx_aperture, Nz, Nx, channels, pixels, d_rx_apod);
		hipGetLastError();
		hipDeviceSynchronize();
	
		//// check for nan values,
		isnan_test_array << <pixels * channels / num_threads + 1, num_threads >> > (d_rx_apod, pixels * channels);
		hipGetLastError();
		hipDeviceSynchronize();
	
		hipFree(d_rx_aperture);
		hipFree(d_rx_ap_distance);
	
		/////////////receive delay calculation /////////////////////////////////////////////
		//float* d_rx_delay = 0;
		hipMalloc((void**)&d_rx_delay, pix_cha * sizeof(float));
		receive_delay << < pixels * channels / num_threads + 1, num_threads >> > (d_theta, d_theta1, rc, d_z_axis, channels, Nx, Nz, del_convert, d_rx_delay);
		hipGetLastError();
		hipDeviceSynchronize();
	
		/////////////////// theta positions for all elements ////////////////////
		//float* d_theta_tx = 0;
		hipMalloc((void**)&d_theta_tx, num_frames * sizeof(float));
		theta1 << < 1, num_frames >> > (d_theta_tx, d_theta, frames, N_active, skip_frames);
	
		/////////////////// Transmit delay calculation ////////////////////
		//float* d_tx_delay = 0;
		hipMalloc((void**)&d_tx_delay, pixels * num_frames * sizeof(float));
		//transmitter delay for 16 frames,  
		transmit_delay << < pixels * num_frames / num_threads + 1, num_threads >> > (d_theta1, d_z_axis, rc, d_theta_tx, Nx, Nz, del_convert, num_frames, zd, d_tx_delay);
		hipGetLastError();
		hipDeviceSynchronize();
	
		hipFree(d_theta1);
		hipFree(d_probe);
		hipFree(d_x_axis);
		hipFree(d_z_axis);
		hipFree(d_theta_tx);
	
		//mFile << "pitch : " << pitch << std::endl << " aper_len:  " << aper_len << " zd : " << zd <<
		//	" sample_spacing : " << sample_spacing << " del_convert : " << del_convert << " rc : " << rc <<
		//	" scan_angle " << scan_angle << " pixels " << pixels << " pix_cha " << pix_cha << std::endl;
	
		zeroC(rximg, samples * N_elements);   // set rx_img array values to zero.
	
		USBDevice = new CCyUSBDevice(NULL);
		// Obtain the control endpoint pointer
		ept = USBDevice->ControlEndPt;
		if (!ept) {
			//printf("Could not get Control endpoint.\n");
			return 1;
		}

		
		return 0;
	}

	// Function to compute the B - mode image if read from Linear prob

	extern double** computeLinearImg() {

		static double pix2 = 0.0;
		// cv::Mat testMat0 = cv::Mat::zeros(250, 1000, CV_8UC1);
		//for (double pix = 0; pix < 255.000; pix++)
		//{
		unsigned char buf[16 * 1024];
		int row = 0;  // Keep track of how many rows have been added
		errno_t err;
		char line[MAX_LINE]; // Max possible line length?
		FILE* fp;
		if ((err = fopen_s(&fp, "out25.txt", "r")) != 0) {
			//printf("Could not open config file for reading.\n");
			//cv::imwrite("errorMat2.png", testMat0);
			exit(1);
		}
		// Send a vendor request (bRequest = 0x05) to the device
		ept->Target = TGT_DEVICE;
		ept->ReqType = REQ_VENDOR;
		ept->Direction = DIR_TO_DEVICE;
		ept->ReqCode = 0x05;
		ept->Value = 1;
		ept->Index = 0;
		ept->TimeOut = 100;  // set timeout to 100ms for quick response
		// Endpoint for reading back data
		CCyBulkEndPoint* ept_in;
		ept_in = USBDevice->BulkInEndPt;
		if (!ept_in) {
			//printf("No IN endpoint??\n");
			exit(1);
		}
		ept_in->MaxPktSize = 16384;
		ept_in->TimeOut = 100;  // set timeout to 100ms for reading
		int iteration = 0;
		int errcount = 0;
		unsigned int addr, data;
		unsigned char recvbuf[2048 * 64 * 2];
		const int MAXROWS = 2040;
		LONG rxlen = MAXROWS * 64 * 2;
	
		// unsigned int start = clock();
		while (fgets(line, MAX_LINE, fp)) {
			line[strcspn(line, "\n")] = 0; // Trim trailing newline
			if ((strlen(line) == 0) || (line[0] == ' ') || (line[0] == '#')) {
				//printf("Skipping [%s]\n", line);
			}
			else if (line[0] == 'O') {
				sscanf_s(line, "O %04X %08X ", &addr, &data);
				//printf("Write %08X to Obelix %04X\n", data, addr);
				row = insert_row(buf, row, addr, data);
			}
			else if (line[0] == 'T') {
				sscanf_s(line, "T %04X %08X ", &addr, &data);
				row = insert_row(buf, row, 0x6, 0x40000000 | addr);
				row = insert_row(buf, row, 0x7, data);
				row = insert_row(buf, row, 0x6, 0xC0000000 | addr);
				//printf("Write %08X to TX %04X\n", data, addr);
			}
			else if (line[0] == 'A') {
				sscanf_s(line, "A %04X %08X ", &addr, &data);
				row = insert_row(buf, row, 0x6, 0x00000000 | addr);
				row = insert_row(buf, row, 0x7, data);
				row = insert_row(buf, row, 0x6, 0x80000000 | addr);
				//printf("Write %08X to AFE %04X\n", data, addr);
			}
			else if (line[0] == 'C') {  // CAPTURE
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x01);
				//write_rows(ept, buf, row);  // Send commands
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x10);
				//write_rows(ept, buf, row);  // Send commands
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x00);
				ept_in->Abort();
				ept_in->Reset();
				write_rows(ept, buf, row);  // Send commands
				wait(1);
				if (read_chunk(ept_in, recvbuf, rxlen)) {
					short* rxdata = (short*)(recvbuf);
					for (int i = 0; i < rxlen / 2; i++) {
						if (rxdata[i] >= 512) rxdata[i] -= 1024;
					}
					// Trying to read only first N-1 rows and discard 1st sample
					for (int i = 0; i < 64; i++) {
						for (int j = 0; j < MAXROWS - 1; j++) {
							//rximg[iteration][i][j] = rxdata[j*64+i+2];
							rximg[i * MAXROWS + j] = rxdata[j * 64 + i + 2];
						}
					}
					//saveToFile(iteration, rxlen, recvbuf);
				}
				else {
					errcount++;
				}
				hipMemcpy(d_data, rximg, sizeof(float) * samples * channels, hipMemcpyHostToDevice);
				beamformingLR3 << <(pixels / 256) * channels, 256 >> > (d_bfHR, d_tx_delay, d_rx_delay, d_data, d_rx_apod, samples, pixels, iteration, num_frames, channels);
				hipGetLastError();
				hipDeviceSynchronize();
				iteration++; // Increment iteration after saving to image
				row = 0;   // Reset buffer for next iteration
			}
			else {
				printf("Don't know how to handle [%s] yet.\n", line);
			}
		}
	
		//// check for nan values,
		isnan_test_array << <pixels / num_threads + 1, num_threads >> > (d_bfHR, pixels);
		hipGetLastError();
		hipDeviceSynchronize();
		//////////// Bandpass filtering using shared memory /////////////////////
		BPfilter1SharedMem << <(pixels + TILE_SIZE - 1) / TILE_SIZE, TILE_SIZE >> > (d_bfHR, d_filt_coeff, pixels, d_bfHRBP);
		hipGetLastError();
		hipDeviceSynchronize();
		//////////////// reshape of the beamformed data ///////////////
		reshape_columnwise << <pixels / num_threads + 1, num_threads >> > (Nx, Nz, dev_beamformed_data1, d_bfHRBP);
		hipGetLastError();
		hipDeviceSynchronize();
		hipMemcpy(env, dev_beamformed_data1, Nz * Nx * sizeof(float), hipMemcpyDeviceToHost);
		char fileout[200];
		//sprintf(fileout, "b_mode_%d.csv", 1); //all the 16 inputs are arranged in a single file
		//csv_write_mat(env, fileout, Nz, Nx);
		double** outArray = convertsingto2darray(env, Nz, Nx);
	
		//outMatCrp = outMat(cv::Range(0, Nz - croppedBot), cv::Range(0, Nx));
		//envolepMat = hilbertTrans4(outMatCrp, 1.0);
		//// log compression
		//logcMat = logTransform(envolepMat);
		//cv::Mat outMat = cv::Mat::zeros(1024, 254, CV_64FC1);
		//for (int r = 0; r < 1024; r++) {
		//	for (int c = 0; c < 254; c++) {
		//		outMat.at<double>(r, c) = pix2;
		//	}
		//}
		//cv::imwrite("sample_output/inLogMat" + std::to_string(pix2) + ".png", logcMat);
		//pix2 = pix2 + 10.0;
		//}
	
		//////////////// Free cuda memory (that will be used again) ///////////////
		hipFree(d_data);
		hipFree(d_bfHR);
		hipFree(d_tx_delay);
		hipFree(d_rx_delay);
		hipFree(d_rx_apod);
		hipFree(dev_beamformed_data1);
		hipFree(d_bfHRBP);
		hipFree(d_filt_coeff);
	
		return outArray;
		//return ConvertMatto2DArray(logcMat);
	}

	// Function to compute the B - mode image if read from Linear prob

	extern double** computeCurveImg(double* probPrms)
	{

		const int MAX_ITER = 128;
		const int N_RX = 64;
		const int MAX_LINE = 256;
		//float PI = 3.14;
		//const int MASK_WIDTH = 364;
		//const int TILE_SIZE = 4;
		//int num_threads = 1024;
		//float rx_f_number = 2.0;
		//int samples = 2040;
		//int N_elements = 128;
		//float sampling_frequency = 32.0e6;
		//float c = 1540.0;
		//int N_active = 8;
		//int channels = 128;
		//int	Nx = 256;
		//int Nz = 1024;
		//int frames = 121;
		//int num_frames = 121;
		//int skip_frames = 1;
		//int	dBvalue = 60;
		//float pitch = 0.000465;
		//float aper_len = 59.055;
		//float zd = 0.00186;
		//float sample_spacing = 2.40625e-05;
		//float del_convert = 20779.2;
		//float rc = 0.0601;
		//float scan_angle = 1.01178;
		//int pixels = 262144;
		//int pix_cha = 33554432;

		//const int MAX_ITER = (const int)probPrms[0]; //mFile2 << MAX_ITER << std::endl;
		//const int N_RX = (const int)probPrms[1];// mFile2 << N_RX << std::endl;
		//const int MAX_LINE = (const int)probPrms[2]; //mFile2 << MAX_LINE << std::endl;
		float PI = (float)probPrms[3]; //mFile2 << PI << std::endl;
		const int MASK_WIDTH = (int)probPrms[4];// mFile2 << MASK_WIDTH << std::endl;
		const int TILE_SIZE = (int)probPrms[5]; //mFile2 << TILE_SIZE << std::endl;
		int num_threads = (int)probPrms[6]; //mFile2 << num_threads << std::endl;
		float rx_f_number = (float)probPrms[7]; //mFile2 << rx_f_number << std::endl;
		int samples = (int)probPrms[8]; ///mFile2 << samples << std::endl;
		int N_elements = (int)probPrms[9]; //mFile2 << N_elements << std::endl;
		float sampling_frequency = (float)probPrms[10]; ///mFile2 << sampling_frequency << std::endl;
		float c = (float)probPrms[11]; //mFile2 << c << std::endl;
		int N_active = (int)probPrms[12]; //mFile2 << N_active << std::endl;
		int channels = (int)probPrms[13]; //mFile2 << channels << std::endl;
		int	Nx = (int)probPrms[14]; //mFile2 << Nx << std::endl;
		int Nz = (int)probPrms[15]; //mFile2 << Nz << std::endl;
		int frames = (int)probPrms[16]; //mFile2 << frames << std::endl;
		int num_frames = (int)probPrms[17]; //mFile2 << num_frames << std::endl;
		int skip_frames = (int)probPrms[18]; //mFile2 << skip_frames << std::endl;
		int	dBvalue = (int)probPrms[19];// mFile2 << dBvalue << std::endl;
		float pitch = (float)probPrms[20]; //mFile2 << pitch << std::endl;
		float aper_len = (float)probPrms[21];// mFile2 << aper_len << std::endl;
		float zd = (float)probPrms[22]; //mFile2 << zd << std::endl;
		float sample_spacing = (float)probPrms[23]; //mFile2 << sample_spacing << std::endl;
		float del_convert = (float)probPrms[24]; //mFile2 << del_convert << std::endl;
		float rc = (float)probPrms[25]; //mFile2 << rc << std::endl;
		float scan_angle = (float)probPrms[26]; //mFile2 << scan_angle  << std::endl;
		int pixels = (int)probPrms[27]; //mFile2 << pixels << std::endl;
		int pix_cha = (int)probPrms[28]; //mFile2 << pix_cha << std::endl;

	
		//for (int i = 0; i < 2; i++) {
		cv::Mat testMat0 = cv::Mat::zeros(250, 1000, CV_8UC1);
		unsigned char buf[16 * 1024];
	
		errno_t err;
		char line[MAX_LINE]; // Max possible line length?
		FILE* fp;
		if ((err = fopen_s(&fp, "out25_curvi.txt", "r")) != 0) {
			//printf("Could not open config file for reading.\n");
			//exit(1);
			return nullptr;
		}
	
		// Send a vendor request (bRequest = 0x05) to the device
		ept->Target = TGT_DEVICE;
		ept->ReqType = REQ_VENDOR;
		ept->Direction = DIR_TO_DEVICE;
		ept->ReqCode = 0x05;
		ept->Value = 1;
		ept->Index = 0;
		ept->TimeOut = 100;  // set timeout to 100ms for quick response
		cv::imwrite("sample_output/testMat3.png", testMat0);
	
		// Endpoint for reading back data
		CCyBulkEndPoint* ept_in;
		ept_in = USBDevice->BulkInEndPt;
		if (!ept_in) {
			//printf("No IN endpoint??\n");
			//exit(1);
			cv::imwrite("sample_output/errorMat3.png", testMat0);
			return nullptr;
		}
		ept_in->MaxPktSize = 16384;
		ept_in->TimeOut = 100;  // set timeout to 100ms for readin
		int iteration = 0;
		int errcount = 0;
		int row = 0;  // Keep track of how many rows have been added
	
		unsigned int addr, data;
		unsigned char recvbuf[2048 * N_RX * 2];
		const int MAXROWS = 2040;
		LONG rxlen = MAXROWS * N_RX * 2;
		cv::imwrite("sample_output/testMat4.png", testMat0);
		//unsigned int start = clock();
		while (fgets(line, MAX_LINE, fp)) {
			line[strcspn(line, "\n")] = 0; // Trim trailing newline
			if ((strlen(line) == 0) || (line[0] == ' ') || (line[0] == '#')) {
				//printf("Skipping [%s]\n", line);
			}
			else if (line[0] == 'O') {
				sscanf_s(line, "O %04X %08X ", &addr, &data);
				//printf("Write %08X to Obelix %04X\n", data, addr);
				row = insert_row(buf, row, addr, data);
			}
			else if (line[0] == 'T') {
				sscanf_s(line, "T %04X %08X ", &addr, &data);
				row = insert_row(buf, row, 0x6, 0x40000000 | addr);
				row = insert_row(buf, row, 0x7, data);
				row = insert_row(buf, row, 0x6, 0xC0000000 | addr);
				//printf("Write %08X to TX %04X\n", data, addr);
			}
			else if (line[0] == 'A') {
				sscanf_s(line, "A %04X %08X ", &addr, &data);
				row = insert_row(buf, row, 0x6, 0x00000000 | addr);
				row = insert_row(buf, row, 0x7, data);
				row = insert_row(buf, row, 0x6, 0x80000000 | addr);
				//printf("Write %08X to AFE %04X\n", data, addr);
			}
			else if (line[0] == 'C') {  // CAPTURE
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x01);
				//write_rows(ept, buf, row);  // Send commands
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x10);
				//write_rows(ept, buf, row);  // Send commands
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x00);
	
				ept_in->Abort();
				ept_in->Reset();
	
				write_rows(ept, buf, row);  // Send commands
				//wait(100);
				//row = insert_row(buf, row, 0x4, 0x03);
				//row = insert_row(buf, row, 0x4, 0x10);
				//row = insert_row(buf, row, 0x4, 0x00);
				//printf("CAPTURE %2d: ", iteration);
				//write_rows(ept, buf, row);  // Send commands
				// One iteration should have 2048 samples * 64 channels * 2 bytes each
	
	
				wait(1);
				if (read_chunk(ept_in, recvbuf, rxlen)) {
					short* rxdata = (short*)(recvbuf);
					for (int i = 0; i < rxlen / 2; i++) {
						if (rxdata[i] >= 512) rxdata[i] -= 1024;
					}
	
					// Trying to read only first N-1 rows and discard 1st sample
					for (int i = 0; i < N_RX; i++) {
						for (int j = 0; j < MAXROWS - 1; j++) {
							//rximg[iteration][i][j] = rxdata[j*64+i+2];
							if (iteration < 29) {      // start from 0 index, so 30-1 
								rximg[i * MAXROWS + j] = rxdata[j * N_RX + i + 2];
							}
							else if (iteration > 91) {
								rximg[(i + 64) * MAXROWS + j] = rxdata[j * N_RX + i + 2];
							}
							else {
								rximg[(i + iteration - 28) * MAXROWS + j] = rxdata[j * N_RX + i + 2];
							}
							//rximg[i * MAXROWS + j] = rxdata[j * N_RX + i + 2];
						}
					}
					//saveToFile(iteration, rxlen, recvbuf);
				}
				else {
					errcount++;
				}
	
				//clock_t begin = clock();   // clock intiated
	
				hipMemcpy(d_data, rximg, sizeof(float) * samples * channels, hipMemcpyHostToDevice);
	
				beamformingLR3 << <(pixels / 256) * channels, 256 >> > (d_bfHR, d_tx_delay, d_rx_delay, d_data, d_rx_apod, samples, pixels, iteration, num_frames, channels);
				hipGetLastError();
				hipDeviceSynchronize();
	
				//clock_t end = clock();
				//float elapsed_secs = float(end - begin) / CLOCKS_PER_SEC;
				//printf("Time for beamforming in ms: %f\n", elapsed_secs * 1000);
	
	
				iteration++; // Increment iteration after saving to image
				row = 0;   // Reset buffer for next iteration
			}
			else {
				printf("Don't know how to handle [%s] yet.\n", line);
				cv::imwrite("sample_output/errorMat3.png", testMat0);
			}
		}
		//unsigned int stop = clock();
		//printf("\n\n\n******\n");
		//printf("Ran %d iterations with %d errors in %d ms\n", iteration, errcount, stop - start);
	
		cv::imwrite("sample_output/testMat5.png", testMat0);
		//////////// Bandpass filtering using shared memory /////////////////////
		BPfilter1SharedMem << <(pixels + TILE_SIZE - 1) / TILE_SIZE, TILE_SIZE >> > (d_bfHR, d_filt_coeff, pixels, d_bfHRBP);
		hipGetLastError();
		hipDeviceSynchronize();
		//////////////// reshape of the beamformed data ///////////////
		reshape_columnwise << <pixels / num_threads + 1, num_threads >> > (Nx, Nz, dev_beamformed_data1, d_bfHRBP);
		hipGetLastError();
		hipDeviceSynchronize();
		hipMemcpy(env, dev_beamformed_data1, Nz * Nx * sizeof(float), hipMemcpyDeviceToHost);
		char fileout[200];
		sprintf(fileout, "sample_output/b_curve_mode.csv"); //all the 16 inputs are arranged in a single file
		csv_write_mat(env, fileout, Nz, Nx);
		cv::imwrite("sample_output/testMat6.png", testMat0);
		double** outArray = convertsingto2darray(env, Nz, Nx);
		//////////////// Free cuda memory (that will be used again) ///////////////
		hipFree(d_data);
		hipFree(d_bfHR);
		hipFree(d_tx_delay);
		hipFree(d_rx_delay);
		hipFree(d_rx_apod);
		hipFree(dev_beamformed_data1);
		return outArray;
	}

}