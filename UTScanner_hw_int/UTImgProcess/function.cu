#include "hip/hip_runtime.h"

// // It reconstructs LR images separately and then add them to get HR image.
// Delay values and apodization are reshaped into [1 1024, 1 1024, ... pixels] ...
#include <stdlib.h>
#include <stdio.h>
#include <iomanip>
#include <ctime>
#include <math.h>
#include <string.h>
//#define PI 3.14159
//#define TILE_SIZE 4
//#define MASK_WIDTH 364

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>      /// Add "cufft.lib" in the linker input to use cufft. 
#include "hip/hip_runtime.h"
#include <windows.h>
#include ""
#include "device_func1.h"
#include "host_func1.h"
#include "beamforming_func1.h"

// include OpenCV Header
#include <opencv2/opencv.hpp>
#include <opencv2/highgui.hpp>
#include "mat_operations.h"


/////BMode functions/////////////
__global__ void log_conv(float* data_hilbert, float* env, hipfftComplex* d_input_value, int row_org, int col)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int index = j * col + i;

	if (i < col && j < row_org)
	{
		// divide by 'size' is to ensure that the FFT equation holds good.
		//real_d_input_value = (d_input_value[i].x / (float)size);   // Extract real value
		float real = d_input_value[index].x / ((float)row_org * (float)col);
		float img = d_input_value[index].y / ((float)row_org * (float)col);
		data_hilbert[index] = fabs(sqrt((real * real) + (img * img))); // Absolute value

		env[index] = 20 * log10(data_hilbert[index]);     // log compression

	}
}

__global__ void db_conv(float* env, float max, int size, int dBvalue)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < size)
	{

		env[i] = env[i] - max;     //env_dB = env_dB - max(max(env_dB));                                    // Normalization
		env[i] = (float)127.0 * (env[i] + (float)dBvalue) / (float)dBvalue;              // dB conversion
	}
}

__global__ void point_wise_product(hipfftComplex* a, int* b, int row_org, int col)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if ((i < col) && (j < row_org))
	{
		a[j * col + i].x = a[j * col + i].x * b[j];
		a[j * col + i].y = a[j * col + i].y * b[j];
	}
}

__global__ void real2complex(float* f, hipfftComplex* fc, int N1, int N2)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int index = j * N2 + i;

	if (i < N2 && j < N1)
	{
		fc[index].x = f[index];
		fc[index].y = 0.0f;

	}

}

/////////////////////////////////////////////////////////////////////////
void Generate_Pointwise_Coeff(int* pointwise_coeff, int size)
{
	if ((size % 2) == 0)
	{
		pointwise_coeff[0] = 1;
		pointwise_coeff[size / 2] = 1;

		for (unsigned int i = 1; i < size / 2; i++)
		{
			pointwise_coeff[i] = 2;
		}
		for (unsigned int i = (size / 2) + 1; i < size; i++)
		{
			pointwise_coeff[i] = 0;
		}
	}
	else
	{
		pointwise_coeff[0] = 1;
		//pointwise_coeff[size / 2] = 1;

		for (unsigned int i = 1; i <= size / 2; i++)
		{
			pointwise_coeff[i] = 2;
		}
		for (unsigned int i = (size / 2) + 1; i < size; i++)
		{
			pointwise_coeff[i] = 0;
		}
	}

}
/////////////////////////////////////////////////////////////

__global__ void aperture_distance(float* mat1, float* mat2, int Nx, int channels, float* mat_out)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	int i = x / channels;
	int j = x % channels;

	if (x < Nx * channels)
	{
		mat_out[i * channels + j] = fabs(mat1[i] - mat2[j]);
	}
}

//__global__ void apodization(float* distance, float* aperture, int Nz, int Nx, int channels, int pixels, float* apod)
//{
//	int x = blockDim.x * blockIdx.x + threadIdx.x;
//	int i = x / Nz;
//	int j = x % Nz;
//
//	if (x < Nz * Nx)
//	{
//		for (int k = 0; k < channels; k++)
//		{
//			bool temp = distance[i * channels + k] <= (aperture[j] / 2);
//			apod[x * channels + k] = temp * (0.5 + 0.5 * cos(2 * PI * distance[i * channels + k] / aperture[j]));
//		}
//		bool temp = distance[i * channels + k] <= (aperture[j] / 2.0);
//		apod[x] = temp * (double)(0.5 + 0.5 * cos(2 * PI * distance[i * channels + k] / aperture[j]));
//	}
//}

__global__ void apodization(float* distance, float* aperture, int Nz, int Nx, int channels, int pixels, float* apod)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int i = x / Nz;
	int ii = i % Nx;
	int j = x % Nz;
	int nrx = x / pixels;
	float PI = 3.14159;

	if (x < pixels * channels)
	{
		bool temp = distance[ii * channels + nrx] <= (aperture[j] / 2);
		apod[i * Nz + j] = temp * (0.5 + 0.5 * cos(2 * PI * distance[ii * channels + nrx] / aperture[j]));
	}
}

// receive_delay calculation
__global__ void receive_delay(float* probe_ge_x, float* x_axis1, float* z_axis1, int channels, int Nx, int Nz, float del_convert, float* rx_delay)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;

	if (x < Nx * Nz * channels)
	{
		int i = x / Nz;
		int ii = i % Nx;
		int j = x % Nz;
		int nrx = x / (Nx * Nz);
		rx_delay[i * Nz + j] = (sqrt((probe_ge_x[nrx] - x_axis1[ii]) * (probe_ge_x[nrx] - x_axis1[ii]) + ((z_axis1[j]) * (z_axis1[j])))) * del_convert;
		// 1867 - 210 = 1657
		//rx_delay[i * Nx + j] = sqrt(rc * rc + (rc + z_axis[ii]) * (rc + z_axis[ii]) - 2 * rc * (rc + z_axis[ii]) * cos(theta[nrx] - theta1[j])) * del_convert;
	}
}

__global__ void theta1(float* theta_active, float* theta, int frames, int N_active, int skip_frames)
{

	int x = threadIdx.x;
	int f = 0;
	for (int i = 1; i <= frames; i += skip_frames)
	{
		theta_active[f] = theta[i - 1];
		f++;
	}
}

// This function calculates TX central aperture position
__global__ void Tx_cen_pos(float* cen_pos, int N_elements, int N_active, float pitch, int skip_frames, int num_frames, float* probe)
{

	int x = threadIdx.x;

	if (x < num_frames)
	{
		//cen_pos[x] = pitch * ((N_active / 2) + (N_active * (x)-N_elements / 2));
		cen_pos[x] = probe[x * skip_frames + 4];
	}
}

//  transmit_delay calculation
__global__ void transmit_delay(float* x_axis1, float* z_axis1, float* k1, float zd, int Nx, int Nz, float del_convert, int num_frames, float* tx_delay)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int i = x / Nz;
	int ii = i % Nx;
	int j = x % Nz;
	int f = x / (Nx * Nz);

	if (x < Nx * Nz * num_frames)
	{
		tx_delay[i * Nz + j] = (sqrt(((k1[f] - x_axis1[ii]) * (k1[f] - x_axis1[ii])) + ((zd + z_axis1[j]) * (zd + z_axis1[j])))) * del_convert;
		// 1875-210 = 1665
		//tx_delay[i * Nx + j] = (zd + sqrt(rc * rc + (rc + z_axis[ii]) * (rc + z_axis[ii]) - 2 * rc * (rc + z_axis[ii]) * cos(theta_tx[f] - theta1[j]))) * del_convert;
		//first 256*1024 for frame 1, next 256*1024 for frame 2........
	}
}

__global__ void beamformingLR3(float* beamformed_data1, float* tx_delay, float* rx_delay, float* data, float* rx_apod, int samples, int pixels, int f, int num_frames, int channels)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	int nrx = x / pixels;   // nrx - nth A-line
	int pix = x & (pixels - 1); // x% pixels;     // pixel location

	int pixel_pos = round((float)tx_delay[f * pixels + pix] + (float)rx_delay[x]);   // delay value estimation from tx and rx delay values

	if (pixel_pos < samples)
	{
		beamformed_data1[pix] += rx_apod[x] * data[(nrx * samples + pixel_pos - 1)];   // Extract data based on the delay values and multiplying with apodization value
	}
}

__global__ void add_ele(float* data, int pixels, float* out_data)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < pixels)
	{
		out_data[x] += data[x];
	}
}

__global__ void sample1(float* tx_delay, float* rx_delay, int pixels, int channels, float c, float sampling_frequency, float* data1, float* rx_apod, float* data, int samples, int columns)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int f = blockDim.y * blockIdx.y + threadIdx.y;
	int nrx = x / pixels; //channels
	int pix = x % pixels; //pixels


	if (f < columns)
	{
		float delay = ((float)tx_delay[f * pixels + pix] + (float)rx_delay[(nrx % channels) * pixels + pix]) / c;
		float p = delay * sampling_frequency;
		int pixel_pos = round(p);

		//data1 = rx_apod[(nrx % channels) + (pix * channels)] * data[((nrx % channels) * samples + pixel_pos - 1) * columns + f];
	}
}

__global__ void add_columns_matrix(float* data, int columns, int pixels, float* out_data)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < pixels)
	{
		for (int f = 0; f < columns; f++)
			out_data[x] += data[x * columns + f];
	}
}

__global__ void reshape_columnwise(int col, int row, float* beamformed_data_reshaped, float* d_bfHR)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	int i = x / row;
	int j = x % row;

	if (x < col * row)
	{
		beamformed_data_reshaped[j * col + i] = d_bfHR[x];

	}
}

__global__ void parallel_try(float* tx_delay, float* rx_delay, float sampling_frequency, float c, int samples,
	int channels, int columns, float* rx_apod, int pixels, float* data, float* beamformed_data)
{
	//__shared__ double* beamformed_data_1;
	int pix = blockDim.x * blockIdx.x + threadIdx.x;
	int nrx = blockDim.y * blockIdx.y + threadIdx.y;
	//int f = x / pixels; int pix = x % pixels; //int nrx = x % 128;
	int f = blockDim.z * blockIdx.z + threadIdx.z;
	//int nrx = x / pixels; //channels
	//int pix = x % pixels; //pixels
	//int f = x / (pixels * channels);

	if (f < columns && pix < pixels && nrx < channels)
	{
		//for (int nrx = 0; nrx < channels; nrx++)
		//{

		float delay = ((float)tx_delay[f * pixels + pix] + (float)rx_delay[(nrx)*pixels + pix]) / c;
		float p = delay * sampling_frequency;
		int pixel_pos = round(p);

		if ((0 < pixel_pos) && (pixel_pos < samples))
		{
			//double ans= beamformed_data[pix] + rx_apod[channels * nrx + pix] * data[nrx * 2600 + pixel_pos];
			beamformed_data[pix * columns + f] += rx_apod[nrx + (pix * channels)] * data[(nrx * samples + pixel_pos - 1) * columns + f];

		}
		//}
	}
}

void read_csv_mat(float* data, char* filename, int col1)
{
	char buffer[6240];  //6240
	char* token;

	int i = 0, j = 0;
	FILE* file;
	file = fopen(filename, "r");
	if (file == NULL)
	{
		// printf("Can't open the file");
	}
	else
	{
		while (fgets(buffer, sizeof(buffer), file) != 0)            // end-of-file indicator
		{
			token = strtok(buffer, ",");
			j = 0;
			while (token != NULL)
			{
				data[i * col1 + j] = atof(token);     //converts the string argument str to float
				token = strtok(NULL, ",");
				j++;
			}

			i++;
		}
		fclose(file);
		// printf("Complete reading from file %s\n", filename);
	}
}

void read_csv_array(float* data, char* filename)
{
	char buffer[6240];  //6240
	char* token;

	int i = 0;// , j = 0;
	FILE* file;
	file = fopen(filename, "r");
	if (file == NULL)
	{
		// printf("Can't open the file");
	}
	else
	{
		while (fgets(buffer, sizeof(buffer), file) != 0)            // end-of-file indicator
		{
			token = strtok(buffer, ",");
			//j = 0;
			while (token != NULL)
			{
				data[i] = atof(token);     //converts the string argument str to float
				token = strtok(NULL, ",");
				//j++;
			}

			i++;
		}
		fclose(file);
		// printf("Complete reading from file %s\n", filename);
	}
}

void read_csv_mat(long double* data, char* filename, int col1)
{
	char buffer[6240];  //6240
	char* token;

	int i = 0, j = 0;
	FILE* file;
	file = fopen(filename, "r");
	if (file == NULL)
	{
		// printf("Can't open the file");
	}
	else
	{
		while (fgets(buffer, sizeof(buffer), file) != 0)            // end-of-file indicator
		{
			token = strtok(buffer, ",");
			j = 0;
			while (token != NULL)
			{
				data[i * col1 + j] = atof(token);     //converts the string argument str to float
				token = strtok(NULL, ",");
				j++;
			}

			i++;
		}
		fclose(file);
		// printf("Complete reading from file %s\n", filename);
	}
}

void csv_write_mat(long double* a, char* filename, int row1, int col1)		//writes data to memory
{
	FILE* fp;
	int i;

	fp = fopen(filename, "w+");

	for (i = 0; i < row1; ++i)
	{
		for (int j = 0; j < col1; j++)
		{
			if (j == col1 - 1)					//for the last value in the column "," is not appended
			{									//matrix dimension error can occur with the presence of extra comma at last of the column
				fprintf(fp, "%g", a[i * col1 + j]);
			}
			else
				fprintf(fp, "%g,", a[i * col1 + j]);
		}



		fprintf(fp, "\n");
	}


	fclose(fp);
	// printf("\n %s file is created\n", filename);
}

void csv_write_mat(double* a, char* filename, int row1, int col1)		//writes data to memory
{
	FILE* fp;
	int i;

	fp = fopen(filename, "w+");

	for (i = 0; i < row1; ++i)
	{
		for (int j = 0; j < col1; j++)
		{
			if (j == col1 - 1)					//for the last value in the column "," is not appended
			{									//matrix dimension error can occur with the presence of extra comma at last of the column
				fprintf(fp, "%g", a[i * col1 + j]);
			}
			else
				fprintf(fp, "%g,", a[i * col1 + j]);
		}



		fprintf(fp, "\n");
	}


	fclose(fp);
	printf("\n %s file is created\n", filename);
}

void csv_write_mat(float* a, char* filename, int row1, int col1)	//for writing integer data "FUNCTION OVERLOADING"
{
	FILE* fp;
	int i;

	fp = fopen(filename, "w+");

	for (i = 0; i < row1; ++i)
	{
		for (int j = 0; j < col1; j++)
		{
			if (j == col1 - 1)
			{
				fprintf(fp, "%f", a[i * col1 + j]);
			}
			else

				fprintf(fp, "%f,", a[i * col1 + j]);
		}



		fprintf(fp, "\n");
	}


	fclose(fp);
	// printf("\n %s file is created\n", filename);
}

//__host__ => to execute the function in the host
//__device__ => to execute the function in the device(GPU)
//__device__ => to execute the function in the device(GPU)
//__host__ __device__ =>executes in both host and device

__host__ __device__ float max_val(float* data, int size1)	//To find max value from an array
{
	float max = 0;
	float temp;
	for (int i = 0; i < size1; i++)
	{
		if (data[i] > max)
		{
			temp = data[i];
			max = temp;
		}
	}
	return max;
}

__host__ __device__ double max_val(double* data, int size1)	//To find max value from an array
{
	double max = data[0];
	double temp;
	for (int i = 0; i < size1; i++)
	{
		if (data[i] > max)
		{
			temp = data[i];
			max = temp;
		}
	}
	return max;
}

__host__ __device__ long double max_val(long double* data, int size1)	//To find max value from an array
{
	long double max = data[0];
	long double temp;
	for (int i = 0; i < size1; i++)
	{
		if (data[i] > max)
		{
			temp = data[i];
			max = temp;
		}
	}
	return max;
}

__host__ __device__ int index(float* data, float value, int size1)		//to find the index of a particular value in the array
{
	int ind = 0;
	for (int i = 0; i < size1; i++)
	{
		if (data[i] == value)
		{
			ind = i;
			break;
		}
	}
	return ind;
}

__host__ __device__ float element_add(float* data, int size1)		//element wise addition of array values
{
	float value = 0;
	for (int i = 0; i < size1; i++)
	{
		value = value + data[i];
	}
	return value;
}

__host__ __device__ void matrix_subset(float* mat, int row1, int col1, int c1, int c2, int r1, int r2, float* mat_out)
{
	for (int idy = 0; idy < (r2 - r1) + 1; idy++)			//matrix sub set generation from a large matrix (ref:"device_func.h")
	{
		for (int idx = 0; idx < ((c2 - c1) + 1); idx++)
		{
			int thread_id = idy * ((c2 - c1) + 1) + idx;
			int thread_id1 = (idy + r1) * col1 + (idx + c1);
			mat_out[thread_id] = mat[thread_id1];
		}
	}
}

__device__ __host__ void matrix_sub(float* mat1, float d0, int row1, float* out)		//subtract a value from the elements of an array
{
	for (int idx = 0; idx < row1; idx++)
	{
		out[idx] = mat1[idx] - d0;
	}
}

__device__ __host__ void element_square_h(float* mat1, int size, float* matout)
{
	for (int idx = 0; idx < size; idx++)
	{
		matout[idx] = mat1[idx] * mat1[idx];
	}
}

__device__ __host__ void element_mult_h(float* mat1, float* mat2, int size, float* matout)
{
	for (int idx = 0; idx < size; idx++)
	{
		matout[idx] = mat1[idx] * mat2[idx];
	}
}

__host__ __device__ float one_skip_add(float* data, int end, int ind)		//element wise addition of array values
{
	float value = 0;
	for (int i = ind; i < end; i++)
	{
		value = value + data[i];
	}
	return value;
}

__device__ __host__ void matrix_mul_h(float* mat1, float val, int size, float* matout)
{
	for (int idx = 0; idx < size; idx++)
	{
		matout[idx] = mat1[idx] * val;
	}
}

__global__ void BPfilter1SharedMem(float* in, float* filt_coeff, int pixels, float* y1) {

	const int TILE_SIZE = 4;
	int MASK_WIDTH = 364;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float N_s[TILE_SIZE];
	N_s[threadIdx.x] = in[x];
	__syncthreads();

	int PtileStartPt = blockIdx.x * blockDim.x;
	int NtileStartPt = (blockIdx.x + 1) * blockDim.x;
	int n_start_pt = x - (MASK_WIDTH / 2);

	float temp = 0;

	for (int j = 0; j < MASK_WIDTH; j++) {
		int N_index = n_start_pt + j;

		if (N_index >= 0 && N_index < pixels) {
			if ((N_index >= PtileStartPt) && (N_index < NtileStartPt)) {
				temp += N_s[threadIdx.x + j - (MASK_WIDTH / 2)] * filt_coeff[j];
			}
			else {
				temp += in[N_index] * filt_coeff[j];
			}
		}
	}
	y1[x] = temp;
}

__global__ void zeros(float* ap_dis, int row1)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	//int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < row1)
	{
		ap_dis[x] = 0;
	}

}

__global__ void zeros(double* ap_dis, int row1)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	//int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < row1)
	{
		ap_dis[x] = 0;
	}

}

__global__ void zeros(long double* ap_dis, int row1)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	//int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < row1)
	{
		ap_dis[x] = 0;
	}

}

__global__ void isnan_test(float* data, int col1, int row1)
{


	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	while (idx < col1) {
		for (int i = 0; i < row1; i++)
		{
			if (isnan(data[(i * col1) + idx]) == 1)
				data[(i * col1) + idx] = 0;
		}

		idx += gridDim.x + blockDim.x;
	}
}

__global__ void isnan_test_array(float* data, int size)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < size)
	{
		if (isnan(data[idx]) == 1)
		{
			data[idx] = 0;
		}
		else
		{
			data[idx] = data[idx];
		}

	}


}

__global__ void down_sampling(float* down_data, float* data, int down_size, int down_val, int col)	//device function for downsampling
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;							//down_size=no.of rows after downsampling
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idy < down_size && idx < col)
	{
		down_data[idy * col + idx] = data[down_val * idy * col + idx];	//down_val=down sampling factor
	}
}

__global__ void down_col(float* down_data, float* data, int down_col_size, int down_val, int col_size, int row)	//device function for downsampling
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;							//down_size=no.of rows after downsampling
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idy < row && idx < down_col_size)
	{
		down_data[idy * down_col_size + idx] = data[idy * col_size + idx * down_val];	//down_val=down sampling factor
	}
}

__global__ void element_division(float* mat_in, float value, int size, float* mat_out)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				   //max=final value of the array
	if (i < size)
	{
		mat_out[i] = mat_in[i] / value;
	}
}

__global__ void element_division(long double* mat_in, float value, int size, long double* mat_out)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				   //max=final value of the array
	if (i < size)
	{
		mat_out[i] = mat_in[i] / value;
	}
}

__global__ void range(int* out_data, int min, int arr_size, int inc)	//creates an array of a range of values
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				//max=final value of the array
	if (i < arr_size)																//arr_size==array size
	{																			//inc=increment needed
		out_data[i] = min + (i * inc);
	}
}

__global__ void range(float* out_data, float min, int arr_size, float inc)	//creates an array of a range of values
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				//max=final value of the array
	if (i < arr_size)																//arr_size==array size
	{																			//inc=increment needed
		out_data[i] = min + (i * inc);
	}
}

__global__ void range(double* out_data, double min, int arr_size, double inc)	//creates an array of a range of values
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				//max=final value of the array
	if (i < arr_size)																//arr_size==array size
	{																			//inc=increment needed
		out_data[i] = min + (i * inc);
	}
}

__global__ void range(long double* out_data, long double min, int arr_size, long double inc)	//creates an array of a range of values
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				//max=final value of the array
	if (i < arr_size)																//arr_size==array size
	{																			//inc=increment needed
		out_data[i] = min + (i * inc);
	}
}

__global__ void mat2D_abs(int* data, int m, int n, int* out_data)	//to find the absolute positive value of each elements in a matrix

{
	int col1 = blockDim.x * blockIdx.x + threadIdx.x;			//m and n are number of rows and colums respectively
	int row1 = blockDim.y * blockIdx.y + threadIdx.y;

	if (row1 < m && col1 < n)
	{
		int thread_id = row1 * n + col1;
		if (data[thread_id] < 0)
		{
			out_data[thread_id] = -1 * data[thread_id];		//negative values are converted to positive values
		}
		else
		{
			out_data[thread_id] = data[thread_id];
		}
	}
}

__global__ void mat2D_abs(float* data, int m, int n, float* out_data)	//to find the absolute positive value of each elements in a matrix

{
	int col1 = blockDim.x * blockIdx.x + threadIdx.x;			//m and n are number of rows and colums respectively
	int row1 = blockDim.y * blockIdx.y + threadIdx.y;

	if (row1 < m && col1 < n)
	{
		int thread_id = row1 * n + col1;
		if (data[thread_id] < 0)
		{
			out_data[thread_id] = -1 * data[thread_id];		//negative values are converted to positive values
		}
		else
		{
			out_data[thread_id] = data[thread_id];
		}
	}
}

__global__ void mat_sub(float* mat1, float d0, int row1, float* out)	//to subtract a specific value from each element in the array
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < row1)
	{
		out[idx] = mat1[idx] - d0;		//d0=value to be subtracted
	}
}

__global__ void mat_subset(float* mat, int row1, int col1, int c1, int c2, int r1, int r2, float* mat_out)	//to take a matrix subset from a large matrix
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	//row1 and col1 are size of large matrix
	int idy = blockIdx.y * blockDim.y + threadIdx.y;	//(r1,c1) (r2,c2)=min and max cordinates of the sub matrix
	if (idx < ((c2 - c1) + 1) && idy < ((r2 - r1) + 1))
	{
		int thread_id = idy * ((c2 - c1) + 1) + idx;
		int thread_id1 = (idy + r1) * col1 + (idx + c1);
		mat_out[thread_id] = mat[thread_id1];
	}
}

__global__ void mat_subset(int* mat, int row1, int col1, int c1, int c2, int r1, int r2, int* mat_out)	//to take a matrix subset from a large matrix
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	//row1 and col1 are size of large matrix
	int idy = blockIdx.y * blockDim.y + threadIdx.y;	//(r1,c1) (r2,c2)=min and max cordinates of the sub matrix
	if (idx < ((c2 - c1) + 1) && idy < ((r2 - r1) + 1))
	{
		int thread_id = idy * ((c2 - c1) + 1) + idx;
		int thread_id1 = (idy + r1) * col1 + (idx + c1);
		mat_out[thread_id] = mat[thread_id1];
	}
}

__global__ void element_square(float* mat, int size, float* out)	//to square each contents of a array
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		out[idx] = mat[idx] * mat[idx];
	}
}

__global__ void element_mul(float* mat1, float* mat2, int size, float* out)	//element wise multiplication of 2 arrays
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		out[idx] = mat1[idx] * mat2[idx];
	}
}

__global__ void mat_add(float* mat1, float* mat2, int row1, int col1)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx < col1 && idy < row1)
	{
		mat2[idy * col1 + idx] = mat1[idy * col1 + idx] + mat2[idy * col1 + idx];
	}
}

__global__ void array_add(double* mat1, double* mat2, int row1)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < row1)
	{
		mat2[idx] = mat1[idx] + mat2[idx];
	}
}

__global__ void mat_subset_1D(int* mat, int size, int first, int last, int* mat_out)	//to take a matrix subset from a large matrix
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	//row1 and col1 are size of large matrix
	//int idy = blockIdx.y * blockDim.y + threadIdx.y;	//(r1,c1) (r2,c2)=min and max cordinates of the sub matrix
	if (idx < ((last - first) + 1))
	{
		int thread_id = idx;
		int thread_id1 = idx + first;
		mat_out[thread_id] = mat[thread_id1];
	}
}

__global__ void mat_subset_1D(float* mat, int size, int first, int last, float* mat_out)	//to take a matrix subset from a large matrix
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	//row1 and col1 are size of large matrix
	//int idy = blockIdx.y * blockDim.y + threadIdx.y;	//(r1,c1) (r2,c2)=min and max cordinates of the sub matrix
	if (idx < ((last - first) + 1))
	{
		int thread_id = idx;
		int thread_id1 = (idx + first);
		mat_out[thread_id] = mat[thread_id1];
	}
}

__global__ void matrix_mult(float* mat1, float val, int size, float* out)	//element wise multiplication of 2 arrays
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		out[idx] = mat1[idx] * val;
	}
}

__global__ void matrix_mult1(float* mat1, float val, int size, float* out)	//element wise multiplication of 2 arrays
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		out[idx] = mat1[idx] * val;
	}
}

__global__ void upsamp_append(float* mat_out, float* mat_in, int first_row, int samp_fact, int row1, int col1)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idy < row1 && idx < col1)
	{
		mat_out[(samp_fact * idy + first_row) * col1 + idx] = mat_in[idy * col1 + idx];
	}
}

__global__ void mat_transpose(float* mat_in, float* mat_out, int row_org, int col_org)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx < col_org && idy < row_org)
	{
		mat_out[idx * row_org + idy] = mat_in[idy * col_org + idx];
	}
}

float** reshapeto2d(float* inArray, int rows, int cols) {
	// Conevert the single array of size rows*col into 2 dimensional array of size rows and col
	// Declare new Array
	float** array2D = (float**)malloc(sizeof(float) * cols);
	for (int i= 0; i < cols; i++) {
		array2D[i] = (float*)malloc(rows);
	}

	// Fill the values
	for (int i = 0; i < cols; i++) {
		for (int j = 0; j < rows; j++) {
			array2D[i][j] = inArray[i * rows + j];
		}
	}

	return array2D;
}


// Extern void bModeGenerationinCUDA(float* t, float* v, int tno)
cv::Mat bModeGen()
{  
	// perform b-mode generation here using cuda
	
	const int TILE_SIZE = 4;
	int MASK_WIDTH = 364;
	//// Computer (NIVIDIA) parametrs
	int num_threads = 1024;

	/// Apodization parameters
	float rx_f_number = 2.0;

	/////// Ultrasound scanner parametrs
	//float depth = 49.28;      // Depth of imaging in mm
	int samples = 2008;                    // # of samples in depth direction
	int N_elements = 64;         // # of transducer elements
	float sampling_frequency = 32e6;   // sampling frequency
	float c = 1540.0;      // speed of sound [m/s]	
	int N_active = 8;                         // Active transmit elmeents
	float pitch = 0.3 / 1000;           // spacing between the elements
	float aper_len = (N_elements - 1) * pitch * 1000;			 //aperture foot print 
	float zd = pitch * N_active / (float)2;            // virtual src distance from transducer array 
	float sample_spacing = c / sampling_frequency / (float)2;
	float del_convert = sampling_frequency / c;  // used in delay calculation

	int channels = 64;	              // number of A-lines data used for beamforming

	//// Beamforming "Grid" parameters
	int Nx = 256;      // 256 Lateral spacing
	int Nz = 1024;            //1024 Axial spacing
	int pixels = Nz * Nx;
	int pix_cha = pixels * channels;     // Nz*Nx*128 This array size is used for Apodization
	int num_frames = 57;   // number of low resolution images
	int skip_frames = 1;  // 

	// Post processing parameters.
	//int dBvalue = 60;

	float* filt_coeff = new float[MASK_WIDTH];
	char filename1[200];
	sprintf(filename1, "C:/Users/navee/Documents/projects/USI_processing/BmodeinCUDA/copy/CudaRuntime/b_10M.csv"); 
	read_csv_array(filt_coeff, filename1);    // csv file read

	float* d_filt_coeff = 0;
	hipMalloc((void**)&d_filt_coeff, sizeof(float) * MASK_WIDTH);
	hipMemcpy(d_filt_coeff, filt_coeff, sizeof(float) * MASK_WIDTH, hipMemcpyHostToDevice);


	////////////// z value////////////////////
	float dz = sample_spacing * samples / Nz;  // depth / (Nz - 1) / 1000;   // spacing in axial (z) direction in mm;
	float* d_z_axis = 0;
	hipMalloc((void**)&d_z_axis, Nz * sizeof(float));
	range << <Nz / num_threads + 1, num_threads >> > (d_z_axis, 0, Nz, dz);
	hipGetLastError();
	hipDeviceSynchronize();

	//////////////////////////////// x value////////////////////////////////
	float dx = aper_len / (Nx - 1);
	float* d_x_axis = 0;
	hipMalloc((void**)&d_x_axis, Nx * sizeof(float));    // 167.939 us
	range << <Nx / num_threads + 1, num_threads >> > (d_x_axis, (-aper_len / 2000), Nx, dx / 1000);
	hipGetLastError();
	hipDeviceSynchronize();

	//////////////// Probe geometry, this info can be taken from transducer file ////////////////////
	float* d_probe = 0;
	hipMalloc((void**)&d_probe, N_elements * sizeof(float));
	range << <1, N_elements >> > (d_probe, (-aper_len / 2000), N_elements, pitch);
	hipGetLastError();
	hipDeviceSynchronize();

	/////////////////rx aperture calculation using Fnumber///////////////////////////////
	// rx_aper=rfsca.z/rf_number
	float* d_rx_aperture = 0;
	hipMalloc((void**)&d_rx_aperture, Nz * sizeof(float));
	element_division << <Nz / num_threads + 1, num_threads >> > (d_z_axis, rx_f_number, Nz, d_rx_aperture);
	hipGetLastError();
	hipDeviceSynchronize();

	////////////////////////rx aerture distance////////
	float* d_rx_ap_distance = 0;
	hipMalloc((void**)&d_rx_ap_distance, channels * Nx * sizeof(float));
	aperture_distance << <Nx * channels / num_threads + 1, num_threads >> > (d_x_axis, d_probe, Nx, channels, d_rx_ap_distance);
	hipGetLastError();
	hipDeviceSynchronize();

	///////////////////apodization/////////////////
	float* d_rx_apod = 0;
	hipMalloc((void**)&d_rx_apod, sizeof(float) * Nz * channels * Nx);
	apodization << <pixels * channels / num_threads + 1, num_threads >> > (d_rx_ap_distance, d_rx_aperture, Nz, Nx, channels, pixels, d_rx_apod);
	hipGetLastError();
	hipDeviceSynchronize();

	hipFree(d_rx_aperture);
	hipFree(d_rx_ap_distance);

	/////////////////// calculate central positions transmit subaperture ////////////////////
	float* d_cen_pos = 0;
	hipMalloc((void**)&d_cen_pos, num_frames * sizeof(float));
	Tx_cen_pos << < 1, num_frames >> > (d_cen_pos, N_elements, N_active, pitch, skip_frames, num_frames, d_probe);

	/////////////receive delay calculation /////////////////////////////////////////////
	float* d_rx_delay = 0;
	hipMalloc((void**)&d_rx_delay, pix_cha * sizeof(float));
	receive_delay << < pixels * channels / num_threads + 1, num_threads >> > (d_probe, d_x_axis, d_z_axis, channels, Nx, Nz, del_convert, d_rx_delay);
	hipGetLastError();
	hipDeviceSynchronize();

	////////////Initialize d_bfHR to store final high-resolution beamformed image /////////////////////////////
	float* d_bfHR = 0;
	hipMalloc((void**)&d_bfHR, pixels * sizeof(float));
	//zeros << <pixels / num_threads + 1, num_threads >> > (d_bfHR, pixels);  
	hipMemset(d_bfHR, 0, pixels * sizeof(float));

	/////////////////// Transmit delay calculation ////////////////////
	float* d_tx_delay = 0;
	hipMalloc((void**)&d_tx_delay, pixels * num_frames * sizeof(float));
	//transmit delay for all frames,   
	transmit_delay << < pixels * num_frames / num_threads + 1, num_threads >> > (d_x_axis, d_z_axis, d_cen_pos, zd, Nx, Nz, del_convert, num_frames, d_tx_delay);
	hipGetLastError();
	hipDeviceSynchronize();


	////////////Free cuda memory (one time use) ///////////////////////////
	hipFree(d_probe);
	hipFree(d_x_axis);
	hipFree(d_z_axis);
	hipFree(d_cen_pos);

	float* data = new float[samples * channels];

	float* d_data = 0;
	hipMalloc((void**)&d_data, sizeof(float) * samples * channels);


	for (int f = 0; f < num_frames; f++)
	{
		char filename[200];
		sprintf(filename, "C:/Users/navee/Documents/projects/USI_processing/BmodeinCUDA/copy/CudaRuntime/inputs/raw_rf_dbsat_Ptsca_arr_%d.csv", f); //all the LR inputs are arranged in a single file

		read_csv_mat(data, filename, 1);    // csv file read

		clock_t begin = clock();   // clock intiated

		hipMemcpy(d_data, data, sizeof(float) * samples * channels, hipMemcpyHostToDevice);

		beamformingLR3 << <(pixels / 256) * channels, 256 >> > (d_bfHR, d_tx_delay, d_rx_delay, d_data, d_rx_apod, samples, pixels, f, num_frames, channels);
		hipGetLastError();
		hipDeviceSynchronize();

		clock_t end = clock();
		float elapsed_secs = float(end - begin) / CLOCKS_PER_SEC;
		printf("Time for beamforming in ms: %f\n", elapsed_secs * 1000);

	}

	//// check for nan values,
	isnan_test_array << <pixels / num_threads + 1, num_threads >> > (d_bfHR, pixels);
	hipGetLastError();
	hipDeviceSynchronize();

	float* d_bfHRBP = 0;  // variable to store beamformed high-resolution bandpass filtered data
	hipMalloc((void**)&d_bfHRBP, sizeof(float) * pixels);

	//////////// Bandpass filtering using shared memory /////////////////////
	BPfilter1SharedMem << <(pixels + TILE_SIZE - 1) / TILE_SIZE, TILE_SIZE >> > (d_bfHR, d_filt_coeff, pixels, d_bfHRBP);
	hipGetLastError();
	hipDeviceSynchronize();

	//////////////// reshape of the beamformed data ///////////////
	float* dev_beamformed_data1 = 0;
	hipMalloc((void**)&dev_beamformed_data1, pixels * sizeof(float));   //234.130 us
	reshape_columnwise << <pixels / num_threads + 1, num_threads >> > (Nx, Nz, dev_beamformed_data1, d_bfHRBP);  //48.864 us
	hipGetLastError();
	hipDeviceSynchronize();

	float* env = new float[pixels];
	hipMemcpy(env, dev_beamformed_data1, Nz * Nx * sizeof(float), hipMemcpyDeviceToHost);
	char* fileout = "C:/Users/navee/Documents/projects/USI_processing/BmodeinCUDA/copy/CudaRuntime/outputs/b_mode.csv";
	csv_write_mat(env, fileout, Nz, Nx);

	//float** bmode2d = reshapeto2d(env, Nz, Nx); // gives error
	cv::Mat bmodMat = cv::Mat::zeros(Nz, Nx, CV_32FC1);
	// cv::Mat bmodMat = converttoMat(env, Nz, Nx);

	//////////////// Free cuda memory (that will be used again) ///////////////
	hipFree(d_data);
	hipFree(d_bfHR);
	hipFree(d_tx_delay);
	hipFree(d_rx_delay);
	hipFree(d_rx_apod);
	hipFree(dev_beamformed_data1);
	hipFree(d_bfHRBP);

	return bmodMat;
}

//extern void imageGenProcessinCUDA() {
//
//	// B-Mode image generation code fully in CUDA
//	double minP, maxP;
//	cv::Mat bmodMat = bModeGen();
//	cv::imwrite("C:/Users/navee/Documents/projects/USI_processing/BmodeinCUDA/copy/CudaRuntime/outputs/bmodMat.png", bmodMat);
//	std::cout << "size of bmodMat : " << bmodMat.rows << " , " << bmodMat.cols << std::endl;
//
//	cv::Mat envolepMat = hilbertTrans4(bmodMat, 1.0);
//	cv::imwrite("C:/Users/navee/Documents/projects/USI_processing/BmodeinCUDA/copy/CudaRuntime/outputs/envolepMat.png", envolepMat);
//	cv::minMaxIdx(envolepMat, &minP, &maxP);
//	std::cout << "range of bmodeMat before log compression: " << minP << " ->" << maxP << std::endl;
//	std::cout << "size of envolepMat : " << envolepMat.rows << " , " << envolepMat.cols << std::endl;
//
//	// perform image processing 
//	cv::Mat deSpeckledimg;
//	DeSpeckle deNoiseImg(envolepMat);
//	deNoiseImg.applySRAD(envolepMat, deSpeckledimg, 1, 10, 0.25, false, false);
//	cv::imwrite("C:/Users/navee/Documents/projects/USI_processing/BmodeinCUDA/copy/CudaRuntime/outputs/dspeckledimg.png", deSpeckledimg);
//	cv::minMaxIdx(deSpeckledimg, &minP, &maxP);
//	std::cout << "range of deSpeckledimg : " << minP << " ->" << maxP << std::endl;
//
//	// // log compression
//	//cv::Mat logcMat = logTransform(envolepMat);
//	//cv::imwrite("./outputs/logcMat.png", logcMat);
//	//cv::minMaxIdx(logcMat, &minP, &maxP);
//	//std::cout << "range of bmodeMat after log compression1: " << minP << " ->" << maxP << std::endl;
//
//	cv::Mat rangedMat = dynamicRangeAdjust(deSpeckledimg, 100.0);
//	cv::imwrite("C:/Users/navee/Documents/projects/USI_processing/BmodeinCUDA/copy/CudaRuntime/outputs/rangedMat.png", rangedMat);
//	cv::minMaxIdx(rangedMat, &minP, &maxP);
//	std::cout << "range of bmodeMat after range adjust: " << minP << " ->" << maxP << std::endl;      
//
//	cv::Mat displayMat = displayRangeAdjust(rangedMat);
//	cv::minMaxIdx(displayMat, &minP, &maxP);
//	std::cout << "range of bmodeMat after disply range adjust: " << minP << " ->" << maxP << std::endl;
//	cv::imwrite("C:/Users/navee/Documents/projects/USI_processing/BmodeinCUDA/copy/CudaRuntime/outputs/b_mode_w_speckle_Red.png", displayMat);
//	//cv::imshow("B-mode image", displayMat);
//	//cv::waitKey();
//}
