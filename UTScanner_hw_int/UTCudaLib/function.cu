#include "hip/hip_runtime.h"

// // It reconstructs LR images separately and then add them to get HR image.
// Delay values and apodization are reshaped into [1 1024, 1 1024, ... pixels] ...
#include <stdlib.h>
#include <stdio.h>
#include <iomanip>
#include <ctime>
#include <math.h>
#include <string.h>
#include <time.h>
//#define PI 3.14159
//#define TILE_SIZE 4
//#define MASK_WIDTH 364

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>      /// Add "cufft.lib" in the linker input to use cufft. 
#include "hip/hip_runtime.h"
#include <windows.h>
#include ""
#include "device_func1.h"
#include "host_func1.h"
#include "beamforming_func1.h"

// include OpenCV Header
#include <opencv2/opencv.hpp>
#include <opencv2/highgui.hpp>
#include "mat_operations.h"
// #include "CyAPI.h"
// #include "UTCudaLib.h"

void wait(unsigned timeout)
{
	timeout += std::clock();
	while (std::clock() < timeout) continue;
}

//void write_rows(CCyControlEndPoint* ept, unsigned char* ptr, unsigned int numRows)
//{
//	int len = numRows * 16; // each row is 16 bytes to send
//	int sent = 0;
//	unsigned char* tPtr = ptr;
//	while (sent < len) {
//		LONG buflen = len - sent;
//		if (buflen > 192) buflen = 192;
//		//for (int i = 0; i < buflen; i += 16) {
//		//	for (int j = 0; j < 16; j++) {
//		//		printf("%02X\t ", ptr[i + j]);
//		//	}
//		//	printf("\n");
//		//}
//		ept->XferData(tPtr, buflen);
//		tPtr += buflen;
//		sent += buflen;
//	}
//	//printf("Sent %d bytes to EPT\n", sent);
//}
//
//bool read_chunk(CCyBulkEndPoint* ept_in, unsigned char* recvBuf, LONG& length)
//{
//	bool result;
//	LONG intlen = length;
//	result = ept_in->XferData(recvBuf, intlen, NULL, true);
//	//if (result) {
//	//	printf("Received data ------------- : %d\n", intlen);
//	//}
//	//else {
//	//	printf("***   ERROR receiving data - expected %d, got %d\n", length, intlen);
//	//}
//	ept_in->Abort();
//	ept_in->Reset();
//	return result;
//}
//
//int insert_row(unsigned char* buf, int row, short addr, int data)
//{
//	int o = row * 16;
//	buf[o] = 0xff; buf[o + 1] = 0xaa; buf[o + 2] = 0x01; buf[o + 3] = 0x07;
//	buf[o + 4] = 0x00; buf[o + 5] = 0x00; buf[o + 6] = 0x00; buf[o + 7] = 0x01;
//	buf[o + 8] = (addr & 0xff); buf[o + 9] = (addr & 0xff00) >> 8;
//	buf[o + 10] = (data & 0xff); buf[o + 11] = (data & 0xff00) >> 8;
//	buf[o + 12] = (data & 0xff0000) >> 16; buf[o + 13] = (data & 0xff000000) >> 24;
//	buf[o + 14] = 0x00; buf[o + 15] = 0x00;
//	return row + 1;
//}
////
////void saveToFile(int count, LONG& rxlen, unsigned char recvbuf[2048 * 64 * 2])
////{
////	char fname[20];
////	sprintf_s(fname, "c:\\test%02d.csv", count);
////	ofstream ofile(fname);
////
////	short* rxdata = (short*)(recvbuf);
////	for (int i = 0; i < rxlen / 2; i++) {
////		if (rxdata[i] >= 512) rxdata[i] -= 1024;
////	}
////
////	for (int i = 0; i < rxlen / 2; i += 64) {
////		for (int j = 0; j < 64; j++) {
////			//printf("%d, ", rxdata[i+j]);
////			ofile << rxdata[i + j] << ", ";
////		}
////		//printf("\n");
////		ofile << std::endl;
////	}
////	printf("Wrote data to %s\n", fname);
////}

// Global variable to store the full image.  Cannot be declared local as memory alloc may fail due to large size.
float rximg[64 * 2040];


/////BMode functions/////////////
__global__ void log_conv(float* data_hilbert, float* env, hipfftComplex* d_input_value, int row_org, int col)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int index = j * col + i;

	if (i < col && j < row_org)
	{
		// divide by 'size' is to ensure that the FFT equation holds good.
		//real_d_input_value = (d_input_value[i].x / (float)size);   // Extract real value
		float real = d_input_value[index].x / ((float)row_org * (float)col);
		float img = d_input_value[index].y / ((float)row_org * (float)col);
		data_hilbert[index] = fabs(sqrt((real * real) + (img * img))); // Absolute value

		env[index] = 20 * log10(data_hilbert[index]);     // log compression

	}
}

__global__ void db_conv(float* env, float max, int size, int dBvalue)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < size)
	{

		env[i] = env[i] - max;     //env_dB = env_dB - max(max(env_dB));                                    // Normalization
		env[i] = (float)127.0 * (env[i] + (float)dBvalue) / (float)dBvalue;              // dB conversion
	}
}

__global__ void point_wise_product(hipfftComplex* a, int* b, int row_org, int col)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if ((i < col) && (j < row_org))
	{
		a[j * col + i].x = a[j * col + i].x * b[j];
		a[j * col + i].y = a[j * col + i].y * b[j];
	}
}

__global__ void real2complex(float* f, hipfftComplex* fc, int N1, int N2)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int index = j * N2 + i;

	if (i < N2 && j < N1)
	{
		fc[index].x = f[index];
		fc[index].y = 0.0f;

	}

}

/////////////////////////////////////////////////////////////////////////
void Generate_Pointwise_Coeff(int* pointwise_coeff, int size)
{
	if ((size % 2) == 0)
	{
		pointwise_coeff[0] = 1;
		pointwise_coeff[size / 2] = 1;

		for (unsigned int i = 1; i < size / 2; i++)
		{
			pointwise_coeff[i] = 2;
		}
		for (unsigned int i = (size / 2) + 1; i < size; i++)
		{
			pointwise_coeff[i] = 0;
		}
	}
	else
	{
		pointwise_coeff[0] = 1;
		//pointwise_coeff[size / 2] = 1;

		for (unsigned int i = 1; i <= size / 2; i++)
		{
			pointwise_coeff[i] = 2;
		}
		for (unsigned int i = (size / 2) + 1; i < size; i++)
		{
			pointwise_coeff[i] = 0;
		}
	}

}
/////////////////////////////////////////////////////////////

__global__ void aperture_distance(float* mat1, float* mat2, int Nx, int channels, float* mat_out)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	int i = x / channels;
	int j = x % channels;

	if (x < Nx * channels)
	{
		mat_out[i * channels + j] = fabs(mat1[i] - mat2[j]);
	}
}

//__global__ void apodization(float* distance, float* aperture, int Nz, int Nx, int channels, int pixels, float* apod)
//{
//	int x = blockDim.x * blockIdx.x + threadIdx.x;
//	int i = x / Nz;
//	int j = x % Nz;
//
//	if (x < Nz * Nx)
//	{
//		for (int k = 0; k < channels; k++)
//		{
//			bool temp = distance[i * channels + k] <= (aperture[j] / 2);
//			apod[x * channels + k] = temp * (0.5 + 0.5 * cos(2 * PI * distance[i * channels + k] / aperture[j]));
//		}
//		bool temp = distance[i * channels + k] <= (aperture[j] / 2.0);
//		apod[x] = temp * (double)(0.5 + 0.5 * cos(2 * PI * distance[i * channels + k] / aperture[j]));
//	}
//}

__global__ void apodization(float* distance, float* aperture, int Nz, int Nx, int channels, int pixels, float* apod)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int i = x / Nz;
	int ii = i % Nx;
	int j = x % Nz;
	int nrx = x / pixels;
	float PI = 3.14159;

	if (x < pixels * channels)
	{
		bool temp = distance[ii * channels + nrx] <= (aperture[j] / 2);
		apod[i * Nz + j] = temp * (0.5 + 0.5 * cos(2 * PI * distance[ii * channels + nrx] / aperture[j]));
	}
}

// receive_delay calculation
__global__ void receive_delay(float* probe_ge_x, float* x_axis1, float* z_axis1, int channels, int Nx, int Nz, float del_convert, float* rx_delay)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;

	if (x < Nx * Nz * channels)
	{
		int i = x / Nz;
		int ii = i % Nx;
		int j = x % Nz;
		int nrx = x / (Nx * Nz);
		rx_delay[i * Nz + j] = (sqrt((probe_ge_x[nrx] - x_axis1[ii]) * (probe_ge_x[nrx] - x_axis1[ii]) + ((z_axis1[j]) * (z_axis1[j])))) * del_convert;
		// 1867 - 210 = 1657
		//rx_delay[i * Nx + j] = sqrt(rc * rc + (rc + z_axis[ii]) * (rc + z_axis[ii]) - 2 * rc * (rc + z_axis[ii]) * cos(theta[nrx] - theta1[j])) * del_convert;
	}
}

__global__ void theta1(float* theta_active, float* theta, int frames, int N_active, int skip_frames)
{

	int x = threadIdx.x;
	int f = 0;
	for (int i = 1; i <= frames; i += skip_frames)
	{
		theta_active[f] = theta[i - 1];
		f++;
	}
}

// This function calculates TX central aperture position
__global__ void Tx_cen_pos(float* cen_pos, int N_elements, int N_active, float pitch, int skip_frames, int num_frames, float* probe)
{

	int x = threadIdx.x;

	if (x < num_frames)
	{
		//cen_pos[x] = pitch * ((N_active / 2) + (N_active * (x)-N_elements / 2));
		cen_pos[x] = probe[x * skip_frames + 4];
	}
}

//  transmit_delay calculation
__global__ void transmit_delay(float* x_axis1, float* z_axis1, float* k1, float zd, int Nx, int Nz, float del_convert, int num_frames, float* tx_delay)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int i = x / Nz;
	int ii = i % Nx;
	int j = x % Nz;
	int f = x / (Nx * Nz);

	if (x < Nx * Nz * num_frames)
	{
		tx_delay[i * Nz + j] = (sqrt(((k1[f] - x_axis1[ii]) * (k1[f] - x_axis1[ii])) + ((zd + z_axis1[j]) * (zd + z_axis1[j])))) * del_convert;
		// 1875-210 = 1665
		//tx_delay[i * Nx + j] = (zd + sqrt(rc * rc + (rc + z_axis[ii]) * (rc + z_axis[ii]) - 2 * rc * (rc + z_axis[ii]) * cos(theta_tx[f] - theta1[j]))) * del_convert;
		//first 256*1024 for frame 1, next 256*1024 for frame 2........
	}
}

__global__ void beamformingLR3(float* beamformed_data1, float* tx_delay, float* rx_delay, float* data, float* rx_apod, int samples, int pixels, int f, int num_frames, int channels)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	int nrx = x / pixels;   // nrx - nth A-line
	int pix = x & (pixels - 1); // x% pixels;     // pixel location

	int pixel_pos = round((float)tx_delay[f * pixels + pix] + (float)rx_delay[x]);   // delay value estimation from tx and rx delay values

	if (pixel_pos < samples)
	{
		beamformed_data1[pix] += rx_apod[x] * data[(nrx * samples + pixel_pos - 1)];   // Extract data based on the delay values and multiplying with apodization value
	}
}

__global__ void add_ele(float* data, int pixels, float* out_data)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < pixels)
	{
		out_data[x] += data[x];
	}
}

__global__ void sample1(float* tx_delay, float* rx_delay, int pixels, int channels, float c, float sampling_frequency, float* data1, float* rx_apod, float* data, int samples, int columns)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int f = blockDim.y * blockIdx.y + threadIdx.y;
	int nrx = x / pixels; //channels
	int pix = x % pixels; //pixels


	if (f < columns)
	{
		float delay = ((float)tx_delay[f * pixels + pix] + (float)rx_delay[(nrx % channels) * pixels + pix]) / c;
		float p = delay * sampling_frequency;
		int pixel_pos = round(p);

		//data1 = rx_apod[(nrx % channels) + (pix * channels)] * data[((nrx % channels) * samples + pixel_pos - 1) * columns + f];
	}
}

__global__ void add_columns_matrix(float* data, int columns, int pixels, float* out_data)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < pixels)
	{
		for (int f = 0; f < columns; f++)
			out_data[x] += data[x * columns + f];
	}
}

__global__ void reshape_columnwise(int col, int row, float* beamformed_data_reshaped, float* d_bfHR)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	int i = x / row;
	int j = x % row;

	if (x < col * row)
	{
		beamformed_data_reshaped[j * col + i] = d_bfHR[x];

	}
}

__global__ void parallel_try(float* tx_delay, float* rx_delay, float sampling_frequency, float c, int samples,
	int channels, int columns, float* rx_apod, int pixels, float* data, float* beamformed_data)
{
	//__shared__ double* beamformed_data_1;
	int pix = blockDim.x * blockIdx.x + threadIdx.x;
	int nrx = blockDim.y * blockIdx.y + threadIdx.y;
	//int f = x / pixels; int pix = x % pixels; //int nrx = x % 128;
	int f = blockDim.z * blockIdx.z + threadIdx.z;
	//int nrx = x / pixels; //channels
	//int pix = x % pixels; //pixels
	//int f = x / (pixels * channels);

	if (f < columns && pix < pixels && nrx < channels)
	{
		//for (int nrx = 0; nrx < channels; nrx++)
		//{

		float delay = ((float)tx_delay[f * pixels + pix] + (float)rx_delay[(nrx)*pixels + pix]) / c;
		float p = delay * sampling_frequency;
		int pixel_pos = round(p);

		if ((0 < pixel_pos) && (pixel_pos < samples))
		{
			//double ans= beamformed_data[pix] + rx_apod[channels * nrx + pix] * data[nrx * 2600 + pixel_pos];
			beamformed_data[pix * columns + f] += rx_apod[nrx + (pix * channels)] * data[(nrx * samples + pixel_pos - 1) * columns + f];

		}
		//}
	}
}

void read_csv_mat(float* data, char* filename, int col1)
{
	char buffer[6240];  //6240
	char* token;

	int i = 0, j = 0;
	FILE* file;
	file = fopen(filename, "r");
	if (file == NULL)
	{
		// printf("Can't open the file");
	}
	else
	{
		while (fgets(buffer, sizeof(buffer), file) != 0)            // end-of-file indicator
		{
			token = strtok(buffer, ",");
			j = 0;
			while (token != NULL)
			{
				data[i * col1 + j] = atof(token);     //converts the string argument str to float
				token = strtok(NULL, ",");
				j++;
			}

			i++;
		}
		fclose(file);
		// printf("Complete reading from file %s\n", filename);
	}
}

void read_csv_array(float* data, char* filename)
{
	char buffer[6240];  //6240
	char* token;

	int i = 0;// , j = 0;
	FILE* file;
	file = fopen(filename, "r");
	if (file == NULL)
	{
		// printf("Can't open the file");
	}
	else
	{
		while (fgets(buffer, sizeof(buffer), file) != 0)            // end-of-file indicator
		{
			token = strtok(buffer, ",");
			//j = 0;
			while (token != NULL)
			{
				data[i] = atof(token);     //converts the string argument str to float
				token = strtok(NULL, ",");
				//j++;
			}

			i++;
		}
		fclose(file);
		// printf("Complete reading from file %s\n", filename);
	}
}

void read_csv_mat(long double* data, char* filename, int col1)
{
	char buffer[6240];  //6240
	char* token;

	int i = 0, j = 0;
	FILE* file;
	file = fopen(filename, "r");
	if (file == NULL)
	{
		// printf("Can't open the file");
	}
	else
	{
		while (fgets(buffer, sizeof(buffer), file) != 0)            // end-of-file indicator
		{
			token = strtok(buffer, ",");
			j = 0;
			while (token != NULL)
			{
				data[i * col1 + j] = atof(token);     //converts the string argument str to float
				token = strtok(NULL, ",");
				j++;
			}

			i++;
		}
		fclose(file);
		// printf("Complete reading from file %s\n", filename);
	}
}

void csv_write_mat(long double* a, char* filename, int row1, int col1)		//writes data to memory
{
	FILE* fp;
	int i;

	fp = fopen(filename, "w+");

	for (i = 0; i < row1; ++i)
	{
		for (int j = 0; j < col1; j++)
		{
			if (j == col1 - 1)					//for the last value in the column "," is not appended
			{									//matrix dimension error can occur with the presence of extra comma at last of the column
				fprintf(fp, "%g", a[i * col1 + j]);
			}
			else
				fprintf(fp, "%g,", a[i * col1 + j]);
		}



		fprintf(fp, "\n");
	}


	fclose(fp);
	// printf("\n %s file is created\n", filename);
}

void csv_write_mat(double* a, char* filename, int row1, int col1)		//writes data to memory
{
	FILE* fp;
	int i;

	fp = fopen(filename, "w+");

	for (i = 0; i < row1; ++i)
	{
		for (int j = 0; j < col1; j++)
		{
			if (j == col1 - 1)					//for the last value in the column "," is not appended
			{									//matrix dimension error can occur with the presence of extra comma at last of the column
				fprintf(fp, "%g", a[i * col1 + j]);
			}
			else
				fprintf(fp, "%g,", a[i * col1 + j]);
		}



		fprintf(fp, "\n");
	}


	fclose(fp);
	printf("\n %s file is created\n", filename);
}

void csv_write_mat(float* a, char* filename, int row1, int col1)	//for writing integer data "FUNCTION OVERLOADING"
{
	FILE* fp;
	int i;

	fp = fopen(filename, "w+");

	for (i = 0; i < row1; ++i)
	{
		for (int j = 0; j < col1; j++)
		{
			if (j == col1 - 1)
			{
				fprintf(fp, "%f", a[i * col1 + j]);
			}
			else

				fprintf(fp, "%f,", a[i * col1 + j]);
		}



		fprintf(fp, "\n");
	}


	fclose(fp);
	// printf("\n %s file is created\n", filename);
}

//__host__ => to execute the function in the host
//__device__ => to execute the function in the device(GPU)
//__device__ => to execute the function in the device(GPU)
//__host__ __device__ =>executes in both host and device

__host__ __device__ float max_val(float* data, int size1)	//To find max value from an array
{
	float max = 0;
	float temp;
	for (int i = 0; i < size1; i++)
	{
		if (data[i] > max)
		{
			temp = data[i];
			max = temp;
		}
	}
	return max;
}

__host__ __device__ double max_val(double* data, int size1)	//To find max value from an array
{
	double max = data[0];
	double temp;
	for (int i = 0; i < size1; i++)
	{
		if (data[i] > max)
		{
			temp = data[i];
			max = temp;
		}
	}
	return max;
}

__host__ __device__ long double max_val(long double* data, int size1)	//To find max value from an array
{
	long double max = data[0];
	long double temp;
	for (int i = 0; i < size1; i++)
	{
		if (data[i] > max)
		{
			temp = data[i];
			max = temp;
		}
	}
	return max;
}

__host__ __device__ int index(float* data, float value, int size1)		//to find the index of a particular value in the array
{
	int ind = 0;
	for (int i = 0; i < size1; i++)
	{
		if (data[i] == value)
		{
			ind = i;
			break;
		}
	}
	return ind;
}

__host__ __device__ float element_add(float* data, int size1)		//element wise addition of array values
{
	float value = 0;
	for (int i = 0; i < size1; i++)
	{
		value = value + data[i];
	}
	return value;
}

__host__ __device__ void matrix_subset(float* mat, int row1, int col1, int c1, int c2, int r1, int r2, float* mat_out)
{
	for (int idy = 0; idy < (r2 - r1) + 1; idy++)			//matrix sub set generation from a large matrix (ref:"device_func.h")
	{
		for (int idx = 0; idx < ((c2 - c1) + 1); idx++)
		{
			int thread_id = idy * ((c2 - c1) + 1) + idx;
			int thread_id1 = (idy + r1) * col1 + (idx + c1);
			mat_out[thread_id] = mat[thread_id1];
		}
	}
}

__device__ __host__ void matrix_sub(float* mat1, float d0, int row1, float* out)		//subtract a value from the elements of an array
{
	for (int idx = 0; idx < row1; idx++)
	{
		out[idx] = mat1[idx] - d0;
	}
}

__device__ __host__ void element_square_h(float* mat1, int size, float* matout)
{
	for (int idx = 0; idx < size; idx++)
	{
		matout[idx] = mat1[idx] * mat1[idx];
	}
}

__device__ __host__ void element_mult_h(float* mat1, float* mat2, int size, float* matout)
{
	for (int idx = 0; idx < size; idx++)
	{
		matout[idx] = mat1[idx] * mat2[idx];
	}
}

__host__ __device__ float one_skip_add(float* data, int end, int ind)		//element wise addition of array values
{
	float value = 0;
	for (int i = ind; i < end; i++)
	{
		value = value + data[i];
	}
	return value;
}

__device__ __host__ void matrix_mul_h(float* mat1, float val, int size, float* matout)
{
	for (int idx = 0; idx < size; idx++)
	{
		matout[idx] = mat1[idx] * val;
	}
}

__global__ void BPfilter1SharedMem(float* in, float* filt_coeff, int pixels, float* y1) {

	const int TILE_SIZE = 4;
	int MASK_WIDTH = 364;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float N_s[TILE_SIZE];
	N_s[threadIdx.x] = in[x];
	__syncthreads();

	int PtileStartPt = blockIdx.x * blockDim.x;
	int NtileStartPt = (blockIdx.x + 1) * blockDim.x;
	int n_start_pt = x - (MASK_WIDTH / 2);

	float temp = 0;

	for (int j = 0; j < MASK_WIDTH; j++) {
		int N_index = n_start_pt + j;

		if (N_index >= 0 && N_index < pixels) {
			if ((N_index >= PtileStartPt) && (N_index < NtileStartPt)) {
				temp += N_s[threadIdx.x + j - (MASK_WIDTH / 2)] * filt_coeff[j];
			}
			else {
				temp += in[N_index] * filt_coeff[j];
			}
		}
	}
	y1[x] = temp;
}

__global__ void zeros(float* ap_dis, int row1)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	//int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < row1)
	{
		ap_dis[x] = 0;
	}

}

__global__ void zeros(double* ap_dis, int row1)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	//int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < row1)
	{
		ap_dis[x] = 0;
	}

}

__global__ void zeros(long double* ap_dis, int row1)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	//int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < row1)
	{
		ap_dis[x] = 0;
	}

}

__global__ void isnan_test(float* data, int col1, int row1)
{


	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	while (idx < col1) {
		for (int i = 0; i < row1; i++)
		{
			if (isnan(data[(i * col1) + idx]) == 1)
				data[(i * col1) + idx] = 0;
		}

		idx += gridDim.x + blockDim.x;
	}
}

__global__ void isnan_test_array(float* data, int size)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < size)
	{
		if (isnan(data[idx]) == 1)
		{
			data[idx] = 0;
		}
		else
		{
			data[idx] = data[idx];
		}

	}


}

__global__ void down_sampling(float* down_data, float* data, int down_size, int down_val, int col)	//device function for downsampling
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;							//down_size=no.of rows after downsampling
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idy < down_size && idx < col)
	{
		down_data[idy * col + idx] = data[down_val * idy * col + idx];	//down_val=down sampling factor
	}
}

__global__ void down_col(float* down_data, float* data, int down_col_size, int down_val, int col_size, int row)	//device function for downsampling
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;							//down_size=no.of rows after downsampling
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idy < row && idx < down_col_size)
	{
		down_data[idy * down_col_size + idx] = data[idy * col_size + idx * down_val];	//down_val=down sampling factor
	}
}

__global__ void element_division(float* mat_in, float value, int size, float* mat_out)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				   //max=final value of the array
	if (i < size)
	{
		mat_out[i] = mat_in[i] / value;
	}
}

__global__ void element_division(long double* mat_in, float value, int size, long double* mat_out)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				   //max=final value of the array
	if (i < size)
	{
		mat_out[i] = mat_in[i] / value;
	}
}

__global__ void range(int* out_data, int min, int arr_size, int inc)	//creates an array of a range of values
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				//max=final value of the array
	if (i < arr_size)																//arr_size==array size
	{																			//inc=increment needed
		out_data[i] = min + (i * inc);
	}
}

__global__ void range(float* out_data, float min, int arr_size, float inc)	//creates an array of a range of values
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				//max=final value of the array
	if (i < arr_size)																//arr_size==array size
	{																			//inc=increment needed
		out_data[i] = min + (i * inc);
	}
}

__global__ void range(double* out_data, double min, int arr_size, double inc)	//creates an array of a range of values
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				//max=final value of the array
	if (i < arr_size)																//arr_size==array size
	{																			//inc=increment needed
		out_data[i] = min + (i * inc);
	}
}

__global__ void range(long double* out_data, long double min, int arr_size, long double inc)	//creates an array of a range of values
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				//max=final value of the array
	if (i < arr_size)																//arr_size==array size
	{																			//inc=increment needed
		out_data[i] = min + (i * inc);
	}
}

__global__ void mat2D_abs(int* data, int m, int n, int* out_data)	//to find the absolute positive value of each elements in a matrix

{
	int col1 = blockDim.x * blockIdx.x + threadIdx.x;			//m and n are number of rows and colums respectively
	int row1 = blockDim.y * blockIdx.y + threadIdx.y;

	if (row1 < m && col1 < n)
	{
		int thread_id = row1 * n + col1;
		if (data[thread_id] < 0)
		{
			out_data[thread_id] = -1 * data[thread_id];		//negative values are converted to positive values
		}
		else
		{
			out_data[thread_id] = data[thread_id];
		}
	}
}

__global__ void mat2D_abs(float* data, int m, int n, float* out_data)	//to find the absolute positive value of each elements in a matrix

{
	int col1 = blockDim.x * blockIdx.x + threadIdx.x;			//m and n are number of rows and colums respectively
	int row1 = blockDim.y * blockIdx.y + threadIdx.y;

	if (row1 < m && col1 < n)
	{
		int thread_id = row1 * n + col1;
		if (data[thread_id] < 0)
		{
			out_data[thread_id] = -1 * data[thread_id];		//negative values are converted to positive values
		}
		else
		{
			out_data[thread_id] = data[thread_id];
		}
	}
}

__global__ void mat_sub(float* mat1, float d0, int row1, float* out)	//to subtract a specific value from each element in the array
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < row1)
	{
		out[idx] = mat1[idx] - d0;		//d0=value to be subtracted
	}
}

__global__ void mat_subset(float* mat, int row1, int col1, int c1, int c2, int r1, int r2, float* mat_out)	//to take a matrix subset from a large matrix
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	//row1 and col1 are size of large matrix
	int idy = blockIdx.y * blockDim.y + threadIdx.y;	//(r1,c1) (r2,c2)=min and max cordinates of the sub matrix
	if (idx < ((c2 - c1) + 1) && idy < ((r2 - r1) + 1))
	{
		int thread_id = idy * ((c2 - c1) + 1) + idx;
		int thread_id1 = (idy + r1) * col1 + (idx + c1);
		mat_out[thread_id] = mat[thread_id1];
	}
}

__global__ void mat_subset(int* mat, int row1, int col1, int c1, int c2, int r1, int r2, int* mat_out)	//to take a matrix subset from a large matrix
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	//row1 and col1 are size of large matrix
	int idy = blockIdx.y * blockDim.y + threadIdx.y;	//(r1,c1) (r2,c2)=min and max cordinates of the sub matrix
	if (idx < ((c2 - c1) + 1) && idy < ((r2 - r1) + 1))
	{
		int thread_id = idy * ((c2 - c1) + 1) + idx;
		int thread_id1 = (idy + r1) * col1 + (idx + c1);
		mat_out[thread_id] = mat[thread_id1];
	}
}

__global__ void element_square(float* mat, int size, float* out)	//to square each contents of a array
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		out[idx] = mat[idx] * mat[idx];
	}
}

__global__ void element_mul(float* mat1, float* mat2, int size, float* out)	//element wise multiplication of 2 arrays
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		out[idx] = mat1[idx] * mat2[idx];
	}
}

__global__ void mat_add(float* mat1, float* mat2, int row1, int col1)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx < col1 && idy < row1)
	{
		mat2[idy * col1 + idx] = mat1[idy * col1 + idx] + mat2[idy * col1 + idx];
	}
}

__global__ void array_add(double* mat1, double* mat2, int row1)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < row1)
	{
		mat2[idx] = mat1[idx] + mat2[idx];
	}
}

__global__ void mat_subset_1D(int* mat, int size, int first, int last, int* mat_out)	//to take a matrix subset from a large matrix
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	//row1 and col1 are size of large matrix
	//int idy = blockIdx.y * blockDim.y + threadIdx.y;	//(r1,c1) (r2,c2)=min and max cordinates of the sub matrix
	if (idx < ((last - first) + 1))
	{
		int thread_id = idx;
		int thread_id1 = idx + first;
		mat_out[thread_id] = mat[thread_id1];
	}
}

__global__ void mat_subset_1D(float* mat, int size, int first, int last, float* mat_out)	//to take a matrix subset from a large matrix
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	//row1 and col1 are size of large matrix
	//int idy = blockIdx.y * blockDim.y + threadIdx.y;	//(r1,c1) (r2,c2)=min and max cordinates of the sub matrix
	if (idx < ((last - first) + 1))
	{
		int thread_id = idx;
		int thread_id1 = (idx + first);
		mat_out[thread_id] = mat[thread_id1];
	}
}

__global__ void matrix_mult(float* mat1, float val, int size, float* out)	//element wise multiplication of 2 arrays
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		out[idx] = mat1[idx] * val;
	}
}

__global__ void matrix_mult1(float* mat1, float val, int size, float* out)	//element wise multiplication of 2 arrays
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		out[idx] = mat1[idx] * val;
	}
}

__global__ void upsamp_append(float* mat_out, float* mat_in, int first_row, int samp_fact, int row1, int col1)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idy < row1 && idx < col1)
	{
		mat_out[(samp_fact * idy + first_row) * col1 + idx] = mat_in[idy * col1 + idx];
	}
}

__global__ void mat_transpose(float* mat_in, float* mat_out, int row_org, int col_org)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx < col_org && idy < row_org)
	{
		mat_out[idx * row_org + idy] = mat_in[idy * col_org + idx];
	}
}

double** reshapeto2D(float* inArray, int rows, int cols) {
	// Conevert the single array of size rows*col into 2 dimensional array of size rows and col
	// Declare new Array
	double** array2D = (double**)malloc(sizeof(double) * cols);
	for (int i = 0; i < cols; i++) {
		array2D[i] = (double*)malloc(rows);
	}

	// Fill the values
	for (int i = 0; i < cols; i++) {
		for (int j = 0; j < rows; j++) {
			array2D[i][j] = (double)inArray[i * rows + j];
		}
	}

	return array2D;
}

double** zeroArray(int rows, int cols) {
	// Conevert the single array of size rows*col into 2 dimensional array of size rows and col
	// Declare new Array
	double** array2D = (double**)malloc(sizeof(double) * cols);
	for (int i = 0; i < cols; i++) {
		array2D[i] = (double*)malloc(rows);
	}

	// Fill the values
	for (int i = 0; i < cols; i++) {
		for (int j = 0; j < rows; j++) {
			array2D[i][j] = (double)0.0;
		}
	}

	return array2D;
}

// Extern void bModeGenerationinCUDA(float* t, float* v, int tno)
double** bModeGen()
{
	const int MAX_LINE = 256;
	const int MASK_WIDTH = 364;
	const int TILE_SIZE = 4;

	//// Computer (NIVIDIA) parametrs
	int num_threads = 1024;

	/// Apodization parameters
	float rx_f_number = 2.0;

	/////// Ultrasound scanner parametrs
	//float depth = 49.28;      // Depth of imaging in mm
	int samples = 2040;                    // # of samples in depth direction
	int N_elements = 64;         // # of transducer elements
	float sampling_frequency = 32e6;   // sampling frequency
	float c = 1540.0;      // speed of sound [m/s]	
	int N_active = 8;                         // Active transmit elmeents
	float pitch = 0.3 / 1000;           // spacing between the elements
	float aper_len = (N_elements - 1) * pitch * 1000;			 //aperture foot print 
	float zd = pitch * N_active / (float)2;            // virtual src distance from transducer array 
	float sample_spacing = c / sampling_frequency / (float)2;
	float del_convert = sampling_frequency / c;  // used in delay calculation

	int channels = 64;	              // number of A-lines data used for beamforming

	//// Beamforming "Grid" parameters
	int Nx = 256;      // 256 Lateral spacing
	int Nz = 1024;            //1024 Axial spacing
	int pixels = Nz * Nx;
	int pix_cha = pixels * channels;     // Nz*Nx*128 This array size is used for Apodization
	int num_frames = 57;   // number of low resolution images
	int skip_frames = 1;  // 

	// Post processing parameters.
	//int dBvalue = 60;

	float* filt_coeff = new float[MASK_WIDTH];
	char filename1[200];
	sprintf(filename1, "C:/Users/CSR_L/source/repos/beamforming_parallel3_PA/b_10M.csv");
	read_csv_array(filt_coeff, filename1);    // csv file read

	float* d_filt_coeff = 0;
	hipMalloc((void**)&d_filt_coeff, sizeof(float) * MASK_WIDTH);
	hipMemcpy(d_filt_coeff, filt_coeff, sizeof(float) * MASK_WIDTH, hipMemcpyHostToDevice);

	////////  Intialization &(or) Memory allocation  //////////////////
	float* d_data = 0;   // variable to store raw rf data
	hipMalloc((void**)&d_data, sizeof(float) * samples * channels);

	float* d_bfHR = 0;  // variable to store beamformed high-resolution beamformed image 
	hipMalloc((void**)&d_bfHR, pixels * sizeof(float));
	//zeros << <pixels / num_threads + 1, num_threads >> > (d_bfHR, pixels);  
	hipMemset(d_bfHR, 0, pixels * sizeof(float));

	float* dev_beamformed_data1 = 0;   // variable to store reshaped beamformed data
	hipMalloc((void**)&dev_beamformed_data1, pixels * sizeof(float));

	float* d_bfHRBP = 0;  // variable to store beamformed high-resolution bandpass filtered data
	hipMalloc((void**)&d_bfHRBP, sizeof(float) * pixels);

	float* env = new float[pixels]; // Host memory variable to store beamformed high-resolution bandpass filtered data


	////////////// z value////////////////////
	float dz = sample_spacing * samples / Nz;  // depth / (Nz - 1) / 1000;   // spacing in axial (z) direction in mm;
	float* d_z_axis = 0;
	hipMalloc((void**)&d_z_axis, Nz * sizeof(float));
	range << <Nz / num_threads + 1, num_threads >> > (d_z_axis, 0, Nz, dz);
	hipGetLastError();
	hipDeviceSynchronize();

	//////////////////////////////// x value////////////////////////////////
	float dx = aper_len / (Nx - 1);
	float* d_x_axis = 0;
	hipMalloc((void**)&d_x_axis, Nx * sizeof(float));    // 167.939 us
	range << <Nx / num_threads + 1, num_threads >> > (d_x_axis, (-aper_len / 2000), Nx, dx / 1000);
	hipGetLastError();
	hipDeviceSynchronize();

	//////////////// Probe geometry, this info can be taken from transducer file ////////////////////
	float* d_probe = 0;
	hipMalloc((void**)&d_probe, N_elements * sizeof(float));
	range << <1, N_elements >> > (d_probe, (-aper_len / 2000), N_elements, pitch);
	hipGetLastError();
	hipDeviceSynchronize();

	/////////////////rx aperture calculation using Fnumber///////////////////////////////
	// rx_aper=rfsca.z/rf_number
	float* d_rx_aperture = 0;
	hipMalloc((void**)&d_rx_aperture, Nz * sizeof(float));
	element_division << <Nz / num_threads + 1, num_threads >> > (d_z_axis, rx_f_number, Nz, d_rx_aperture);
	hipGetLastError();
	hipDeviceSynchronize();

	////////////////////////rx aperture distance////////
	float* d_rx_ap_distance = 0;
	hipMalloc((void**)&d_rx_ap_distance, channels * Nx * sizeof(float));
	aperture_distance << <Nx * channels / num_threads + 1, num_threads >> > (d_x_axis, d_probe, Nx, channels, d_rx_ap_distance);
	hipGetLastError();
	hipDeviceSynchronize();

	///////////////////apodization/////////////////
	float* d_rx_apod = 0;
	hipMalloc((void**)&d_rx_apod, sizeof(float) * Nz * channels * Nx);
	apodization << <pixels * channels / num_threads + 1, num_threads >> > (d_rx_ap_distance, d_rx_aperture, Nz, Nx, channels, pixels, d_rx_apod);
	hipGetLastError();
	hipDeviceSynchronize();

	hipFree(d_rx_aperture);
	hipFree(d_rx_ap_distance);

	/////////////////// calculate central positions transmit subaperture ////////////////////
	float* d_cen_pos = 0;
	hipMalloc((void**)&d_cen_pos, num_frames * sizeof(float));
	Tx_cen_pos << < 1, num_frames >> > (d_cen_pos, N_elements, N_active, pitch, skip_frames, num_frames, d_probe);

	/////////////receive delay calculation /////////////////////////////////////////////
	float* d_rx_delay = 0;
	hipMalloc((void**)&d_rx_delay, pix_cha * sizeof(float));
	receive_delay << < pixels * channels / num_threads + 1, num_threads >> > (d_probe, d_x_axis, d_z_axis, channels, Nx, Nz, del_convert, d_rx_delay);
	hipGetLastError();
	hipDeviceSynchronize();

	/////////////////// Transmit delay calculation ////////////////////
	float* d_tx_delay = 0;
	hipMalloc((void**)&d_tx_delay, pixels * num_frames * sizeof(float));
	//transmit delay for all frames,   
	transmit_delay << < pixels * num_frames / num_threads + 1, num_threads >> > (d_x_axis, d_z_axis, d_cen_pos, zd, Nx, Nz, del_convert, num_frames, d_tx_delay);
	hipGetLastError();
	hipDeviceSynchronize();

	////////////Free cuda memory (one time use) ///////////////////////////
	hipFree(d_probe);
	hipFree(d_x_axis);
	hipFree(d_z_axis);
	hipFree(d_cen_pos);

	//for (int i = 0; i < 2; i++) {
	//	unsigned char buf[16 * 1024];
	//	int row = 0;  // Keep track of how many rows have been added
	//	errno_t err;
	//	char line[MAX_LINE]; // Max possible line length?
	//	FILE* fp;
	//	if ((err = fopen_s(&fp, "out25.txt", "r")) != 0) {
	//		printf("Could not open config file for reading.\n");
	//		exit(1);
	//	}
	//	CCyUSBDevice* USBDevice = new CCyUSBDevice(NULL);
	//	// Obtain the control endpoint pointer
	//	CCyControlEndPoint* ept = USBDevice->ControlEndPt;
	//	if (!ept) {
	//		printf("Could not get Control endpoint.\n");
	//		exit(1);
	//	}
	//	// Send a vendor request (bRequest = 0x05) to the device
	//	ept->Target = TGT_DEVICE;
	//	ept->ReqType = REQ_VENDOR;
	//	ept->Direction = DIR_TO_DEVICE;
	//	ept->ReqCode = 0x05;
	//	ept->Value = 1;
	//	ept->Index = 0;
	//	ept->TimeOut = 100;  // set timeout to 100ms for quick response
	//	// Endpoint for reading back data
	//	CCyBulkEndPoint* ept_in;
	//	ept_in = USBDevice->BulkInEndPt;
	//	if (!ept_in) {
	//		//printf("No IN endpoint??\n");
	//		exit(1);
	//	}
	//	ept_in->MaxPktSize = 16384;
	//	ept_in->TimeOut = 100;  // set timeout to 100ms for reading
	//	int iteration = 0;
	//	int errcount = 0;
	//	unsigned int addr, data;
	//	unsigned char recvbuf[2048 * 64 * 2];
	//	const int MAXROWS = 2040;
	//	LONG rxlen = MAXROWS * 64 * 2;
	//	unsigned int start = clock();
	//	while (fgets(line, MAX_LINE, fp)) {
	//		line[strcspn(line, "\n")] = 0; // Trim trailing newline
	//		if ((strlen(line) == 0) || (line[0] == ' ') || (line[0] == '#')) {
	//			printf("Skipping [%s]\n", line);
	//		}
	//		else if (line[0] == 'O') {
	//			sscanf_s(line, "O %04X %08X ", &addr, &data);
	//			printf("Write %08X to Obelix %04X\n", data, addr);
	//			row = insert_row(buf, row, addr, data);
	//		}
	//		else if (line[0] == 'T') {
	//			sscanf_s(line, "T %04X %08X ", &addr, &data);
	//			row = insert_row(buf, row, 0x6, 0x40000000 | addr);
	//			row = insert_row(buf, row, 0x7, data);
	//			row = insert_row(buf, row, 0x6, 0xC0000000 | addr);
	//			printf("Write %08X to TX %04X\n", data, addr);
	//		}
	//		else if (line[0] == 'A') {
	//			sscanf_s(line, "A %04X %08X ", &addr, &data);
	//			row = insert_row(buf, row, 0x6, 0x00000000 | addr);
	//			row = insert_row(buf, row, 0x7, data);
	//			row = insert_row(buf, row, 0x6, 0x80000000 | addr);
	//			printf("Write %08X to AFE %04X\n", data, addr);
	//		}
	//		else if (line[0] == 'C') {  // CAPTURE
	//		 //wait(100);
	//			row = insert_row(buf, row, 0x4, 0x01);
	//			//write_rows(ept, buf, row);  // Send commands
	//			//wait(100);
	//			row = insert_row(buf, row, 0x4, 0x10);
	//			//write_rows(ept, buf, row);  // Send commands
	//			//wait(100);
	//			row = insert_row(buf, row, 0x4, 0x00);
	//			ept_in->Abort();
	//			ept_in->Reset();
	//			write_rows(ept, buf, row);  // Send commands
	//			//wait(100);
	//			//row = insert_row(buf, row, 0x4, 0x03);
	//			//row = insert_row(buf, row, 0x4, 0x10);
	//			//row = insert_row(buf, row, 0x4, 0x00);
	//			//printf("CAPTURE %2d: ", iteration);
	//			//write_rows(ept, buf, row);  // Send commands
	//			// One iteration should have 2048 samples * 64 channels * 2 bytes each
	//			wait(1);
	//			if (read_chunk(ept_in, recvbuf, rxlen)) {
	//				short* rxdata = (short*)(recvbuf);
	//				for (int i = 0; i < rxlen / 2; i++) {
	//					if (rxdata[i] >= 512) rxdata[i] -= 1024;
	//				}
	//				// Trying to read only first N-1 rows and discard 1st sample
	//				for (int i = 0; i < 64; i++) {
	//					for (int j = 0; j < MAXROWS - 1; j++) {
	//						//rximg[iteration][i][j] = rxdata[j*64+i+2];
	//						rximg[i * MAXROWS + j] = rxdata[j * 64 + i + 2];
	//					}
	//				}
	//				//saveToFile(iteration, rxlen, recvbuf);
	//			}
	//			else {
	//				errcount++;
	//			}
	//			//clock_t begin = clock();   // clock intiated
	//			hipMemcpy(d_data, rximg, sizeof(float) * samples * channels, hipMemcpyHostToDevice);
	//			beamformingLR3 << <(pixels / 256) * channels, 256 >> > (d_bfHR, d_tx_delay, d_rx_delay, d_data, d_rx_apod, samples, pixels, iteration, num_frames, channels);
	//			hipGetLastError();
	//			hipDeviceSynchronize();
	//			//clock_t end = clock();
	//			//float elapsed_secs = float(end - begin) / CLOCKS_PER_SEC;
	//			//printf("Time for beamforming in ms: %f\n", elapsed_secs * 1000);
	//			iteration++; // Increment iteration after saving to image
	//			row = 0;   // Reset buffer for next iteration
	//		}
	//		else {
	//			//printf("Don't know how to handle [%s] yet.\n", line);
	//		}
	//	}
	//	//unsigned int stop = clock();
	//	//printf("\n\n\n******\n");
	//	//printf("Ran %d iterations with %d errors in %d ms\n", iteration, errcount, stop - start);
	//	//// check for nan values,
	//	isnan_test_array << <pixels / num_threads + 1, num_threads >> > (d_bfHR, pixels);
	//	hipGetLastError();
	//	hipDeviceSynchronize();
	//	//////////// Bandpass filtering using shared memory /////////////////////
	//	BPfilter1SharedMem << <(pixels + TILE_SIZE - 1) / TILE_SIZE, TILE_SIZE >> > (d_bfHR, d_filt_coeff, pixels, d_bfHRBP);
	//	hipGetLastError();
	//	hipDeviceSynchronize();
	//	//////////////// reshape of the beamformed data ///////////////
	//	reshape_columnwise << <pixels / num_threads + 1, num_threads >> > (Nx, Nz, dev_beamformed_data1, d_bfHRBP);
	//	hipGetLastError();
	//	hipDeviceSynchronize();
	//	hipMemcpy(env, dev_beamformed_data1, Nz * Nx * sizeof(float), hipMemcpyDeviceToHost);
	//	char fileout[200];
	//	sprintf(fileout, "C:/Users/CSR_L/Downloads/UTScannerApp_realTime/b_mode_%d.csv", i); //all the 16 inputs are arranged in a single file
	//	csv_write_mat(env, fileout, Nz, Nx);
	//	//cv::Mat bmodMat = converttoMat(env, Nz, Nx);
	//	double** bmodArray = reshapeto2D(env, Nz, Nx);
	//	return bmodArray;
	//}

	////////////////// Free cuda memory (that will be used again) ///////////////
	//hipFree(d_data);
	//hipFree(d_bfHR);
	//hipFree(d_tx_delay);
	//hipFree(d_rx_delay);
	//hipFree(d_rx_apod);
	//hipFree(dev_beamformed_data1);
	//hipFree(d_bfHRBP);
	double** dummyData = zeroArray(Nz, Nx);

	return dummyData;

}

cv::Mat  bModeGenTest1()
{

	// perform b-mode generation here using cuda

	const int TILE_SIZE = 4;
	int MASK_WIDTH = 364;
	//// Computer (NIVIDIA) parametrs
	int num_threads = 1024;

	/// Apodization parameters
	float rx_f_number = 2.0;

	/////// Ultrasound scanner parametrs
	//float depth = 49.28;      // Depth of imaging in mm
	int samples = 2008;                    // # of samples in depth direction
	int N_elements = 64;         // # of transducer elements
	float sampling_frequency = 32e6;   // sampling frequency
	float c = 1540.0;      // speed of sound [m/s]	
	int N_active = 8;                         // Active transmit elmeents
	float pitch = 0.3 / 1000;           // spacing between the elements
	float aper_len = (N_elements - 1) * pitch * 1000;			 //aperture foot print 
	float zd = pitch * N_active / (float)2;            // virtual src distance from transducer array 
	float sample_spacing = c / sampling_frequency / (float)2;
	float del_convert = sampling_frequency / c;  // used in delay calculation

	int channels = 64;	              // number of A-lines data used for beamforming

	//// Beamforming "Grid" parameters
	int Nx = 256;      // 256 Lateral spacing
	int Nz = 1024;            //1024 Axial spacing
	int pixels = Nz * Nx;
	int pix_cha = pixels * channels;     // Nz*Nx*128 This array size is used for Apodization
	int num_frames = 57;   // number of low resolution images
	int skip_frames = 1;  // 

	// Post processing parameters.
	//int dBvalue = 60;

	float* filt_coeff = new float[MASK_WIDTH];
	char filename1[200];
	sprintf(filename1, "C:/Users/CSR_L/source/repos/beamforming_parallel3_PA/b_10M.csv");
	read_csv_array(filt_coeff, filename1);    // csv file read

	float* d_filt_coeff = 0;
	hipMalloc((void**)&d_filt_coeff, sizeof(float) * MASK_WIDTH);
	hipMemcpy(d_filt_coeff, filt_coeff, sizeof(float) * MASK_WIDTH, hipMemcpyHostToDevice);


	////////////// z value////////////////////
	float dz = sample_spacing * samples / Nz;  // depth / (Nz - 1) / 1000;   // spacing in axial (z) direction in mm;
	float* d_z_axis = 0;
	hipMalloc((void**)&d_z_axis, Nz * sizeof(float));
	range << <Nz / num_threads + 1, num_threads >> > (d_z_axis, 0, Nz, dz);
	hipGetLastError();
	hipDeviceSynchronize();

	//////////////////////////////// x value////////////////////////////////
	float dx = aper_len / (Nx - 1);
	float* d_x_axis = 0;
	hipMalloc((void**)&d_x_axis, Nx * sizeof(float));    // 167.939 us
	range << <Nx / num_threads + 1, num_threads >> > (d_x_axis, (-aper_len / 2000), Nx, dx / 1000);
	hipGetLastError();
	hipDeviceSynchronize();

	//////////////// Probe geometry, this info can be taken from transducer file ////////////////////
	float* d_probe = 0;
	hipMalloc((void**)&d_probe, N_elements * sizeof(float));
	range << <1, N_elements >> > (d_probe, (-aper_len / 2000), N_elements, pitch);
	hipGetLastError();
	hipDeviceSynchronize();

	/////////////////rx aperture calculation using Fnumber///////////////////////////////
	// rx_aper=rfsca.z/rf_number
	float* d_rx_aperture = 0;
	hipMalloc((void**)&d_rx_aperture, Nz * sizeof(float));
	element_division << <Nz / num_threads + 1, num_threads >> > (d_z_axis, rx_f_number, Nz, d_rx_aperture);
	hipGetLastError();
	hipDeviceSynchronize();

	////////////////////////rx aerture distance////////
	float* d_rx_ap_distance = 0;
	hipMalloc((void**)&d_rx_ap_distance, channels * Nx * sizeof(float));
	aperture_distance << <Nx * channels / num_threads + 1, num_threads >> > (d_x_axis, d_probe, Nx, channels, d_rx_ap_distance);
	hipGetLastError();
	hipDeviceSynchronize();

	///////////////////apodization/////////////////
	float* d_rx_apod = 0;
	hipMalloc((void**)&d_rx_apod, sizeof(float) * Nz * channels * Nx);
	apodization << <pixels * channels / num_threads + 1, num_threads >> > (d_rx_ap_distance, d_rx_aperture, Nz, Nx, channels, pixels, d_rx_apod);
	hipGetLastError();
	hipDeviceSynchronize();

	hipFree(d_rx_aperture);
	hipFree(d_rx_ap_distance);

	/////////////////// calculate central positions transmit subaperture ////////////////////
	float* d_cen_pos = 0;
	hipMalloc((void**)&d_cen_pos, num_frames * sizeof(float));
	Tx_cen_pos << < 1, num_frames >> > (d_cen_pos, N_elements, N_active, pitch, skip_frames, num_frames, d_probe);

	/////////////receive delay calculation /////////////////////////////////////////////
	float* d_rx_delay = 0;
	hipMalloc((void**)&d_rx_delay, pix_cha * sizeof(float));
	receive_delay << < pixels * channels / num_threads + 1, num_threads >> > (d_probe, d_x_axis, d_z_axis, channels, Nx, Nz, del_convert, d_rx_delay);
	hipGetLastError();
	hipDeviceSynchronize();

	////////////Initialize d_bfHR to store final high-resolution beamformed image /////////////////////////////
	float* d_bfHR = 0;
	hipMalloc((void**)&d_bfHR, pixels * sizeof(float));
	//zeros << <pixels / num_threads + 1, num_threads >> > (d_bfHR, pixels);  
	hipMemset(d_bfHR, 0, pixels * sizeof(float));

	/////////////////// Transmit delay calculation ////////////////////
	float* d_tx_delay = 0;
	hipMalloc((void**)&d_tx_delay, pixels * num_frames * sizeof(float));
	//transmit delay for all frames,   
	transmit_delay << < pixels * num_frames / num_threads + 1, num_threads >> > (d_x_axis, d_z_axis, d_cen_pos, zd, Nx, Nz, del_convert, num_frames, d_tx_delay);
	hipGetLastError();
	hipDeviceSynchronize();


	////////////Free cuda memory (one time use) ///////////////////////////
	hipFree(d_probe);
	hipFree(d_x_axis);
	hipFree(d_z_axis);
	hipFree(d_cen_pos);

	float* data = new float[samples * channels];

	float* d_data = 0;
	hipMalloc((void**)&d_data, sizeof(float) * samples * channels);

	cv::Mat outMat = cv::Mat::zeros(1024, 254, CV_64FC1);

	return outMat;
}

cv::Mat  bModeGenTest2()
{
	//// perform b-mode generation here using cuda
	//const int TILE_SIZE = 4;
	//int MASK_WIDTH = 364;
	//const int MAX_LINE = 256;

	////// Computer (NIVIDIA) parametrs
	//int num_threads = 1024;
	///// Apodization parameters
	//float rx_f_number = 2.0;
	///////// Ultrasound scanner parametrs
	////float depth = 49.28;      // Depth of imaging in mm
	//int samples = 2040;                    // # of samples in depth direction
	//int N_elements = 64;         // # of transducer elements
	//float sampling_frequency = 32e6;   // sampling frequency
	//float c = 1540.0;      // speed of sound [m/s]	
	//int N_active = 8;                         // Active transmit elmeents
	//float pitch = 0.3 / 1000;           // spacing between the elements
	//float aper_len = (N_elements - 1) * pitch * 1000;			 //aperture foot print 
	//float zd = pitch * N_active / (float)2;            // virtual src distance from transducer array 
	//float sample_spacing = c / sampling_frequency / (float)2;
	//float del_convert = sampling_frequency / c;  // used in delay calculation
	//int channels = 64;	              // number of A-lines data used for beamforming
	////// Beamforming "Grid" parameters
	//int Nx = 256;      // 256 Lateral spacing
	//int Nz = 1024;            //1024 Axial spacing
	//int pixels = Nz * Nx;
	//int pix_cha = pixels * channels;     // Nz*Nx*128 This array size is used for Apodization
	//int num_frames = 57;   // number of low resolution images
	//int skip_frames = 1;  //

	//// parameters for matrix processing
	//int croppedBot = 300;
	//cv::Mat outMat, outMatCrp, envolepMat, logcMat;

	//// Post processing parameters.
	////int dBvalue = 60;

	//float* filt_coeff = new float[MASK_WIDTH];
	//char filename1[200];
	//sprintf(filename1, "C:\\Users\\CSR_L\\source\\repos\\beamforming_parallel3_PA\\b_10M.csv");
	//read_csv_array(filt_coeff, filename1);    // csv file read

	//float* d_filt_coeff = 0;
	//hipMalloc((void**)&d_filt_coeff, sizeof(float) * MASK_WIDTH);
	//hipMemcpy(d_filt_coeff, filt_coeff, sizeof(float) * MASK_WIDTH, hipMemcpyHostToDevice);

	//////////  Intialization &(or) Memory allocation  //////////////////
	//float* d_data = 0;   // variable to store raw rf data
	//hipMalloc((void**)&d_data, sizeof(float) * samples * channels);

	//float* d_bfHR = 0;  // variable to store beamformed high-resolution beamformed image 
	//hipMalloc((void**)&d_bfHR, pixels * sizeof(float));
	////zeros << <pixels / num_threads + 1, num_threads >> > (d_bfHR, pixels);  
	//hipMemset(d_bfHR, 0, pixels * sizeof(float));

	//float* dev_beamformed_data1 = 0;   // variable to store reshaped beamformed data
	//hipMalloc((void**)&dev_beamformed_data1, pixels * sizeof(float));

	//float* d_bfHRBP = 0;  // variable to store beamformed high-resolution bandpass filtered data
	//hipMalloc((void**)&d_bfHRBP, sizeof(float) * pixels);

	//float* env = new float[pixels]; // Host memory variable to store beamformed high-resolution bandpass filtered data

	//////////////// z value////////////////////
	//float dz = sample_spacing * samples / Nz;  // depth / (Nz - 1) / 1000;   // spacing in axial (z) direction in mm;
	//float* d_z_axis = 0;
	//hipMalloc((void**)&d_z_axis, Nz * sizeof(float));
	//range << <Nz / num_threads + 1, num_threads >> > (d_z_axis, 0, Nz, dz);
	//hipGetLastError();
	//hipDeviceSynchronize();

	////////////////////////////////// x value////////////////////////////////
	//float dx = aper_len / (Nx - 1);
	//float* d_x_axis = 0;
	//hipMalloc((void**)&d_x_axis, Nx * sizeof(float));    // 167.939 us
	//range << <Nx / num_threads + 1, num_threads >> > (d_x_axis, (-aper_len / 2000), Nx, dx / 1000);
	//hipGetLastError();
	//hipDeviceSynchronize();

	////////////////// Probe geometry, this info can be taken from transducer file ////////////////////
	//float* d_probe = 0;
	//hipMalloc((void**)&d_probe, N_elements * sizeof(float));
	//range << <1, N_elements >> > (d_probe, (-aper_len / 2000), N_elements, pitch);
	//hipGetLastError();
	//hipDeviceSynchronize();

	///////////////////rx aerture calculation using Fnumber///////////////////////////////
	//// rx_aper=rfsca.z/rf_number
	//float* d_rx_aperture = 0;
	//hipMalloc((void**)&d_rx_aperture, Nz * sizeof(float));
	//element_division << <Nz / num_threads + 1, num_threads >> > (d_z_axis, rx_f_number, Nz, d_rx_aperture);
	//hipGetLastError();
	//hipDeviceSynchronize();

	//////////////////////////rx aerture distance////////
	//float* d_rx_ap_distance = 0;
	//hipMalloc((void**)&d_rx_ap_distance, channels * Nx * sizeof(float));
	//aperture_distance << <Nx * channels / num_threads + 1, num_threads >> > (d_x_axis, d_probe, Nx, channels, d_rx_ap_distance);
	//hipGetLastError();
	//hipDeviceSynchronize();

	/////////////////////apodization/////////////////
	//float* d_rx_apod = 0;
	//hipMalloc((void**)&d_rx_apod, sizeof(float) * Nz * channels * Nx);
	//apodization << <pixels * channels / num_threads + 1, num_threads >> > (d_rx_ap_distance, d_rx_aperture, Nz, Nx, channels, pixels, d_rx_apod);
	//hipGetLastError();
	//hipDeviceSynchronize();

	//hipFree(d_rx_aperture);
	//hipFree(d_rx_ap_distance);

	///////////////////// calculate central positions transmit subaperture ////////////////////
	//float* d_cen_pos = 0;
	//hipMalloc((void**)&d_cen_pos, num_frames * sizeof(float));
	//Tx_cen_pos << < 1, num_frames >> > (d_cen_pos, N_elements, N_active, pitch, skip_frames, num_frames, d_probe);

	///////////////receive delay calculation /////////////////////////////////////////////
	//float* d_rx_delay = 0;
	//hipMalloc((void**)&d_rx_delay, pix_cha * sizeof(float));
	//receive_delay << < pixels * channels / num_threads + 1, num_threads >> > (d_probe, d_x_axis, d_z_axis, channels, Nx, Nz, del_convert, d_rx_delay);
	//hipGetLastError();
	//hipDeviceSynchronize();

	///////////////////// Transmit delay calculation ////////////////////
	//float* d_tx_delay = 0;
	//hipMalloc((void**)&d_tx_delay, pixels * num_frames * sizeof(float));
	////transmit delay for all frames,   
	//transmit_delay << < pixels * num_frames / num_threads + 1, num_threads >> > (d_x_axis, d_z_axis, d_cen_pos, zd, Nx, Nz, del_convert, num_frames, d_tx_delay);
	//hipGetLastError();
	//hipDeviceSynchronize();

	//////////////Free cuda memory (one time use) ///////////////////////////
	//hipFree(d_probe);
	//hipFree(d_x_axis);
	//hipFree(d_z_axis);
	//hipFree(d_cen_pos);

	//CCyUSBDevice* USBDevice = new CCyUSBDevice(NULL);
	//// Obtain the control endpoint pointer
	//CCyControlEndPoint* ept = USBDevice->ControlEndPt;
	//if (!ept) {
	//	printf("Could not get Control endpoint.\n");
	//	exit(1);
	//}


	//for (int i = 0; i < 1; i++) {
	//	unsigned char buf[16 * 1024];
	//	int row = 0;  // Keep track of how many rows have been added
	//	errno_t err;
	//	char line[MAX_LINE]; // Max possible line length?
	//	FILE* fp;
	//	if ((err = fopen_s(&fp, "C:/Users/CSR_L/Downloads/UTScannerApp_realTime/out25.txt", "r")) != 0) {
	//		printf("Could not open config file for reading.\n");
	//		exit(1);
	//	}

	//	// Send a vendor request (bRequest = 0x05) to the device
	//	ept->Target = TGT_DEVICE;
	//	ept->ReqType = REQ_VENDOR;
	//	ept->Direction = DIR_TO_DEVICE;
	//	ept->ReqCode = 0x05;
	//	ept->Value = 1;
	//	ept->Index = 0;
	//	ept->TimeOut = 100;  // set timeout to 100ms for quick response


	//	// Endpoint for reading back data
	//	CCyBulkEndPoint* ept_in;
	//	ept_in = USBDevice->BulkInEndPt;
	//	if (!ept_in) {
	//		//printf("No IN endpoint??\n");
	//		exit(1);
	//	}

	//	ept_in->MaxPktSize = 16384;
	//	ept_in->TimeOut = 100;  // set timeout to 100ms for reading
	//	int iteration = 0;
	//	int errcount = 0;
	//	unsigned int addr, data;
	//	unsigned char recvbuf[2048 * 64 * 2];
	//	const int MAXROWS = 2040;
	//	LONG rxlen = MAXROWS * 64 * 2;

	//	// unsigned int start = clock();
	//	while (fgets(line, MAX_LINE, fp)) {
	//		line[strcspn(line, "\n")] = 0; // Trim trailing newline
	//		if ((strlen(line) == 0) || (line[0] == ' ') || (line[0] == '#')) {
	//			//printf("Skipping [%s]\n", line);
	//		}
	//		else if (line[0] == 'O') {
	//			sscanf_s(line, "O %04X %08X ", &addr, &data);
	//			//printf("Write %08X to Obelix %04X\n", data, addr);
	//			row = insert_row(buf, row, addr, data);
	//		}
	//		else if (line[0] == 'T') {
	//			sscanf_s(line, "T %04X %08X ", &addr, &data);
	//			row = insert_row(buf, row, 0x6, 0x40000000 | addr);
	//			row = insert_row(buf, row, 0x7, data);
	//			row = insert_row(buf, row, 0x6, 0xC0000000 | addr);
	//			//printf("Write %08X to TX %04X\n", data, addr);
	//		}
	//		else if (line[0] == 'A') {
	//			sscanf_s(line, "A %04X %08X ", &addr, &data);
	//			row = insert_row(buf, row, 0x6, 0x00000000 | addr);
	//			row = insert_row(buf, row, 0x7, data);
	//			row = insert_row(buf, row, 0x6, 0x80000000 | addr);
	//			//printf("Write %08X to AFE %04X\n", data, addr);
	//		}
	//		else if (line[0] == 'C') {  // CAPTURE
	//			//wait(100);
	//			row = insert_row(buf, row, 0x4, 0x01);
	//			//write_rows(ept, buf, row);  // Send commands
	//			//wait(100);
	//			row = insert_row(buf, row, 0x4, 0x10);
	//			//write_rows(ept, buf, row);  // Send commands
	//			//wait(100);
	//			row = insert_row(buf, row, 0x4, 0x00);
	//			ept_in->Abort();
	//			ept_in->Reset();
	//			write_rows(ept, buf, row);  // Send commands
	//			wait(1);
	//			if (read_chunk(ept_in, recvbuf, rxlen)) {
	//				short* rxdata = (short*)(recvbuf);
	//				for (int i = 0; i < rxlen / 2; i++) {
	//					if (rxdata[i] >= 512) rxdata[i] -= 1024;
	//				}
	//				// Trying to read only first N-1 rows and discard 1st sample
	//				for (int i = 0; i < 64; i++) {
	//					for (int j = 0; j < MAXROWS - 1; j++) {
	//						//rximg[iteration][i][j] = rxdata[j*64+i+2];
	//						rximg[i * MAXROWS + j] = rxdata[j * 64 + i + 2];
	//					}
	//				}
	//				//saveToFile(iteration, rxlen, recvbuf);
	//			}
	//			else {
	//				errcount++;
	//			}
	//			hipMemcpy(d_data, rximg, sizeof(float) * samples * channels, hipMemcpyHostToDevice);
	//			beamformingLR3 << <(pixels / 256) * channels, 256 >> > (d_bfHR, d_tx_delay, d_rx_delay, d_data, d_rx_apod, samples, pixels, iteration, num_frames, channels);
	//			hipGetLastError();
	//			hipDeviceSynchronize();
	//			iteration++; // Increment iteration after saving to image
	//			row = 0;   // Reset buffer for next iteration
	//		}
	//		else {
	//			printf("Don't know how to handle [%s] yet.\n", line);
	//		}
	//	}

	//	//// check for nan values,
	//	isnan_test_array << <pixels / num_threads + 1, num_threads >> > (d_bfHR, pixels);
	//	hipGetLastError();
	//	hipDeviceSynchronize();

	//	//////////// Bandpass filtering using shared memory /////////////////////
	//	BPfilter1SharedMem << <(pixels + TILE_SIZE - 1) / TILE_SIZE, TILE_SIZE >> > (d_bfHR, d_filt_coeff, pixels, d_bfHRBP);
	//	hipGetLastError();
	//	hipDeviceSynchronize();

	//	//////////////// reshape of the beamformed data ///////////////
	//	reshape_columnwise << <pixels / num_threads + 1, num_threads >> > (Nx, Nz, dev_beamformed_data1, d_bfHRBP);
	//	hipGetLastError();
	//	hipDeviceSynchronize();

	//	hipMemcpy(env, dev_beamformed_data1, Nz * Nx * sizeof(float), hipMemcpyDeviceToHost);

	//	//char fileout[200];
	//	// sprintf(fileout, "C:\\Users\\CSR_L\\Downloads\\UTScannerApp_realTime\\b_mode_%d.csv", i); //all the 16 inputs are arranged in a single file
	//	// csv_write_mat(env, fileout, Nz, Nx);

	//	outMat = converttoMat(env, Nz, Nx);
	//	outMatCrp = outMat(cv::Range(0, Nz - croppedBot), cv::Range(0, Nx));
	//	envolepMat = hilbertTrans4(outMatCrp, 1.0);
	//	// log compression
	//	logcMat = logTransform(envolepMat);

	//	return logcMat;
	//}

	// dummy Mat
	cv::Mat outMat = cv::Mat::zeros(1024 - 300, 254, CV_64FC1);
	return outMat;
}

double** ConvertMatto2DArray(cv::Mat img)
{
	double** array2D = (double**)malloc(img.rows * sizeof(double*));
	for (int i = 0; i < img.rows; i++) {
		array2D[i] = (double*)malloc(img.cols * sizeof(double));
	}


	//int** arr = (int**)malloc(r * sizeof(int*));
	//for (i = 0; i < r; i++)
	//	arr[i] = (int*)malloc(c * sizeof(int));

	// Fill the values
	for (int i = 0; i < img.rows; i++) {
		for (int j = 0; j < img.cols; j++) {
			array2D[i][j] = img.at<double>(i, j);
		}
	}

	return array2D;
}

extern double** imageGenProcessinCUDA() {

	// B-Mode image generation code fully in CUDA
	double minP, maxP;
	cv::Mat bmodMat = bModeGenTest2();
	//double** bmodArray = bModeGen();

	return ConvertMatto2DArray(bmodMat);

	//cv::imwrite("C:/Users/navee/Documents/projects/USI_processing/BmodeinCUDA/copy/CudaRuntime/outputs/bmodMat.png", bmodMat);
	//std::cout << "size of bmodMat : " << bmodMat.rows << " , " << bmodMat.cols << std::endl;
	//cv::Mat envolepMat = hilbertTrans4(bmodMat, 1.0);
	//cv::imwrite("C:/Users/navee/Documents/projects/USI_processing/BmodeinCUDA/copy/CudaRuntime/outputs/envolepMat.png", envolepMat);
	//cv::minMaxIdx(envolepMat, &minP, &maxP);
	//std::cout << "range of bmodeMat before log compression: " << minP << " ->" << maxP << std::endl;
	//std::cout << "size of envolepMat : " << envolepMat.rows << " , " << envolepMat.cols << std::endl;
	//// perform image processing 
	//cv::Mat deSpeckledimg;
	//DeSpeckle deNoiseImg(envolepMat);
	//deNoiseImg.applySRAD(envolepMat, deSpeckledimg, 1, 10, 0.25, false, false);
	//cv::imwrite("C:/Users/navee/Documents/projects/USI_processing/BmodeinCUDA/copy/CudaRuntime/outputs/dspeckledimg.png", deSpeckledimg);
	//cv::minMaxIdx(deSpeckledimg, &minP, &maxP);
	//std::cout << "range of deSpeckledimg : " << minP << " ->" << maxP << std::endl;
	//// // log compression
	////cv::Mat logcMat = logTransform(envolepMat);
	////cv::imwrite("./outputs/logcMat.png", logcMat);
	////cv::minMaxIdx(logcMat, &minP, &maxP);
	////std::cout << "range of bmodeMat after log compression1: " << minP << " ->" << maxP << std::endl;
	//cv::Mat rangedMat = dynamicRangeAdjust(deSpeckledimg, 100.0);
	//cv::imwrite("C:/Users/navee/Documents/projects/USI_processing/BmodeinCUDA/copy/CudaRuntime/outputs/rangedMat.png", rangedMat);
	//cv::minMaxIdx(rangedMat, &minP, &maxP);
	//std::cout << "range of bmodeMat after range adjust: " << minP << " ->" << maxP << std::endl;      
	//cv::Mat displayMat = displayRangeAdjust(rangedMat);
	//cv::minMaxIdx(displayMat, &minP, &maxP);
	//std::cout << "range of bmodeMat after disply range adjust: " << minP << " ->" << maxP << std::endl;
	//cv::imwrite("C:/Users/navee/Documents/projects/USI_processing/BmodeinCUDA/copy/CudaRuntime/outputs/b_mode_w_speckle_Red.png", displayMat);
	//cv::imshow("B-mode image", displayMat);
	//cv::waitKey();
}

//cv::Mat testCLass::testFun() {
//	cv::Mat outMat = cv::Mat::zeros(1024 - 300, 254, CV_64FC1);
//	return outMat;
//}