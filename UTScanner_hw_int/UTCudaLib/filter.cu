#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <fstream>
#include <vector>
#include <stdio.h>
#include <string>
#include <iostream>
#include <exception>
#include <chrono>
#include <math.h>
#include <complex>
#include <hipfft/hipfft.h>      /// From "cufft.lib" 
#include <array>
#include <iomanip>
#include <opencv2/opencv.hpp>

void read_data2(const char* path, float* outArray, int rows, int cols, bool display) {
    /// -------------------READING THE DATA FILE------------------------------------------

    std::string line, word;
    std::ifstream wholeStream;
    std::stringstream lineStream;
    std::cout << "rows : " << rows << "cols : " << cols << std::endl;
    wholeStream.open(path, std::ios::in);

    if (wholeStream.is_open()) {
        int i = 0, j = 0;
        float v;
        while (getline(wholeStream, line)) {

            // convert string line to line stream
            lineStream = std::stringstream(line);
            while (getline(lineStream, word, ',')) {
                //std::cout << "word :" << word;
                outArray[i] = stof(word);
                i++;
            }
        }
    }
    wholeStream.close();


    if (display) {
        std::cout << "----------------" << std::endl;
        for (int i = 0; i < rows; i++) {
            for (int j = 0; j < cols; j++) {
                std::cout << outArray[cols * i + j] << ", ";
            }
            std::cout << std::endl;
        }
    }
}

__global__ void real2complex(float* f, hipfftComplex* fc) {
    //int i = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    fc[i].x = f[i];
    fc[i].y = 0.0f;
}

__global__ void splitComplex(hipfftComplex* inComplex, float* outReal, float* outImag) {

    int i = threadIdx.x;
    outReal[i] = inComplex[i].x;
    outImag[i] = inComplex[i].y;

}

__global__ void scalarMult(float* inArray, float* outArray, float c) {
    int i = threadIdx.x;

    outArray[i] = inArray[i] * c;

}

__global__ void magnitide(float* inX, float* inY, float* outW) {
    int i = threadIdx.x;

    outW[i] = std::sqrtf(std::pow(inX[i], 2) + std::pow(inY[i], 2));

}

cv::Mat converttoMat(float* imgArray, int rows, int cols) {

    // Converting the B-mode image into OpenCV Mat
    cv::Mat outMat = cv::Mat::zeros(rows, cols, CV_32FC1);
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            outMat.at<float>(i, j) = imgArray[i * cols + j];
        }

    }

    return outMat;
}

class cudaDisplay
{
private:

    hipfftHandle plan;
    int NBK;
    dim3 BKZ;
    float* d_xflat, * d_ifftI, * d_ifftR, * d_envelop;
    hipfftComplex* d_xflatComplex;
    hipfftComplex* d_fftComplex;
    hipfftComplex* d_ifftComplex;
    hipfftComplex* xflatComplex;
    hipfftComplex* fftComplex;
    hipfftComplex* ifftComplex;

public:
	void init(int rows, int cols) {

		// init the cufft handles here
        NBK = cols;
        BKZ = dim3(rows);
        hipfftPlan2d(&plan, cols, rows, HIPFFT_C2C);

        hipMalloc((void**)&d_xflat, sizeof(float) * rows * cols);
        hipMalloc((void**)&d_ifftI, sizeof(float) * rows * cols);
        hipMalloc((void**)&d_ifftR, sizeof(float) * rows * cols);
        hipMalloc((void**)&d_envelop, sizeof(float) * rows * cols);

        xflatComplex = new hipfftComplex[rows * cols];
        fftComplex = new hipfftComplex[rows * cols];
        ifftComplex = new hipfftComplex[rows * cols];
        hipMalloc((void**)&d_fftComplex, sizeof(hipfftComplex) * rows * cols);
        hipMalloc((void**)&d_ifftComplex, sizeof(hipfftComplex) * rows * cols);
        hipMalloc((void**)&d_xflatComplex, sizeof(hipfftComplex) * rows * cols);

        
	}

	void fetchEnvolep(float* inImg, float* outEnvolep, int rows, int cols) {
		// calculate the hilber transform here
        hipMemcpy(d_xflat, inImg, sizeof(float) * rows * cols, hipMemcpyHostToDevice); //input in device
        real2complex << <NBK, BKZ >> > (d_xflat, d_xflatComplex);
        hipfftExecC2C(plan, d_xflatComplex, d_fftComplex, HIPFFT_FORWARD);
        hipfftExecC2C(plan, d_fftComplex, d_ifftComplex, HIPFFT_BACKWARD);
        // convert t real and imaginary parts
        splitComplex << <NBK, BKZ >> > (d_ifftComplex, d_ifftR, d_ifftI);
        scalarMult << <NBK, BKZ >> > (d_ifftI, d_ifftI, (float)(1.0 / rows));
        magnitide << <NBK, BKZ >> > (d_xflat, d_ifftI, d_envelop);

        hipMemcpy(outEnvolep, d_envelop, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
	}

};

int main()
{
    int rows = 1024;
    int cols = 256;
    float* xflat = new float[rows * cols];
    float* yflat = new float[rows * cols];
    cv::Mat envolepMat;
    const char* data_path = "./inputs/b_curve_mode.csv";
    read_data2(data_path, xflat, rows, cols, false);

    cudaDisplay cudaDisplayHandle = cudaDisplay();
    cudaDisplayHandle.init(rows, cols);
    cudaDisplayHandle.fetchEnvolep(xflat, yflat, rows, cols);

    std::cout << "--------- envelop------------" << std::endl;
    std::cout << std::setprecision(2);
    for (int j = 0; j < rows*cols; j++) {
            std::cout << yflat[j] << ",";
            if ((j + 1) % rows == 0)
                std::cout << std::endl;
    }

    envolepMat = converttoMat(yflat, rows, cols);

    cv::imwrite("./sample_outputs/envolepMat_cuda.png", envolepMat);
    
    return 0;
}