#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>      /// From "cufft.lib" 
#include "hip/hip_runtime.h"
#include <fstream>
#include "testheader.cuh"
#include ""


__global__ void range(int* out_data, int min, int arr_size, int inc)	//creates an array of a range of values
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				//max=final value of the array
	if (i < arr_size)																//arr_size==array size
	{																			//inc=increment needed
		out_data[i] = min + (i * inc);
	}
}

__global__ void range(float* out_data, float min, int arr_size, float inc)	//creates an array of a range of values
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				//max=final value of the array
	if (i < arr_size)																//arr_size==array size
	{																			//inc=increment needed
		out_data[i] = min + (i * inc);
	}
}

__global__ void range(double* out_data, double min, int arr_size, double inc)	//creates an array of a range of values
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				//max=final value of the array
	if (i < arr_size)																//arr_size==array size
	{																			//inc=increment needed
		out_data[i] = min + (i * inc);
	}
}

__global__ void range(long double* out_data, long double min, int arr_size, long double inc)	//creates an array of a range of values
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				//max=final value of the array
	if (i < arr_size)																//arr_size==array size
	{																			//inc=increment needed
		out_data[i] = min + (i * inc);
	}
}

__global__ void element_division(float* mat_in, float value, int size, float* mat_out)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				   //max=final value of the array
	if (i < size)
	{
		mat_out[i] = mat_in[i] / value;
	}
}

__global__ void element_division(long double* mat_in, float value, int size, long double* mat_out)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;								//min=starting value of array
																				   //max=final value of the array
	if (i < size)
	{
		mat_out[i] = mat_in[i] / value;
	}
}

__global__ void aperture_distance(float* mat1, float* mat2, int Nx, int channels, float* mat_out)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	int i = x / channels;
	int j = x % channels;

	if (x < Nx * channels)
	{
		mat_out[i * channels + j] = fabs(mat1[i] - mat2[j]);
	}
}

__global__ void apodization(float* distance, float* aperture, int Nz, int Nx, int channels, int pixels, float* apod)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int i = x / Nz;
	int ii = i % Nx;
	int j = x % Nz;
	int nrx = x / pixels;
	float PI = 3.14159;

	if (x < pixels * channels)
	{
		bool temp = distance[ii * channels + nrx] <= (aperture[j] / 2);
		apod[i * Nz + j] = temp * (0.5 + 0.5 * cos(2 * PI * distance[ii * channels + nrx] / aperture[j]));
	}
}

// This function calculates TX central aperture position
__global__ void Tx_cen_pos(float* cen_pos, int N_elements, int N_active, float pitch, int skip_frames, int num_frames, float* probe)
{

	int x = threadIdx.x;

	if (x < num_frames)
	{
		//cen_pos[x] = pitch * ((N_active / 2) + (N_active * (x)-N_elements / 2));
		cen_pos[x] = probe[x * skip_frames + 4];
	}
}

// receive_delay calculation
__global__ void receive_delay(float* probe_ge_x, float* x_axis1, float* z_axis1, int channels, int Nx, int Nz, float del_convert, float* rx_delay)
{

}

//  transmit_delay calculation
__global__ void transmit_delay(float* x_axis1, float* z_axis1, float* k1, float zd, int Nx, int Nz, float del_convert, int num_frames, float* tx_delay)
{

}

__global__ void beamformingLR3(float* beamformed_data1, float* tx_delay, float* rx_delay, float* data, float* rx_apod, int samples, int pixels, int f, int num_frames, int channels)
{

}

__global__ void isnan_test_array(float* data, int size)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < size)
	{
		if (isnan(data[idx]) == 1)
		{
			data[idx] = 0;
		}
		else
		{
			data[idx] = data[idx];
		}

	}


}

__global__ void BPfilter1SharedMem(float* in, float* filt_coeff, int pixels, float* y1) {


}

__global__ void reshape_columnwise(int col, int row, float* beamformed_data_reshaped, float* d_bfHR)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	int i = x / row;
	int j = x % row;

	if (x < col * row)
	{
		beamformed_data_reshaped[j * col + i] = d_bfHR[x];

	}
}

//** <Curvilinear Prob> **//

// receive_delay calculation
__global__ void receive_delay(float* theta, float* theta1, float rc, float* z_axis, int channels, int Nx, int Nz, float del_convert, float* rx_delay)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;


}

__global__ void theta1(float* theta_active, float* theta, int frames, int N_active, int skip_frames)
{

	int x = threadIdx.x;
	int f = 0;
	for (int i = 1; i <= frames; i += skip_frames)
	{
		theta_active[f] = theta[i + 3 - 1];
		f++;
	}
}

//  transmit_delay calculation
__global__ void transmit_delay(float* theta1, float* z_axis, float rc, float* theta_tx, int Nx, int Nz, float del_convert, int columns, float zd, float* tx_delay)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int i = x / Nz;
	int j = x % Nz;
	int f = x / (Nx * Nz);



}

__global__ void add_ele(float* data, int pixels, float* out_data)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < pixels)
	{
		out_data[x] += data[x];
	}
}

__global__ void sample1(float* tx_delay, float* rx_delay, int pixels, int channels, float c, float sampling_frequency, float* data1, float* rx_apod, float* data, int samples, int columns)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int f = blockDim.y * blockIdx.y + threadIdx.y;
	int nrx = x / pixels; //channels
	int pix = x % pixels; //pixels


	if (f < columns)
	{
		float delay = ((float)tx_delay[f * pixels + pix] + (float)rx_delay[(nrx % channels) * pixels + pix]) / c;
		float p = delay * sampling_frequency;
		int pixel_pos = round(p);

		//data1 = rx_apod[(nrx % channels) + (pix * channels)] * data[((nrx % channels) * samples + pixel_pos - 1) * columns + f];
	}
}

__global__ void add_columns_matrix(float* data, int columns, int pixels, float* out_data)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < pixels)
	{
		for (int f = 0; f < columns; f++)
			out_data[x] += data[x * columns + f];
	}
}

__global__ void parallel_try(float* tx_delay, float* rx_delay, float sampling_frequency, float c, int samples,
	int channels, int columns, float* rx_apod, int pixels, float* data, float* beamformed_data)
{
	//__shared__ double* beamformed_data_1;
	int pix = blockDim.x * blockIdx.x + threadIdx.x;
	int nrx = blockDim.y * blockIdx.y + threadIdx.y;
	//int f = x / pixels; int pix = x % pixels; //int nrx = x % 128;
	int f = blockDim.z * blockIdx.z + threadIdx.z;
	//int nrx = x / pixels; //channels
	//int pix = x % pixels; //pixels
	//int f = x / (pixels * channels);

	if (f < columns && pix < pixels && nrx < channels)
	{
		//for (int nrx = 0; nrx < channels; nrx++)
		//{

		float delay = ((float)tx_delay[f * pixels + pix] + (float)rx_delay[(nrx)*pixels + pix]) / c;
		float p = delay * sampling_frequency;
		int pixel_pos = round(p);

		if ((0 < pixel_pos) && (pixel_pos < samples))
		{
			//double ans= beamformed_data[pix] + rx_apod[channels * nrx + pix] * data[nrx * 2600 + pixel_pos];
			beamformed_data[pix * columns + f] += rx_apod[nrx + (pix * channels)] * data[(nrx * samples + pixel_pos - 1) * columns + f];

		}
		//}
	}
}

//cudaBackEnd::cudaBackEnd() {
//
//	int num_threads = 1024;
//	float rx_f_number = 2.0;
//	int samples = 2040;						// # of samples in depth direction
//	int N_elements = 128;					// # of transducer elements
//	float sampling_frequency = 32e6;		// sampling frequency
//	float c = 1540.0;						// speed of sound [m/s]	
//	int N_active = 8;						// Active transmit elmeents
//	float pitch = 0.465 / 1000;				// spacing between the elements
//	float aper_len = (N_elements - 1) * pitch * 1000;	//aperture foot print 
//	float zd = pitch * N_active / (float)2;				// virtual src distance from transducer array 
//	float sample_spacing = c / sampling_frequency / (float)2;
//	float del_convert = sampling_frequency / c;			// used in delay calculation
//	float rc = 60.1 / 1000;					// radius_of_curvature
//	float scan_angle = (58 * PI) / 180;
//	int channels = 128;						// number of A-lines data used for beamforming
//	int Nx = 256;							// 256 Lateral spacing
//	int Nz = 1024;							//1024 Axial spacing
//	int pixels = Nz * Nx;
//	int pix_cha = pixels * channels;		// Nz*Nx*128 This array size is used for Apodization
//	int frames = 121;
//	int num_frames = 121;					// number of low resolution images
//	int skip_frames = 1;					// 
//	int dBvalue = 60;
//	float rximg[128 * 2040];
//	int croppedBot = 300;
//	//float* filt_coeff = new float[364];
//	//float* d_z_axis = 0;
//	//float* d_x_axis = 0;
//	//float* d_probe = 0;
//	//float* d_rx_aperture = 0;
//	//float* d_rx_ap_distance = 0;
//	//float* d_cen_pos = 0;
//	//float* d_data = 0;   // variable to store raw rf data
//	//float* d_bfHR = 0;  // variable to store beamformed high-resolution beamformed image 
//	//float* d_tx_delay = 0;
//	//float* d_rx_delay = 0;// delay calculation
//	//float* d_rx_apod = 0; //apodization
//	//float* d_filt_coeff = 0; //to read filter coeff CSV
//	//float* d_bfHRBP = 0;  // variable to store beamformed high-resolution bandpass filtered data
//	//float* dev_beamformed_data1 = 0;   // variable to store reshaped beamformed data
//	//float* env = new float[pixels]; // Host memory variable to store beamformed high-resolution bandpass filtered data
//	//float* d_theta = 0;
//	//float* d_theta1 = 0;
//	//float* d_theta_tx = 0;
//
//	USBDevice = new CCyUSBDevice(NULL);
//	// Obtain the control endpoint pointer
//	ept = USBDevice->ControlEndPt;
//	if (!ept) {
//		printf("Could not get Control endpoint.\n");
//		//return 1;
//	}
//
//}

//void cudaBackEnd::read_csv_array_test(float* data, char* filename)
//	{
//		char buffer[6240];  //6240
//		char* token;
//		int i = 0;
//		FILE* file;
//
//		file = fopen(filename, "r");
//		if (file == NULL)
//		{
//			throw std::exception("File did not open");
//		}
//
//		while (fgets(buffer, sizeof(buffer), file) != 0)            // end-of-file indicator
//		{
//			token = strtok(buffer, ",");
//			//j = 0;
//			while (token != NULL)
//			{
//				data[i] = atof(token);     //converts the string argument str to float
//				token = strtok(NULL, ",");
//				//j++;
//			}
//
//			i++;
//		}
//		fclose(file);
//		// printf("Complete reading from file %s\n", filename);
//
//	}
//
//void cudaBackEnd::setMemmory(int a) {
//
//		read_csv_array_test(filt_coeff, "b_10M.csv");    // csv file read
//		hipMalloc((void**)&d_filt_coeff, sizeof(float) * MASK_WIDTH);
//		hipMemcpy(d_filt_coeff, filt_coeff, sizeof(float) * MASK_WIDTH, hipMemcpyHostToDevice);
//
//		mfile.open("sample_output/test.txt");
//		mfile << "Test OK" << std::endl;
//		mfile.close();
//
//	}

void cudaBackEnd::wait(unsigned timeout)
{
	timeout += std::clock();
	while (std::clock() < timeout) continue;
}

void cudaBackEnd::write_rows(CCyControlEndPoint* ept, unsigned char* ptr, unsigned int numRows)
{
	int len = numRows * 16; // each row is 16 bytes to send
	int sent = 0;
	unsigned char* tPtr = ptr;
	while (sent < len) {
		LONG buflen = len - sent;
		if (buflen > 192) buflen = 192;
		//for (int i = 0; i < buflen; i += 16) {
		//	for (int j = 0; j < 16; j++) {
		//		printf("%02X\t ", ptr[i + j]);
		//	}
		//	printf("\n");
		//}
		ept->XferData(tPtr, buflen);
		tPtr += buflen;
		sent += buflen;
	}
	//printf("Sent %d bytes to EPT\n", sent);
}

bool cudaBackEnd::read_chunk(CCyBulkEndPoint* ept_in, unsigned char* recvBuf, LONG& length)
{
	bool result;
	LONG intlen = length;
	result = ept_in->XferData(recvBuf, intlen, NULL, true);
	//if (result) {
	//	printf("Received data ------------- : %d\n", intlen);
	//}
	//else {
	//	printf("***   ERROR receiving data - expected %d, got %d\n", length, intlen);
	//}
	ept_in->Abort();
	ept_in->Reset();
	return result;
}

int cudaBackEnd::insert_row(unsigned char* buf, int row, short addr, int data)
{
	int o = row * 16;
	buf[o] = 0xff; buf[o + 1] = 0xaa; buf[o + 2] = 0x01; buf[o + 3] = 0x07;
	buf[o + 4] = 0x00; buf[o + 5] = 0x00; buf[o + 6] = 0x00; buf[o + 7] = 0x01;
	buf[o + 8] = (addr & 0xff); buf[o + 9] = (addr & 0xff00) >> 8;
	buf[o + 10] = (data & 0xff); buf[o + 11] = (data & 0xff00) >> 8;
	buf[o + 12] = (data & 0xff0000) >> 16; buf[o + 13] = (data & 0xff000000) >> 24;
	buf[o + 14] = 0x00; buf[o + 15] = 0x00;
	return row + 1;
}

void cudaBackEnd::read_csv_mat(float* data, char* filename, int col1)
{
	char buffer[6240];  //6240
	char* token;

	int i = 0, j = 0;
	FILE* file;
	file = fopen(filename, "r");
	if (file == NULL)
	{
		// printf("Can't open the file");
	}
	else
	{
		while (fgets(buffer, sizeof(buffer), file) != 0)            // end-of-file indicator
		{
			token = strtok(buffer, ",");
			j = 0;
			while (token != NULL)
			{
				data[i * col1 + j] = atof(token);     //converts the string argument str to float
				token = strtok(NULL, ",");
				j++;
			}

			i++;
		}
		fclose(file);
		// printf("Complete reading from file %s\n", filename);
	}
}

void cudaBackEnd::read_csv_mat(long double* data, char* filename, int col1)
{
	char buffer[6240];  //6240
	char* token;

	int i = 0, j = 0;
	FILE* file;
	file = fopen(filename, "r");
	if (file == NULL)
	{
		// printf("Can't open the file");
	}
	else
	{
		while (fgets(buffer, sizeof(buffer), file) != 0)            // end-of-file indicator
		{
			token = strtok(buffer, ",");
			j = 0;
			while (token != NULL)
			{
				data[i * col1 + j] = atof(token);     //converts the string argument str to float
				token = strtok(NULL, ",");
				j++;
			}

			i++;
		}
		fclose(file);
		// printf("Complete reading from file %s\n", filename);
	}
}

void cudaBackEnd::read_csv_array(float* data, char* filename)
{
	char buffer[6240];  //6240
	char* token;
	int i = 0;
	FILE* file;

	file = fopen(filename, "r");
	if (file == NULL)
	{
		throw std::exception("File did not open");
	}

	while (fgets(buffer, sizeof(buffer), file) != 0)            // end-of-file indicator
	{
		token = strtok(buffer, ",");
		//j = 0;
		while (token != NULL)
		{
			data[i] = atof(token);     //converts the string argument str to float
			token = strtok(NULL, ",");
			//j++;
		}

		i++;
	}
	fclose(file);
	// printf("Complete reading from file %s\n", filename);

}

void cudaBackEnd::zeroC(float* bfHR, int pixels)
{
	for (int j = 0; j < pixels; j++)
	{
		bfHR[j] = 0;
	}
}

//** <Curvilinear Prob> **//

void cudaBackEnd::csv_write_mat(long double* a, const char* filename, int row1, int col1)		//writes data to memory
{
	FILE* fp;
	int i;

	fp = fopen(filename, "w+");

	for (i = 0; i < row1; ++i)
	{
		for (int j = 0; j < col1; j++)
		{
			if (j == col1 - 1)					//for the last value in the column "," is not appended
			{									//matrix dimension error can occur with the presence of extra comma at last of the column
				fprintf(fp, "%g", a[i * col1 + j]);
			}
			else
				fprintf(fp, "%g,", a[i * col1 + j]);
		}



		fprintf(fp, "\n");
	}


	fclose(fp);
	// printf("\n %s file is created\n", filename);
}

void cudaBackEnd::csv_write_mat(double* a, const char* filename, int row1, int col1)		//writes data to memory
{
	FILE* fp;
	int i;

	fp = fopen(filename, "w+");

	for (i = 0; i < row1; ++i)
	{
		for (int j = 0; j < col1; j++)
		{
			if (j == col1 - 1)					//for the last value in the column "," is not appended
			{									//matrix dimension error can occur with the presence of extra comma at last of the column
				fprintf(fp, "%g", a[i * col1 + j]);
			}
			else
				fprintf(fp, "%g,", a[i * col1 + j]);
		}



		fprintf(fp, "\n");
	}


	fclose(fp);
	printf("\n %s file is created\n", filename);
}

void cudaBackEnd::csv_write_mat(float* a, const char* filename, int row1, int col1)	//for writing integer data "FUNCTION OVERLOADING"
{
	FILE* fp;
	int i;

	fp = fopen(filename, "w+");

	for (i = 0; i < row1; ++i)
	{
		for (int j = 0; j < col1; j++)
		{
			if (j == col1 - 1)
			{
				fprintf(fp, "%f", a[i * col1 + j]);
			}
			else

				fprintf(fp, "%f,", a[i * col1 + j]);
		}



		fprintf(fp, "\n");
	}


	fclose(fp);
	// printf("\n %s file is created\n", filename);
}

double** cudaBackEnd::convertsingto2darray(float* imgArray, int rows, int cols) {

	double** array2D = (double**)malloc(rows * sizeof(double*));
	for (int i = 0; i < rows; i++) {
		array2D[i] = (double*)malloc(cols * sizeof(double));
	}

	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			array2D[i][j] = (double)imgArray[i * cols + j];
		}
	}

	return array2D;
}

int cudaBackEnd::num_threads = 1024;
int cudaBackEnd::N_active = 8;							// Active transmit elmeents
int cudaBackEnd::samples = 2040;						// # of samples in depth direction
int cudaBackEnd::N_elements = 128;						// # of transducer elements
float cudaBackEnd::rx_f_number = 2.0;
float cudaBackEnd::PI = 3.14;
float cudaBackEnd::sampling_frequency = 32e6;			// sampling frequency
float cudaBackEnd::c = 1540.0;							// speed of sound [m/s]	
float cudaBackEnd::pitch = 0.465 / 1000;				// spacing between the elements
float cudaBackEnd::aper_len = (N_elements - 1) * pitch * 1000;	//aperture foot print 
float cudaBackEnd::zd = pitch * N_active / (float)2;			// virtual src distance from transducer array 
float cudaBackEnd::sample_spacing = c / sampling_frequency / (float)2;
float cudaBackEnd::del_convert = sampling_frequency / c;		// used in delay calculation
float cudaBackEnd::rc = 60.1 / 1000;					// radius_of_curvature
float cudaBackEnd::scan_angle = (58 * PI) / 180;
int cudaBackEnd::channels = 128;						// number of A-lines data used for beamforming
int cudaBackEnd::Nx = 256;								// 256 Lateral spacing
int cudaBackEnd::Nz = 1024;								//1024 Axial spacing
int cudaBackEnd::pixels = Nz * Nx;
int cudaBackEnd::pix_cha = pixels * channels;			// Nz*Nx*128 This array size is used for Apodization
int cudaBackEnd::frames = 121;
int cudaBackEnd::num_frames = 121;						// number of low resolution images
int cudaBackEnd::skip_frames = 1;						// 
int cudaBackEnd::dBvalue = 60;

float* cudaBackEnd::filt_coeff = new float[364];
float* cudaBackEnd::env = new float[cudaBackEnd::pixels];
float* cudaBackEnd::d_filt_coeff = 0;
float* cudaBackEnd::d_z_axis = 0;
float* cudaBackEnd::d_x_axis = 0;
float* cudaBackEnd::d_probe = 0;
float* cudaBackEnd::d_rx_aperture = 0;
float* cudaBackEnd::d_rx_ap_distance = 0;
float* cudaBackEnd::d_cen_pos = 0;
float* cudaBackEnd::d_data = 0;
float* cudaBackEnd::d_bfHR = 0;
float* cudaBackEnd::d_tx_delay = 0;
float* cudaBackEnd::d_rx_delay = 0;
float* cudaBackEnd::d_rx_apod = 0;
float* cudaBackEnd::d_bfHRBP = 0;
float* cudaBackEnd::dev_beamformed_data1 = 0;
//** for curveLiner Prob  **//
float* cudaBackEnd::d_theta = 0;
float* cudaBackEnd::d_theta1 = 0;
float* cudaBackEnd::d_theta_tx = 0;
//float cudaBackEnd::rximg[128 * 2040] = { 0 };
float* cudaBackEnd::rximg = new float[cudaBackEnd::N_elements* cudaBackEnd::samples];
FILE* cudaBackEnd::fp = 0;

CCyUSBDevice* cudaBackEnd::USBDevice = new CCyUSBDevice(NULL);
CCyControlEndPoint* cudaBackEnd::ept = cudaBackEnd::USBDevice->ControlEndPt;
CCyBulkEndPoint* cudaBackEnd::ept_in = cudaBackEnd::USBDevice->BulkInEndPt;

int cudaBackEnd::initHW() 
{
	cudaBackEnd::USBDevice	= new CCyUSBDevice(NULL);
	cudaBackEnd::ept		= cudaBackEnd::USBDevice->ControlEndPt;
	cudaBackEnd::ept_in		= cudaBackEnd::USBDevice->BulkInEndPt;

	if (!cudaBackEnd::ept) {
		//printf("Could not get Control endpoint.\n");
		return 3;
	}

	if (!cudaBackEnd::ept_in) {
		//printf("No IN endpoint??\n");
		return 4;
	}

	// Send a vendor request (bRequest = 0x05) to the device
	cudaBackEnd::ept->Target	= TGT_DEVICE;
	cudaBackEnd::ept->ReqType	= REQ_VENDOR;
	cudaBackEnd::ept->Direction = DIR_TO_DEVICE;
	cudaBackEnd::ept->ReqCode	= 0x05;
	cudaBackEnd::ept->Value		= 1;
	cudaBackEnd::ept->Index		= 0;
	cudaBackEnd::ept->TimeOut	= 100;				// set timeout to 100ms for quick response

	cudaBackEnd::ept_in->MaxPktSize = 16384;
	cudaBackEnd::ept_in->TimeOut	= 100;			// set timeout to 100ms for readin

	return 0;
}

int cudaBackEnd::initSettingFile(const char* path)
{
	errno_t err;
	//FILE* fp;
	// path = "out25_curvi.txt"; for curvilieanr prob
	if ((err = fopen_s(&cudaBackEnd::fp, path, "r")) != 0) {
		//printf("Could not open config file for reading.\n");
		return 3;
	}

	return 0;
}

int cudaBackEnd::initGPUprobeC(double* prob_params) {

	const int MASK_WIDTH = 364;
	std::ofstream mFile;
	mFile.open("sample_output/testclass.txt");
	mFile << "OK" << std::endl;
	//------------------------------------

	try 
	{
		cudaBackEnd::PI = (float)prob_params[3];	//mFile2 << PI << std::endl;
		//const int MASK_WIDTH = (int)probPrms[4];// mFile2 << MASK_WIDTH << std::endl;
		//const int TILE_SIZE = (int)prob_params[5]; //mFile2 << TILE_SIZE << std::endl;
		cudaBackEnd::num_threads = (int)prob_params[6];		//mFile2 << num_threads << std::endl;
		cudaBackEnd::rx_f_number = (float)prob_params[7];	//mFile2 << rx_f_number << std::endl;
		cudaBackEnd::samples = (int)prob_params[8];		///mFile2 << samples << std::endl;
		cudaBackEnd::N_elements = (int)prob_params[9];		//mFile2 << N_elements << std::endl;
		cudaBackEnd::sampling_frequency = (float)prob_params[10]; ///mFile2 << sampling_frequency << std::endl;
		cudaBackEnd::c = (float)prob_params[11];	//mFile2 << c << std::endl;
		cudaBackEnd::N_active = (int)prob_params[12];		//mFile2 << N_active << std::endl;
		cudaBackEnd::channels = (int)prob_params[13];		//mFile2 << channels << std::endl;
		cudaBackEnd::Nx = (int)prob_params[14];		//mFile2 << Nx << std::endl;
		cudaBackEnd::Nz = (int)prob_params[15];		//mFile2 << Nz << std::endl;
		cudaBackEnd::frames = (int)prob_params[16];		//mFile2 << frames << std::endl;
		cudaBackEnd::num_frames = (int)prob_params[17];		//mFile2 << num_frames << std::endl;
		cudaBackEnd::skip_frames = (int)prob_params[18];
		cudaBackEnd::dBvalue = (int)prob_params[19];
		cudaBackEnd::pitch = (float)prob_params[20];
		cudaBackEnd::aper_len = (float)prob_params[21];
		cudaBackEnd::zd = (float)prob_params[22];
		cudaBackEnd::sample_spacing = (float)prob_params[23];
		cudaBackEnd::del_convert = (float)prob_params[24]; ;
		cudaBackEnd::rc = (float)prob_params[25];
		cudaBackEnd::scan_angle = (float)prob_params[26];
		cudaBackEnd::pixels = (int)prob_params[27];
		cudaBackEnd::pix_cha = (int)prob_params[28];

		char filename1[200];
		sprintf(filename1, "b_10M.csv");
		cudaBackEnd::read_csv_array(cudaBackEnd::filt_coeff, filename1);    // csv file read

		hipMalloc((void**)&cudaBackEnd::d_filt_coeff, sizeof(float) * MASK_WIDTH);
		hipMemcpy(cudaBackEnd::d_filt_coeff, filt_coeff, sizeof(float) * MASK_WIDTH, hipMemcpyHostToDevice);

		////////  Intialization &(or) Memory allocation  //////////////////
		hipMalloc((void**)&cudaBackEnd::d_data, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels);// variable to store raw rf data

		hipMalloc((void**)&cudaBackEnd::d_bfHR, cudaBackEnd::pixels * sizeof(float)); // variable to store beamformed high-resolution beamformed image 
		hipMemset(cudaBackEnd::d_bfHR, 0, cudaBackEnd::pixels * sizeof(float));

		hipMalloc((void**)&cudaBackEnd::dev_beamformed_data1, cudaBackEnd::pixels * sizeof(float));// variable to store reshaped beamformed data

		hipMalloc((void**)&cudaBackEnd::d_bfHRBP, sizeof(float) * cudaBackEnd::pixels);// variable to store beamformed high-resolution bandpass filtered data

		/////////////////// theta positions for all elements ////////////////////
		hipMalloc((void**)&cudaBackEnd::d_theta, cudaBackEnd::N_elements * sizeof(float));
		range << <cudaBackEnd::Nx / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_theta, (-cudaBackEnd::scan_angle / 2), cudaBackEnd::N_elements, (cudaBackEnd::scan_angle / (cudaBackEnd::N_elements - 1)));


		///////////// theta for grid /////////////////  theta = -scan_angle / 2 : scan_angle / (elements - 1) : scan_angle / 2;
		hipMalloc((void**)&cudaBackEnd::d_theta1, cudaBackEnd::Nx * sizeof(float));
		range << <cudaBackEnd::Nx / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_theta1, (-cudaBackEnd::scan_angle / 2), cudaBackEnd::Nx, (cudaBackEnd::scan_angle / (cudaBackEnd::Nx - 1)));
		hipGetLastError();
		hipDeviceSynchronize();

		////////////// z value////////////////////
		float dz = cudaBackEnd::sample_spacing * cudaBackEnd::samples / cudaBackEnd::Nz;  // depth / (Nz - 1) / 1000;   // spacing in axial (z) direction in mm;
		hipMalloc((void**)&cudaBackEnd::d_z_axis, cudaBackEnd::Nz * sizeof(float));
		range << <cudaBackEnd::Nz / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_z_axis, 0, cudaBackEnd::Nz, dz);
		hipGetLastError();
		hipDeviceSynchronize();

		//////////////////////////////// x value////////////////////////////////
		float dx = aper_len / (Nx - 1);
		hipMalloc((void**)&cudaBackEnd::d_x_axis, cudaBackEnd::Nx * sizeof(float));
		range << <cudaBackEnd::Nx / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_x_axis, (-cudaBackEnd::aper_len / 2000), cudaBackEnd::Nx, dx / 1000);
		hipGetLastError();
		hipDeviceSynchronize();

		//////////////// Probe geometry, this info can be taken from transducer file ////////////////////
		hipMalloc((void**)&cudaBackEnd::d_probe, cudaBackEnd::N_elements * sizeof(float));
		range << <1, cudaBackEnd::N_elements >> > (cudaBackEnd::d_probe, (-cudaBackEnd::aper_len / 2000), cudaBackEnd::N_elements, cudaBackEnd::pitch);
		hipGetLastError();
		hipDeviceSynchronize();

		/////////////////rx aerture calculation using Fnumber///////////////////////////////
		hipMalloc((void**)&cudaBackEnd::d_rx_aperture, cudaBackEnd::Nz * sizeof(float));
		element_division << <cudaBackEnd::Nz / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_z_axis, cudaBackEnd::rx_f_number, cudaBackEnd::Nz, cudaBackEnd::d_rx_aperture);
		hipGetLastError();
		hipDeviceSynchronize();

		////////////////////////rx aerture distance////////
		hipMalloc((void**)&cudaBackEnd::d_rx_ap_distance, cudaBackEnd::channels * cudaBackEnd::Nx * sizeof(float));  //20.087 us
		aperture_distance << <cudaBackEnd::Nx * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_x_axis, cudaBackEnd::d_probe, cudaBackEnd::Nx, cudaBackEnd::channels, cudaBackEnd::d_rx_ap_distance);
		hipGetLastError();
		hipDeviceSynchronize();

		///////////////////apodization/////////////////
		hipMalloc((void**)&cudaBackEnd::d_rx_apod, sizeof(float) * cudaBackEnd::Nz * cudaBackEnd::channels * cudaBackEnd::Nx);
		apodization << <cudaBackEnd::pixels * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_rx_ap_distance, cudaBackEnd::d_rx_aperture, cudaBackEnd::Nz, cudaBackEnd::Nx, cudaBackEnd::channels, cudaBackEnd::pixels, cudaBackEnd::d_rx_apod);
		hipGetLastError();
		hipDeviceSynchronize();

		//// check for nan values,
		isnan_test_array << <cudaBackEnd::pixels * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_rx_apod, cudaBackEnd::pixels * cudaBackEnd::channels);
		hipGetLastError();
		hipDeviceSynchronize();

		hipFree(cudaBackEnd::d_rx_aperture);
		hipFree(cudaBackEnd::d_rx_ap_distance);

		/////////////receive delay calculation /////////////////////////////////////////////
		hipMalloc((void**)&cudaBackEnd::d_rx_delay, cudaBackEnd::pix_cha * sizeof(float));
		receive_delay << < cudaBackEnd::pixels * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_theta, cudaBackEnd::d_theta1, cudaBackEnd::rc, cudaBackEnd::d_z_axis, cudaBackEnd::channels, cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::del_convert, cudaBackEnd::d_rx_delay);
		hipGetLastError();
		hipDeviceSynchronize();

		/////////////////// theta positions for all elements ////////////////////
		hipMalloc((void**)&cudaBackEnd::d_theta_tx, cudaBackEnd::num_frames * sizeof(float));
		theta1 << < 1, cudaBackEnd::num_frames >> > (cudaBackEnd::d_theta_tx, cudaBackEnd::d_theta, cudaBackEnd::frames, cudaBackEnd::N_active, cudaBackEnd::skip_frames);

		/////////////////// Transmit delay calculation ////////////////////
		hipMalloc((void**)&cudaBackEnd::d_tx_delay, cudaBackEnd::pixels * cudaBackEnd::num_frames * sizeof(float));
		//transmitter delay for 16 frames,  
		transmit_delay << < cudaBackEnd::pixels * cudaBackEnd::num_frames / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_theta1, cudaBackEnd::d_z_axis, cudaBackEnd::rc, cudaBackEnd::d_theta_tx, cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::del_convert, cudaBackEnd::num_frames, cudaBackEnd::zd, cudaBackEnd::d_tx_delay);
		hipGetLastError();
		hipDeviceSynchronize();

		hipFree(cudaBackEnd::d_theta1);
		hipFree(cudaBackEnd::d_probe);
		hipFree(cudaBackEnd::d_x_axis);
		hipFree(cudaBackEnd::d_z_axis);
		hipFree(cudaBackEnd::d_theta_tx);

		zeroC(cudaBackEnd::rximg, cudaBackEnd::samples * cudaBackEnd::N_elements);   // set rx_img array values to zero.
	}
	catch(std::exception& e)
	{
		return 1;
	}

	return 0;

	mFile << "Memmory init completed" << std::endl;
	mFile.close();
}

double** cudaBackEnd::computeBModeImgC() {

	std::ofstream mFile;
	mFile.open("sample_output/testcomputeimg.txt");
	mFile << "OK" << std::endl;
	//------------------------

	errno_t err;
	char line[MAX_LINE]; // Max possible line length?
	FILE* fp;
	if ((err = fopen_s(&fp, "out25_curvi.txt", "r")) != 0) {
		//printf("Could not open config file for reading.\n");
		return nullptr;
	}
	mFile << "Read config files" << std::endl;
	//----------------------------

	CCyUSBDevice* USBDevice;	// H/W initilization1
	CCyControlEndPoint* ept;	// H/W initilization2
	CCyBulkEndPoint* ept_in;	// Endpoint for reading back data
	mFile << "h/w init done1" << std::endl;

	USBDevice = new CCyUSBDevice(NULL);
	// Obtain the control endpoint pointer
	ept = USBDevice->ControlEndPt;
	if (!ept) {
		printf("Could not get Control endpoint.\n");
		//return 1;
		return nullptr;
	}

	// Send a vendor request (bRequest = 0x05) to the device
	ept->Target = TGT_DEVICE;
	ept->ReqType = REQ_VENDOR;
	ept->Direction = DIR_TO_DEVICE;
	ept->ReqCode = 0x05;
	ept->Value = 1;
	ept->Index = 0;
	ept->TimeOut = 100;				// set timeout to 100ms for quick response
	
	mFile << "h/w init done3" << std::endl;

	ept_in = USBDevice->BulkInEndPt;
	if (!ept_in) {
		//printf("No IN endpoint??\n");
		return nullptr;
	}
	ept_in->MaxPktSize = 16384;
	ept_in->TimeOut = 100;			// set timeout to 100ms for readin

	//-------------------------------------

	int iteration = 0;
	int errcount = 0;
	int row = 0;					// Keep track of how many rows have been added

	const int MAX_LINE = 256;
	const int N_RX = 64;
	unsigned char buf[16 * 1024];

	unsigned int addr, data;
	unsigned char recvbuf[2048 * N_RX * 2];
	const int MAXROWS = 2040;
	LONG rxlen = MAXROWS * N_RX * 2;

	double** outArray; // holds 2d array with B-Mode image

	try
	{
		//unsigned int start = clock();
		while (fgets(line, cudaBackEnd::MAX_LINE, fp)) {
			line[strcspn(line, "\n")] = 0; // Trim trailing newline
			if ((strlen(line) == 0) || (line[0] == ' ') || (line[0] == '#')) {
				//printf("Skipping [%s]\n", line);
			}
			else if (line[0] == 'O') {
				sscanf_s(line, "O %04X %08X ", &addr, &data);
				//printf("Write %08X to Obelix %04X\n", data, addr);
				row = insert_row(buf, row, addr, data);
			}
			else if (line[0] == 'T') {
				sscanf_s(line, "T %04X %08X ", &addr, &data);
				row = insert_row(buf, row, 0x6, 0x40000000 | addr);
				row = insert_row(buf, row, 0x7, data);
				row = insert_row(buf, row, 0x6, 0xC0000000 | addr);
				//printf("Write %08X to TX %04X\n", data, addr);
			}
			else if (line[0] == 'A') {
				sscanf_s(line, "A %04X %08X ", &addr, &data);
				row = insert_row(buf, row, 0x6, 0x00000000 | addr);
				row = insert_row(buf, row, 0x7, data);
				row = insert_row(buf, row, 0x6, 0x80000000 | addr);
				//printf("Write %08X to AFE %04X\n", data, addr);
			}
			else if (line[0] == 'C') {  // CAPTURE STARTS
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x01);
				//write_rows(ept, buf, row);  // Send commands
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x10);
				//write_rows(ept, buf, row);  // Send commands
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x00);

				ept_in->Abort();
				ept_in->Reset();

				write_rows(ept, buf, row);  // Send commands

				//wait(100);
				//row = insert_row(buf, row, 0x4, 0x03);
				//row = insert_row(buf, row, 0x4, 0x10);
				//row = insert_row(buf, row, 0x4, 0x00);
				//printf("CAPTURE %2d: ", iteration);
				//write_rows(ept, buf, row);  // Send commands
				// One iteration should have 2048 samples * 64 channels * 2 bytes each

				wait(1);
				if (read_chunk(ept_in, recvbuf, rxlen)) {
					short* rxdata = (short*)(recvbuf);
					for (int i = 0; i < rxlen / 2; i++) {
						if (rxdata[i] >= 512) rxdata[i] -= 1024;
					}
					// Trying to read only first N-1 rows and discard 1st sample
					for (int i = 0; i < N_RX; i++) {
						for (int j = 0; j < MAXROWS - 1; j++) {
							//rximg[iteration][i][j] = rxdata[j*64+i+2];
							if (iteration < 29) {      // start from 0 index, so 30-1 
								cudaBackEnd::rximg[i * MAXROWS + j] = rxdata[j * N_RX + i + 2];
							}
							else if (iteration > 91) {
								cudaBackEnd::rximg[(i + 64) * MAXROWS + j] = rxdata[j * N_RX + i + 2];
							}
							else {
								cudaBackEnd::rximg[(i + iteration - 28) * MAXROWS + j] = rxdata[j * N_RX + i + 2];
							}
							//rximg[i * MAXROWS + j] = rxdata[j * N_RX + i + 2];
						}
					}
					//saveToFile(iteration, rxlen, recvbuf);
				}
				else {
					errcount++;
				}

				//clock_t begin = clock();   // clock intiated

				hipMemcpy(cudaBackEnd::d_data, cudaBackEnd::rximg, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels, hipMemcpyHostToDevice);

				beamformingLR3 << <(cudaBackEnd::pixels / 256) * cudaBackEnd::channels, 256 >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_tx_delay, cudaBackEnd::d_rx_delay, cudaBackEnd::d_data, cudaBackEnd::d_rx_apod, cudaBackEnd::samples, cudaBackEnd::pixels, iteration, cudaBackEnd::num_frames, cudaBackEnd::channels);
				hipGetLastError();
				hipDeviceSynchronize();

				//clock_t end = clock();
				//float elapsed_secs = float(end - begin) / CLOCKS_PER_SEC;
				//printf("Time for beamforming in ms: %f\n", elapsed_secs * 1000);

				iteration++;	// Increment iteration after saving to image
				row = 0;		// Reset buffer for next iteration
			}
			else {
				printf("Don't know how to handle [%s] yet.\n", line);
			}
		}

		//////////// Bandpass filtering using shared memory /////////////////////
		BPfilter1SharedMem << <(cudaBackEnd::pixels + cudaBackEnd::TILE_SIZE - 1) / cudaBackEnd::TILE_SIZE, cudaBackEnd::TILE_SIZE >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_filt_coeff, cudaBackEnd::pixels, cudaBackEnd::d_bfHRBP);
		hipGetLastError();
		hipDeviceSynchronize();
		//////////////// reshape of the beamformed data ///////////////
		reshape_columnwise << <cudaBackEnd::pixels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::d_bfHRBP);
		hipGetLastError();
		hipDeviceSynchronize();
		hipMemcpy(cudaBackEnd::env, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::Nz * cudaBackEnd::Nx * sizeof(float), hipMemcpyDeviceToHost);
		char fileout[200];
		sprintf(fileout, "sample_output/b_curve_mode.csv"); //all the 16 inputs are arranged in a single file
		csv_write_mat(cudaBackEnd::env, fileout, cudaBackEnd::Nz, cudaBackEnd::Nx);

		mFile << "CSV written" << std::endl;

		outArray = convertsingto2darray(env, Nz, Nx);
	}
	catch (std::exception& e)
	{
		return nullptr;
	}

	mFile << "while loop completed" << std::endl;

	
	//////////////// Free cuda memory (that will be used again) ///////////////
	//hipFree(cudaBackEnd::d_data);
	//hipFree(cudaBackEnd::d_bfHR);
	//hipFree(cudaBackEnd::d_tx_delay);
	//hipFree(cudaBackEnd::d_rx_delay);
	//hipFree(cudaBackEnd::d_rx_apod);
	//hipFree(cudaBackEnd::dev_beamformed_data1);

	// For next iteration
	zeroC(cudaBackEnd::rximg, cudaBackEnd::samples* cudaBackEnd::N_elements);   // set rx_img array values to zero.
	hipMemset(cudaBackEnd::d_bfHR, 0, cudaBackEnd::pixels * sizeof(float));
	hipMemset(cudaBackEnd::dev_beamformed_data1, 0, cudaBackEnd::pixels * sizeof(float));
	
	mFile.close();
	return outArray;
}

int cudaBackEnd::initGPUprobeL(double* prob_params)
{
	// perform b-mode generation here using cuda
	const int TILE_SIZE = prob_params[0];
	int MASK_WIDTH		= prob_params[1];
	const int MAX_LINE	= prob_params[2];


	cudaBackEnd::num_threads	= prob_params[3];
	cudaBackEnd::rx_f_number	= prob_params[4];	// Apodization parameters
	cudaBackEnd::samples		= prob_params[5];	// # of samples in depth direction
	cudaBackEnd::N_elements		= prob_params[6];	// # of transducer elements
	cudaBackEnd::sampling_frequency = prob_params[7];   // sampling frequency
	cudaBackEnd::c				= prob_params[8];	// speed of sound [m/s]	
	cudaBackEnd::N_active		= prob_params[9];   // Active transmit elmeents
	cudaBackEnd::pitch			= prob_params[10];	// spacing between the elements
	cudaBackEnd::aper_len		= prob_params[11];  // aperture foot print 
	cudaBackEnd::zd				= prob_params[12];  // virtual src distance from transducer array 
	cudaBackEnd::sample_spacing	= prob_params[13];
	cudaBackEnd::del_convert	= prob_params[14];  // used in delay calculation
	cudaBackEnd::channels		= prob_params[15];	// number of A-lines data used for beamforming
	cudaBackEnd::Nx				= prob_params[16];	// 256 Lateral spacing Beamforming "Grid" parameters
	cudaBackEnd::Nz				= prob_params[17];	// 1024 Axial spacing
	cudaBackEnd::pixels			= prob_params[18];
	cudaBackEnd::pix_cha		= prob_params[19];	// Nz*Nx*128 This array size is used for Apodization
	cudaBackEnd::num_frames		= prob_params[20];	// number of low resolution images
	cudaBackEnd::skip_frames	= prob_params[21];	//

	// Device and Host memmoey used in initializer
	// float* filt_coeff = new float[MASK_WIDTH];


	try
	{
		//////-<initializing memmory>-///////

		char filename3[200];
		sprintf(filename3, "b_10M.csv");
		read_csv_array(cudaBackEnd::filt_coeff, filename3);    // csv file read
		//cv::imwrite("okMat3.png", testMat0);

		// float* d_filt_coeff = 0;
		hipMalloc((void**)&cudaBackEnd::d_filt_coeff, sizeof(float) * MASK_WIDTH);
		hipMemcpy(cudaBackEnd::d_filt_coeff, cudaBackEnd::filt_coeff, sizeof(float) * MASK_WIDTH, hipMemcpyHostToDevice);

		////////  Intialization &(or) Memory allocation  //////////////////
		hipMalloc((void**)&cudaBackEnd::d_data, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels);// variable to store raw rf data

		hipMalloc((void**)&cudaBackEnd::d_bfHR, cudaBackEnd::pixels * sizeof(float)); // variable to store beamformed high-resolution beamformed image 
		hipMemset(cudaBackEnd::d_bfHR, 0, cudaBackEnd::pixels * sizeof(float));

		hipMalloc((void**)&cudaBackEnd::dev_beamformed_data1, cudaBackEnd::pixels * sizeof(float));// variable to store reshaped beamformed data

		hipMalloc((void**)&cudaBackEnd::d_bfHRBP, sizeof(float) * cudaBackEnd::pixels);// variable to store beamformed high-resolution bandpass filtered data

		////////////// z value////////////////////
		float dz = sample_spacing * samples / Nz;  // depth / (Nz - 1) / 1000;   // spacing in axial (z) direction in mm;
		hipMalloc((void**)&cudaBackEnd::d_z_axis, cudaBackEnd::Nz * sizeof(float));
		range << <cudaBackEnd::Nz / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_z_axis, 0, cudaBackEnd::Nz, dz);
		hipGetLastError();
		hipDeviceSynchronize();

		//////////////////////////////// x value////////////////////////////////
		float dx = aper_len / (cudaBackEnd::Nx - 1);
		// float* d_x_axis = 0;
		hipMalloc((void**)&cudaBackEnd::d_x_axis, cudaBackEnd::Nx * sizeof(float));    // 167.939 us
		range << <cudaBackEnd::Nx / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_x_axis, (-cudaBackEnd::aper_len / 2000), cudaBackEnd::Nx, dx / 1000);
		hipGetLastError();
		hipDeviceSynchronize();

		//////////////// Probe geometry, this info can be taken from transducer file ////////////////////
		//float* d_probe = 0;
		hipMalloc((void**)&cudaBackEnd::d_probe, cudaBackEnd::N_elements * sizeof(float));
		range << <1, cudaBackEnd::N_elements >> > (cudaBackEnd::d_probe, (-cudaBackEnd::aper_len / 2000), cudaBackEnd::N_elements, cudaBackEnd::pitch);
		hipGetLastError();
		hipDeviceSynchronize();

		/////////////////rx aerture calculation using Fnumber///////////////////////////////
		hipMalloc((void**)&cudaBackEnd::d_rx_aperture, cudaBackEnd::Nz * sizeof(float));
		element_division << <cudaBackEnd::Nz / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_z_axis, cudaBackEnd::rx_f_number, cudaBackEnd::Nz, cudaBackEnd::d_rx_aperture);
		hipGetLastError();
		hipDeviceSynchronize();

		////////////////////////rx aerture distance////////
		hipMalloc((void**)&cudaBackEnd::d_rx_ap_distance, cudaBackEnd::channels * cudaBackEnd::Nx * sizeof(float));
		aperture_distance << <cudaBackEnd::Nx * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_x_axis, cudaBackEnd::d_probe, cudaBackEnd::Nx, cudaBackEnd::channels, cudaBackEnd::d_rx_ap_distance);
		hipGetLastError();
		hipDeviceSynchronize();

		///////////////////apodization/////////////////
		// float* d_rx_apod = 0;
		hipMalloc((void**)&cudaBackEnd::d_rx_apod, sizeof(float) * cudaBackEnd::Nz * cudaBackEnd::channels * cudaBackEnd::Nx);
		apodization << <cudaBackEnd::pixels * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_rx_ap_distance, cudaBackEnd::d_rx_aperture, cudaBackEnd::Nz, cudaBackEnd::Nx, cudaBackEnd::channels, cudaBackEnd::pixels, cudaBackEnd::d_rx_apod);
		hipGetLastError();
		hipDeviceSynchronize();

		hipFree(d_rx_aperture);
		hipFree(d_rx_ap_distance);

		/////////////////// calculate central positions transmit subaperture ////////////////////
		hipMalloc((void**)&cudaBackEnd::d_cen_pos, cudaBackEnd::num_frames * sizeof(float));
		Tx_cen_pos << < 1, cudaBackEnd::num_frames >> > (cudaBackEnd::d_cen_pos, cudaBackEnd::N_elements, cudaBackEnd::N_active, cudaBackEnd::pitch, cudaBackEnd::skip_frames, cudaBackEnd::num_frames, cudaBackEnd::d_probe);

		/////////////receive delay calculation /////////////////////////////////////////////
		hipMalloc((void**)&cudaBackEnd::d_rx_delay, cudaBackEnd::pix_cha * sizeof(float));
		receive_delay << < cudaBackEnd::pixels * cudaBackEnd::channels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_probe, cudaBackEnd::d_x_axis, cudaBackEnd::d_z_axis, cudaBackEnd::channels, cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::del_convert, cudaBackEnd::d_rx_delay);
		hipGetLastError();
		hipDeviceSynchronize();

		/////////////////// Transmit delay calculation ////////////////////
		hipMalloc((void**)&cudaBackEnd::d_tx_delay, cudaBackEnd::pixels * cudaBackEnd::num_frames * sizeof(float));
		transmit_delay << < cudaBackEnd::pixels * cudaBackEnd::num_frames / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_x_axis, cudaBackEnd::d_z_axis, cudaBackEnd::d_cen_pos, cudaBackEnd::zd, cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::del_convert, cudaBackEnd::num_frames, cudaBackEnd::d_tx_delay);
		hipGetLastError();
		hipDeviceSynchronize();

		////////////Free cuda memory (one time use) ///////////////////////////
		hipFree(d_probe);
		hipFree(d_x_axis);
		hipFree(d_z_axis);
		hipFree(d_cen_pos);

	}

	catch (std::exception& e)
	{
		return 10;
	}

	return 0;

}

double** cudaBackEnd::computeBModeImgL()
{
	errno_t err;
	FILE* fp;
	if ((err = fopen_s(&fp, "out25.txt", "r")) != 0) {
		//printf("Could not open config file for reading.\n");
		//cv::imwrite("errorMat2.png", testMat0);
		exit(1);
	}
	//------------------------------------------

	// Send a vendor request (bRequest = 0x05) to the device
	cudaBackEnd::ept->Target = TGT_DEVICE;
	cudaBackEnd::ept->ReqType = REQ_VENDOR;
	cudaBackEnd::ept->Direction = DIR_TO_DEVICE;
	cudaBackEnd::ept->ReqCode = 0x05;
	cudaBackEnd::ept->Value = 1;
	cudaBackEnd::ept->Index = 0;
	cudaBackEnd::ept->TimeOut = 100;  // set timeout to 100ms for quick response

	cudaBackEnd::ept_in = USBDevice->BulkInEndPt;
	if (!cudaBackEnd::ept_in) {
		//printf("No IN endpoint??\n");
		exit(1);
	}
	cudaBackEnd::ept_in->MaxPktSize = 16384;
	cudaBackEnd::ept_in->TimeOut = 100;  // set timeout to 100ms for reading

	//-------------------------------------------

	const int TILE_SIZE = 4;
	int MASK_WIDTH		= 364;
	const int MAX_LINE	= 256; 
	static double pix2	= 0.0;
	unsigned char buf[16 * 1024];
	
	int row = 0;  // Keep track of how many rows have been added
	char line[MAX_LINE]; // Max possible line length?

	int iteration = 0;
	int errcount = 0;
	unsigned int addr, data;
	unsigned char recvbuf[2048 * 64 * 2];
	const int MAXROWS = 2040;
	LONG rxlen = MAXROWS * 64 * 2;

	zeroC(cudaBackEnd::rximg, cudaBackEnd::samples * cudaBackEnd::N_elements);   // set rx_img array values to zero.
	double** outArray;

	try
	{
		// unsigned int start = clock();
		while (fgets(line, MAX_LINE, fp)) {
			line[strcspn(line, "\n")] = 0; // Trim trailing newline
			if ((strlen(line) == 0) || (line[0] == ' ') || (line[0] == '#')) {
				//printf("Skipping [%s]\n", line);
			}
			else if (line[0] == 'O') {
				sscanf_s(line, "O %04X %08X ", &addr, &data);
				//printf("Write %08X to Obelix %04X\n", data, addr);
				row = insert_row(buf, row, addr, data);
			}
			else if (line[0] == 'T') {
				sscanf_s(line, "T %04X %08X ", &addr, &data);
				row = insert_row(buf, row, 0x6, 0x40000000 | addr);
				row = insert_row(buf, row, 0x7, data);
				row = insert_row(buf, row, 0x6, 0xC0000000 | addr);
				//printf("Write %08X to TX %04X\n", data, addr);
			}
			else if (line[0] == 'A') {
				sscanf_s(line, "A %04X %08X ", &addr, &data);
				row = insert_row(buf, row, 0x6, 0x00000000 | addr);
				row = insert_row(buf, row, 0x7, data);
				row = insert_row(buf, row, 0x6, 0x80000000 | addr);
				//printf("Write %08X to AFE %04X\n", data, addr);
			}
			else if (line[0] == 'C') {  // CAPTURE
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x01);
				//write_rows(ept, buf, row);  // Send commands
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x10);
				//write_rows(ept, buf, row);  // Send commands
				//wait(100);
				row = insert_row(buf, row, 0x4, 0x00);
				cudaBackEnd::ept_in->Abort();
				cudaBackEnd::ept_in->Reset();
				write_rows(ept, buf, row);  // Send commands
				wait(1);
				if (read_chunk(cudaBackEnd::ept_in, recvbuf, rxlen)) {
					short* rxdata = (short*)(recvbuf);
					for (int i = 0; i < rxlen / 2; i++) {
						if (rxdata[i] >= 512) rxdata[i] -= 1024;
					}
					// Trying to read only first N-1 rows and discard 1st sample
					for (int i = 0; i < 64; i++) {
						for (int j = 0; j < MAXROWS - 1; j++) {
							cudaBackEnd::rximg[i * MAXROWS + j] = rxdata[j * 64 + i + 2];
						}
					}
					//saveToFile(iteration, rxlen, recvbuf);
				}
				else {
					errcount++;
				}
				hipMemcpy(cudaBackEnd::d_data, cudaBackEnd::rximg, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels, hipMemcpyHostToDevice);
				beamformingLR3 << <(cudaBackEnd::pixels / 256) * cudaBackEnd::channels, 256 >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_tx_delay, cudaBackEnd::d_rx_delay, cudaBackEnd::d_data, cudaBackEnd::d_rx_apod, cudaBackEnd::samples, cudaBackEnd::pixels, iteration, cudaBackEnd::num_frames, cudaBackEnd::channels);
				hipGetLastError();
				hipDeviceSynchronize();
				iteration++; // Increment iteration after saving to image
				row = 0;   // Reset buffer for next iteration
			}
			else {
				printf("Don't know how to handle [%s] yet.\n", line);
			}
		}

		//// check for nan values,
		isnan_test_array << <cudaBackEnd::pixels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::d_bfHR, cudaBackEnd::pixels);
		hipGetLastError();
		hipDeviceSynchronize();
		//////////// Bandpass filtering using shared memory /////////////////////
		BPfilter1SharedMem << <(cudaBackEnd::pixels + TILE_SIZE - 1) / TILE_SIZE, TILE_SIZE >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_filt_coeff, cudaBackEnd::pixels, cudaBackEnd::d_bfHRBP);
		hipGetLastError();
		hipDeviceSynchronize();
		//////////////// reshape of the beamformed data ///////////////
		reshape_columnwise << <cudaBackEnd::pixels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::d_bfHRBP);
		hipGetLastError();
		hipDeviceSynchronize();
		hipMemcpy(cudaBackEnd::env, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::Nz * cudaBackEnd::Nx * sizeof(float), hipMemcpyDeviceToHost);
		//char fileout[200];
		//sprintf(fileout, "b_mode_%d.csv", 1); //all the 16 inputs are arranged in a single file
		//csv_write_mat(env, fileout, Nz, Nx);
		// outMat = converttoMat(env, Nz, Nx);
		outArray = convertsingto2darray(cudaBackEnd::env, cudaBackEnd::Nz, cudaBackEnd::Nx);
	}
	catch (std::exception& e)
	{
		return nullptr;
	}

	// For next iteration
	zeroC(cudaBackEnd::rximg, cudaBackEnd::samples * cudaBackEnd::N_elements);   // set rx_img array values to zero.
	hipMemset(cudaBackEnd::d_bfHR, 0, cudaBackEnd::pixels * sizeof(float));
	hipMemset(cudaBackEnd::dev_beamformed_data1, 0, cudaBackEnd::pixels * sizeof(float));

	//////////////// Free cuda memory (that will be used again) ///////////////
	//hipFree(d_data);
	//hipFree(d_bfHR);
	//hipFree(d_tx_delay);
	//hipFree(d_rx_delay);
	//hipFree(d_rx_apod);
	//hipFree(dev_beamformed_data1);
	//hipFree(d_bfHRBP);
	//hipFree(d_filt_coeff);

	return outArray;
}

double** cudaBackEnd::computeBModeImg(int a) {

	std::ofstream mFile;
	mFile.open("sample_output/testcomputeimg.txt");
	mFile << "OK" << std::endl;

	//////////---<H/W INIT>---////////

	CCyUSBDevice* USBDevice;	// H/W initilization1
	CCyControlEndPoint* ept;	// H/W initilization2
	CCyBulkEndPoint* ept_in;	// Endpoint for reading back data
	USBDevice = new CCyUSBDevice(NULL);
	ept = USBDevice->ControlEndPt; // Obtain the control endpoint pointer
	if (!ept) {
		printf("Could not get Control endpoint.\n");
		//return 1;
	}
	// Send a vendor request (bRequest = 0x05) to the device
	ept->Target = TGT_DEVICE;
	ept->ReqType = REQ_VENDOR;
	ept->Direction = DIR_TO_DEVICE;
	ept->ReqCode = 0x05;
	ept->Value = 1;
	ept->Index = 0;
	ept->TimeOut = 100;				// set timeout to 100ms for quick response
	ept_in = USBDevice->BulkInEndPt;
	if (!ept_in) {
		//printf("No IN endpoint??\n");
		return nullptr;
	}
	ept_in->MaxPktSize = 16384;
	ept_in->TimeOut = 100;			// set timeout to 100ms for readin


	//////////-<set reading params>-/////////

	const int MAX_LINE = 256;
	const int N_RX = 64;
	unsigned char buf[16 * 1024];

	errno_t err;
	char line[MAX_LINE]; // Max possible line length?
	FILE* fp;
	if ((err = fopen_s(&fp, "out25_curvi.txt", "r")) != 0) {
		//printf("Could not open config file for reading.\n");
		return nullptr;
	}

	int iteration = 0;
	int errcount = 0;
	int row = 0;					// Keep track of how many rows have been added
	unsigned int addr, data;
	unsigned char recvbuf[2048 * N_RX * 2];
	const int MAXROWS = 2040;
	LONG rxlen = MAXROWS * N_RX * 2;


	//unsigned int start = clock();
	while (fgets(line, cudaBackEnd::MAX_LINE, fp)) {
		line[strcspn(line, "\n")] = 0; // Trim trailing newline
		if ((strlen(line) == 0) || (line[0] == ' ') || (line[0] == '#')) {
		}
		else if (line[0] == 'O') {
			sscanf_s(line, "O %04X %08X ", &addr, &data);
			row = insert_row(buf, row, addr, data);
		}
		else if (line[0] == 'T') {
			sscanf_s(line, "T %04X %08X ", &addr, &data);
			row = insert_row(buf, row, 0x6, 0x40000000 | addr);
			row = insert_row(buf, row, 0x7, data);
			row = insert_row(buf, row, 0x6, 0xC0000000 | addr);
		}
		else if (line[0] == 'A') {
			sscanf_s(line, "A %04X %08X ", &addr, &data);
			row = insert_row(buf, row, 0x6, 0x00000000 | addr);
			row = insert_row(buf, row, 0x7, data);
			row = insert_row(buf, row, 0x6, 0x80000000 | addr);
		}
		else if (line[0] == 'C') {  // CAPTURE STARTS
			row = insert_row(buf, row, 0x4, 0x01);
			row = insert_row(buf, row, 0x4, 0x10);
			row = insert_row(buf, row, 0x4, 0x00);

			ept_in->Abort();
			ept_in->Reset();

			write_rows(ept, buf, row);  // Send commands

			wait(1);
			if (read_chunk(ept_in, recvbuf, rxlen)) {
				short* rxdata = (short*)(recvbuf);
				for (int i = 0; i < rxlen / 2; i++) {
					if (rxdata[i] >= 512) rxdata[i] -= 1024;
				}
				// Trying to read only first N-1 rows and discard 1st sample
				for (int i = 0; i < N_RX; i++) {
					for (int j = 0; j < MAXROWS - 1; j++) {
						//rximg[iteration][i][j] = rxdata[j*64+i+2];
						if (iteration < 29) {      // start from 0 index, so 30-1 
							cudaBackEnd::rximg[i * MAXROWS + j] = rxdata[j * N_RX + i + 2];
						}
						else if (iteration > 91) {
							cudaBackEnd::rximg[(i + 64) * MAXROWS + j] = rxdata[j * N_RX + i + 2];
						}
						else {
							cudaBackEnd::rximg[(i + iteration - 28) * MAXROWS + j] = rxdata[j * N_RX + i + 2];
						}
						//rximg[i * MAXROWS + j] = rxdata[j * N_RX + i + 2];
					}
				}
				//saveToFile(iteration, rxlen, recvbuf);
			}
			else {
				errcount++;
			}

			hipMemcpy(cudaBackEnd::d_data, cudaBackEnd::rximg, sizeof(float) * cudaBackEnd::samples * cudaBackEnd::channels, hipMemcpyHostToDevice);
			beamformingLR3 << <(cudaBackEnd::pixels / 256) * cudaBackEnd::channels, 256 >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_tx_delay, cudaBackEnd::d_rx_delay, cudaBackEnd::d_data, cudaBackEnd::d_rx_apod, cudaBackEnd::samples, cudaBackEnd::pixels, iteration, cudaBackEnd::num_frames, cudaBackEnd::channels);
			hipGetLastError();
			hipDeviceSynchronize();

			iteration++;	// Increment iteration after saving to image
			row = 0;		// Reset buffer for next iteration
		}
		else {
			printf("Don't know how to handle [%s] yet.\n", line);
		}
	}

	mFile << "while loop completed" << std::endl;

	//////////// Bandpass filtering using shared memory /////////////////////
	BPfilter1SharedMem << <(cudaBackEnd::pixels + cudaBackEnd::TILE_SIZE - 1) / cudaBackEnd::TILE_SIZE, cudaBackEnd::TILE_SIZE >> > (cudaBackEnd::d_bfHR, cudaBackEnd::d_filt_coeff, cudaBackEnd::pixels, cudaBackEnd::d_bfHRBP);
	hipGetLastError();
	hipDeviceSynchronize();
	//////////////// reshape of the beamformed data ///////////////
	reshape_columnwise << <cudaBackEnd::pixels / cudaBackEnd::num_threads + 1, cudaBackEnd::num_threads >> > (cudaBackEnd::Nx, cudaBackEnd::Nz, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::d_bfHRBP);
	hipGetLastError();
	hipDeviceSynchronize();
	hipMemcpy(cudaBackEnd::env, cudaBackEnd::dev_beamformed_data1, cudaBackEnd::Nz * cudaBackEnd::Nx * sizeof(float), hipMemcpyDeviceToHost);
	char fileout[200];
	sprintf(fileout, "sample_output/b_curve_mode.csv"); //all the 16 inputs are arranged in a single file
	csv_write_mat(cudaBackEnd::env, fileout, cudaBackEnd::Nz, cudaBackEnd::Nx);

	mFile << "CSV written" << std::endl;

	double** outArray = convertsingto2darray(env, Nz, Nx);
	//////////////// Free cuda memory (that will be used again) ///////////////
	//hipFree(cudaBackEnd::d_data);
	//hipFree(cudaBackEnd::d_bfHR);
	//hipFree(cudaBackEnd::d_tx_delay);
	//hipFree(cudaBackEnd::d_rx_delay);
	//hipFree(cudaBackEnd::d_rx_apod);
	//hipFree(cudaBackEnd::dev_beamformed_data1);

	///////-<free up for next iteration>-/////////
	zeroC(cudaBackEnd::rximg, cudaBackEnd::samples * cudaBackEnd::N_elements);   // set rx_img array values to zero.

	hipMemset(cudaBackEnd::d_bfHR, 0, cudaBackEnd::pixels * sizeof(float));
	hipMemset(cudaBackEnd::dev_beamformed_data1, 0, cudaBackEnd::pixels * sizeof(float));

	mFile.close();
	return outArray;
}
